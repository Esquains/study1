#include "hip/hip_runtime.h"
#include "THCUNN.h"
#include "THCHalf.h"
#include "THCHalfAutoNumerics.cuh"
#include <THC/THCApply.cuh>

template <typename T>
struct logSigmoid_updateOutput_functor
{
  __device__ void operator()(T *output, const T *input) const {
    const T max = fmaxType(0, - *input);
    const T z = THCNumerics<T>::exp(-max) + THCNumerics<T>::exp(-*input -max);
    *output = -(max + THCNumerics<T>::log(z));
    //*output = -THCNumerics<T>::log(1.f + THCNumerics<T>::exp(- *input));
  }
};

template <typename T>
struct logSigmoid_updateGradInput_functor
{
  __device__ void operator()(T *gradInput, const T *input, const T *gradOutput) const {
    const T max = fmaxType(0, *-input);
    const T z = THCNumerics<T>::exp(-max) + THCNumerics<T>::exp(-*input -max);
    T max_deriv = 0;
    T sign = -1;
    if (*input < 0){
        max_deriv = -1;
        sign = 1;
    }
    *gradInput = *gradOutput * (-max_deriv * sign*((z - 1)/z)); 
    //const T z = THCNumerics<T>::exp(- *input);
    //*gradInput = *gradOutput * z / (1.f + z);
  }
};

#ifdef CUDA_HALF_TENSOR
template <>
struct logSigmoid_updateOutput_functor<half> {
  __device__ __forceinline__ void operator()(half* output, const half *input) const {
#ifdef CUDA_HALF_INSTRUCTIONS
    const half one = __float2half(1.f);
    *output = __hneg(THCNumerics<half>::log(one + THCNumerics<half>::exp(__hneg(*input))));
#else
    float in = __half2float(*input);
    *output = __float2half(-THCNumerics<float>::log(1.f + THCNumerics<float>::exp(-in)));
#endif
  }
};

template <>
struct logSigmoid_updateGradInput_functor<half> {
  __device__ __forceinline__ void operator()(half* gradInput, const half *input, const half *gradOutput) const {
#ifdef CUDA_HALF_INSTRUCTIONS
    const half one = __float2half(1.f);
    const half in_exp = THCNumerics<half>::exp(__hneg(*input));
    *gradInput = hdiv(__hmul(*gradOutput, in_exp), __hadd(one, in_exp));
#else
    const float in_exp = THCNumerics<float>::exp(-(__half2float(*input)));
    const float go = __half2float(*gradOutput);
    *gradInput = __float2half(go * in_exp / (1.f + in_exp));
#endif
  }
};
#endif

#include "generic/LogSigmoid.cu"
#include "THCGenerateFloatTypes.h"
