
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "THC/generic/THCTensorMathReduce.cu"
#else

#include <c10/cuda/CUDAException.h>

#if !defined(THC_REAL_IS_BOOL)

void THCTensor_(prod)(THCState* state, THCTensor *self, THCTensor *src, int dimension, int keepdim) {
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, self, src));
  if (!THC_reduceDim<scalar_t>(state, self, src,
                           thrust::identity<accreal>{},
                           ReduceMultiply<accreal>{},
                           thrust::identity<accreal>{},
                           scalar_cast<accreal>(1),
                           dimension,
                           keepdim)) {
    THArgCheck(false, 2, CUTORCH_DIM_WARNING);
  }

  THCudaCheck(cudaGetLastError());
}

#endif

#endif
