
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/SpatialMaxUnpooling.cu"
#else

void THNN_(SpatialMaxUnpooling_updateOutput)(
           THCState *state,
           THCTensor *input,
           THCTensor *output,
           THCIndexTensor *indices,
           int owidth, int oheight)
{
  AT_ERROR("Deprecated");
}

void THNN_(SpatialMaxUnpooling_updateGradInput)(
           THCState *state,
           THCTensor *input,
           THCTensor *gradOutput,
           THCTensor *gradInput,
           THCIndexTensor *indices,
           int owidth, int oheight)
{
  THCUNN_assertSameGPU(state, 4, input, gradOutput, indices, gradInput);
  THCUNN_check_shape_indices(state, indices, input);

  int64_t nInputCols, nInputRows, nInputPlane, batchSize;
  int dimw = 2;
  int dimh = 1;

  if (input->dim() == 3) {
    nInputPlane = input->size(0);
    batchSize = 1;
  }
  else
  {
    ++dimw;
    ++dimh;
    nInputPlane = input->size(1);
    batchSize = input->size(0);
  }
  nInputCols = input->size(dimw);
  nInputRows = input->size(dimh);

  if(owidth!=gradOutput->size(dimw) || oheight!=gradOutput->size(dimh)){
     THError("Inconsistent gradOutput size. oheight= %d, owidth= %d, gradOutput: %dx%d",
             oheight, owidth,gradOutput->size(dimh),gradOutput->size(dimw));
  }

  input = THCTensor_(newContiguous)(state, input);
  indices = THCIndexTensor_(newContiguous)(state, indices);
  gradOutput = THCTensor_(newContiguous)(state, gradOutput);
  THCTensor_(resizeAs)(state, gradInput, input);

  int count = THCTensor_(nElement)(state, input);

  MaxUnpoolBackward <<< GET_BLOCKS(count), CUDA_NUM_THREADS, 0, THCState_getCurrentStream(state) >>>
      (count, THCTensor_(data)(state, gradOutput), THCIndexTensor_(data)(state, indices),
      batchSize, nInputPlane, nInputRows, nInputCols, oheight, owidth, THCTensor_(data)(state, gradInput));
  THCudaCheck(cudaGetLastError());

  // clean
  THCTensor_(free)(state, input);
  THCIndexTensor_(free)(state, indices);
  THCTensor_(free)(state, gradOutput);
}

#endif
