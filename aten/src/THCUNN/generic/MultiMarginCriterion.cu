
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "THCUNN/generic/MultiMarginCriterion.cu"
#else

static inline void THNN_(MultiMarginCriterion_shapeCheck)(
  THCState *state,
  THCTensor *input, THCTensor *target) {
if (input->dim() <= 1) {
int dim = input->dim() == 0 ? 1 : input->size(0);
int target_size = target->dim() == 0 ? 1 : target->size(0);
TORCH_CHECK(!target->is_empty() && (target->dim() <= 1) && (target_size == dim),
  "inconsistent target size: ", target->sizes(), " for input of size: ", input->sizes());
} else if (input->dim() == 2) {
  int nframe = input->size(0);
  int dim = input->size(1);
  TORCH_CHECK((input->size(1) != 0) && (target->dim() == 2) && (target->size(0) == nframe) && (target->size(1) == dim),
  "inconsistent target size: ", target->sizes(), " for input of size: ", input->sizes());
} else {
  TORCH_CHECK(false, "non-empty vector or matrix expected, got size: ", input->sizes());
}
}

// TODO: improve error messages
void THNN_(MultiMarginCriterion_updateOutput)(
           THCState *state,
           THCTensor *input,
           THCIndexTensor *target,
           THCTensor *output,
           int64_t reduction,
           int p,
           THCTensor *weights,
           accreal margin_)
{
  THNN_(MultiMarginCriterion_shapeCheck)(state, input, target);
  if (input->numel() == 0) {
    return;
  }
  scalar_t margin = ScalarConvert<accreal, scalar_t>::to(margin_);
  THCUNN_assertSameGPU(state, 2, input, target);
  input = THCTensor_(newContiguous)(state, input);
  if(weights)
    weights = THCTensor_(newContiguous)(state, weights);
  if (THTensor_nDimensionLegacyNoScalars(input) == 1)
  {
    int nframe = 1;
    THArgCheck(!target->is_empty() && (THTensor_nDimensionLegacyNoScalars(target) == 1) && (THTensor_sizeLegacyNoScalars(target, 0) == nframe), 3,
               "inconsistent target size");
    dim3 blocks(1);
    dim3 threads(MULTIMARGIN_THREADS);
    if (reduction == at::Reduction::None) {
      THCTensor_(resizeAs)(state, output, target);
    } else {
      THCTensor_(resize0d)(state, output);
    }
    if (p == 1)
    {
      cunn_MultiMarginCriterion_updateOutput_kernel<1, scalar_t, accreal> <<<blocks,threads, 0, c10::cuda::getCurrentCUDAStream()>>>(
        THCTensor_(data)(state, output),
        THCTensor_(data)(state, input),
        THCIndexTensor_(data)(state, target),
        weights ? THCTensor_(data)(state, weights) : NULL,
        1, THTensor_sizeLegacyNoScalars(input, 0),
        reduction == at::Reduction::Mean,
        margin
      );
    }
    else if (p == 2)
    {
      cunn_MultiMarginCriterion_updateOutput_kernel<2, scalar_t, accreal> <<<blocks,threads, 0, c10::cuda::getCurrentCUDAStream()>>>(
        THCTensor_(data)(state, output),
        THCTensor_(data)(state, input),
        THCIndexTensor_(data)(state, target),
        weights ? THCTensor_(data)(state, weights) : NULL,
        1, THTensor_sizeLegacyNoScalars(input, 0),
        reduction == at::Reduction::Mean,
        margin
      );
    }
    THCudaCheck(cudaGetLastError());
  }
  else if (input->dim() == 2)
  {
    int nframe = input->size(0);
    // allow zero-dim target for 2D input.
    THArgCheck((input->size(1) != 0) && (THTensor_nDimensionLegacyNoScalars(target) == 1) && (THTensor_sizeLegacyNoScalars(target, 0) == nframe), 3,
               "inconsistent target size");
    dim3 blocks(input->size(0));
    dim3 threads(MULTIMARGIN_THREADS);

    if (reduction == at::Reduction::None)
    {
      THCTensor_(resizeAs)(state, output, target);
      if (p == 1)
      {
        cunn_MultiMarginCriterion_updateOutput_kernel<1, scalar_t, accreal> <<<blocks,threads, 0, c10::cuda::getCurrentCUDAStream()>>>(
          THCTensor_(data)(state, output),
          THCTensor_(data)(state, input),
          THCIndexTensor_(data)(state, target),
          weights ? THCTensor_(data)(state, weights) : NULL,
          nframe, input->size(1),
          false,
          margin
        );
      }
      else if (p == 2)
      {
        cunn_MultiMarginCriterion_updateOutput_kernel<2, scalar_t, accreal> <<<blocks,threads, 0, c10::cuda::getCurrentCUDAStream()>>>(
          THCTensor_(data)(state, output),
          THCTensor_(data)(state, input),
          THCIndexTensor_(data)(state, target),
          weights ? THCTensor_(data)(state, weights) : NULL,
          nframe, input->size(1),
          false,
          margin
        );
      }
      THCudaCheck(cudaGetLastError());
    }
    else
    {
      THCTensor_(resize0d)(state, output);
      THCTensor *output_ = THCTensor_(newWithSize1d)(state, input->size(0));  // tmp output buffer
      if (p == 1)
      {
        cunn_MultiMarginCriterion_updateOutput_kernel<1, scalar_t, accreal> <<<blocks,threads, 0, c10::cuda::getCurrentCUDAStream()>>>(
          THCTensor_(data)(state, output_),
          THCTensor_(data)(state, input),
          THCIndexTensor_(data)(state, target),
          weights ? THCTensor_(data)(state, weights) : NULL,
          nframe, input->size(1),
          reduction == at::Reduction::Mean,
          margin
        );
      }
      else if (p == 2)
      {
        cunn_MultiMarginCriterion_updateOutput_kernel<2, scalar_t, accreal> <<<blocks,threads, 0, c10::cuda::getCurrentCUDAStream()>>>(
          THCTensor_(data)(state, output_),
          THCTensor_(data)(state, input),
          THCIndexTensor_(data)(state, target),
          weights ? THCTensor_(data)(state, weights) : NULL,
          input->size(0), input->size(1),
          reduction == at::Reduction::Mean,
          margin
        );
      }
      THCudaCheck(cudaGetLastError());
      auto t = THTensor_wrap(output_);
      auto r = THTensor_wrap(output);
      at::native::sum_out(r, t, at::IntArrayRef(std::vector<int64_t>{}), false, r.scalar_type());
      THCTensor_(free)(state, output_);
    }
  }
  else
  {
    TORCH_CHECK(false, "Expected 2D input with optional zero batch dim, or 1D input with non-zero dims, but got sizes: ",
    input->sizes());
  }

  THCTensor_(free)(state, input);
  if(weights)
    THCTensor_(free)(state, weights);
}

void THNN_(MultiMarginCriterion_updateGradInput)(
           THCState *state,
           THCTensor *input,
           THCIndexTensor *target,
           THCTensor *gradOutput,
           THCTensor *gradInput,
           int64_t reduction,
           int p,
           THCTensor *weights,
           accreal margin_)
{
  THNN_(MultiMarginCriterion_shapeCheck)(state, input, target);
  input = THCTensor_(newContiguous)(state, input);
  THCTensor_(resizeAs)(state, gradInput, input);
  if (input->numel() == 0) {
    THCTensor_(free)(state, input);
    return;
  }
  scalar_t margin = ScalarConvert<accreal, scalar_t>::to(margin_);
  THCUNN_assertSameGPU(state, 3, input, gradInput, target);
  gradOutput = THCTensor_(newContiguous)(state, gradOutput);

  if(weights)
    weights = THCTensor_(newContiguous)(state, weights);

  if (THTensor_nDimensionLegacyNoScalars(input) == 1)
  {
    dim3 blocks(1);
    dim3 threads(MULTIMARGIN_THREADS);

    if (p == 1)
    {
      cunn_MultiMarginCriterion_updateGradInput_kernel<1, scalar_t, accreal> <<<blocks,threads, 0, c10::cuda::getCurrentCUDAStream()>>>(
        THCTensor_(data)(state, gradInput),
        THCTensor_(data)(state, gradOutput),
        THCTensor_(data)(state, input),
        THCIndexTensor_(data)(state, target),
        weights ? THCTensor_(data)(state, weights) : NULL,
        1, THTensor_sizeLegacyNoScalars(gradInput, 0),
        reduction == at::Reduction::Mean,
        margin,
        reduction != at::Reduction::None
      );
    }
    else if (p == 2)
    {
      cunn_MultiMarginCriterion_updateGradInput_kernel<2, scalar_t, accreal> <<<blocks,threads, 0, c10::cuda::getCurrentCUDAStream()>>>(
        THCTensor_(data)(state, gradInput),
        THCTensor_(data)(state, gradOutput),
        THCTensor_(data)(state, input),
        THCIndexTensor_(data)(state, target),
        weights ? THCTensor_(data)(state, weights) : NULL,
        1, THTensor_sizeLegacyNoScalars(gradInput, 0),
        reduction == at::Reduction::Mean,
        margin,
        reduction != at::Reduction::None
      );
    }
    THCudaCheck(cudaGetLastError());
  }
  else if (input->dim() == 2)
  {
    int nframe = gradInput->size(0);
    THArgCheck((input->size(1) != 0) && (THTensor_nDimensionLegacyNoScalars(target) == 1) && (THTensor_sizeLegacyNoScalars(target, 0) == nframe), 3,
               "inconsistent target size");
    dim3 blocks(gradInput->size(0));
    dim3 threads(MULTIMARGIN_THREADS);

    if (p == 1)
    {
      cunn_MultiMarginCriterion_updateGradInput_kernel<1, scalar_t, accreal> <<<blocks,threads, 0, c10::cuda::getCurrentCUDAStream()>>>(
        THCTensor_(data)(state, gradInput),
        THCTensor_(data)(state, gradOutput),
        THCTensor_(data)(state, input),
        THCIndexTensor_(data)(state, target),
        weights ? THCTensor_(data)(state, weights) : NULL,
        nframe, gradInput->size(1),
        reduction == at::Reduction::Mean,
        margin,
        reduction != at::Reduction::None
      );
    }
    else if (p == 2)
    {
      cunn_MultiMarginCriterion_updateGradInput_kernel<2, scalar_t, accreal> <<<blocks,threads, 0, c10::cuda::getCurrentCUDAStream()>>>(
        THCTensor_(data)(state, gradInput),
        THCTensor_(data)(state, gradOutput),
        THCTensor_(data)(state, input),
        THCIndexTensor_(data)(state, target),
        weights ? THCTensor_(data)(state, weights) : NULL,
        nframe, gradInput->size(1),
        reduction == at::Reduction::Mean,
        margin,
        reduction != at::Reduction::None
      );
    }
    THCudaCheck(cudaGetLastError());
  }
  else
  {
    TORCH_CHECK(false, "Expected 2D input with optional zero batch dim, or 1D input with non-zero dims, but got sizes: ", 
    input->sizes());
  }

  THCTensor_(free)(state, input);
  THCTensor_(free)(state, gradOutput);
  if(weights)
    THCTensor_(free)(state, weights);
}

#endif
