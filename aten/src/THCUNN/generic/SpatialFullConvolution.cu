
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/SpatialFullConvolution.cu"
#else

void THNN_(SpatialFullConvolution_updateOutput)(
           THCState *state,
           THCTensor *input,
           THCTensor *output,
           THCTensor *weight,
           THCTensor *bias,
           THCTensor *columns,
           THCTensor *ones,
           int64_t kW, int64_t kH,
           int64_t dW, int64_t dH,
           int64_t padW, int64_t padH,
           int64_t adjW, int64_t adjH)
{
  THNN_(SpatialFullDilatedConvolution_updateOutput)(
      state, input, output, weight, bias, columns, ones,
      kW, kH, dW, dH, padW, padH, 1, 1, adjW, adjH);
}

void THNN_(SpatialFullConvolution_updateGradInput)(
           THCState *state,
           THCTensor *input,
           THCTensor *gradOutput,
           THCTensor *gradInput,
           THCTensor *weight,
           THCTensor *gradColumns,
           int64_t kW, int64_t kH,
           int64_t dW, int64_t dH,
           int64_t padW, int64_t padH,
           int64_t adjW, int64_t adjH)
{
  THNN_(SpatialFullDilatedConvolution_updateGradInput)(
      state, input, gradOutput, gradInput, weight, gradColumns,
      kW, kH, dW, dH, padW, padH, 1, 1, adjW, adjH);
}


void THNN_(SpatialFullConvolution_accGradParameters)(
           THCState *state,
           THCTensor *input,
           THCTensor *gradOutput,
           THCTensor *gradWeight,
           THCTensor *gradBias,
           THCTensor *columns,
           THCTensor *ones,
           int64_t kW, int64_t kH,
           int64_t dW, int64_t dH,
           int64_t padW, int64_t padH,
           int64_t adjW, int64_t adjH,
           accreal scale_)
{
  THNN_(SpatialFullDilatedConvolution_accGradParameters)(
      state, input, gradOutput, gradWeight, gradBias,
      columns, ones,
      kW, kH, dW, dH, padW, padH, 1, 1, adjW, adjH, scale_);
}

#endif