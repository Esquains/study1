#include "hip/hip_runtime.h"
#include "THCUNN.h"
#include "THCTensor.hpp"
#include "common.h"
#include "THCDeviceTensor.cuh"
#include "THCDeviceTensorUtils.cuh"
#include "THCDeviceUtils.cuh"
#include "THCReduceApplyUtils.cuh"
#include <THC/THCApply.cuh>

#include "THCHalf.h"
#include "THCHalfAutoNumerics.cuh"
#include "THCAtomics.cuh"

template<typename Dtype>
__global__ void SpatialReflectionPadding_updateOutput(
  THCDeviceTensor<Dtype, 4> input,
  THCDeviceTensor<Dtype, 4> output,
  int64_t padT, int64_t padB, int64_t padL, int64_t padR) {

  int64_t outputPointId = threadIdx.x + blockIdx.x * blockDim.x;
  int64_t plane = blockIdx.y;
  int64_t batch = blockIdx.z;
  if (outputPointId >= output.getSize(2) * output.getSize(3)) {
    return;
  }
  int64_t outputPointX = outputPointId % output.getSize(3);
  int64_t outputPointY = outputPointId / output.getSize(3);

  int64_t iStartX = max(0, -padL);
  int64_t iStartY = max(0, -padT);
  int64_t oStartX = max(0, padL);
  int64_t oStartY = max(0, padT);

  int64_t inputPointX = abs(outputPointX - padL)
                  - abs(outputPointX - (input.getSize(3) + padL - 1))
                  - outputPointX
                  + 2 * padL + input.getSize(3) - 1
                  - oStartX + iStartX;

  int64_t inputPointY = abs(outputPointY - padT)
                  - abs(outputPointY - (input.getSize(2) + padT - 1))
                  - outputPointY
                  + 2 * padT + input.getSize(2) - 1
                  - oStartY + iStartY;

  Dtype valueToCopy = input[batch][plane][inputPointY][inputPointX];
  output[batch][plane][outputPointY][outputPointX] = valueToCopy;
}

template <typename Dtype>
__global__ void SpatialReflectionPadding_updateGradInput(
  THCDeviceTensor<Dtype, 4> gradInput,
  THCDeviceTensor<Dtype, 4> gradOutput,
  int64_t padT, int64_t padB, int64_t padL, int64_t padR) {

  int64_t outputPointId = threadIdx.x + blockIdx.x * blockDim.x;
  int64_t plane = blockIdx.y;
  int64_t batch = blockIdx.z;
  if (outputPointId >= gradOutput.getSize(2) * gradOutput.getSize(3)) {
    return;
  }
  int64_t outputPointX = outputPointId % gradOutput.getSize(3);
  int64_t outputPointY = outputPointId / gradOutput.getSize(3);

  int64_t iStartX = max(0, -padL);
  int64_t iStartY = max(0, -padT);
  int64_t oStartX = max(0, padL);
  int64_t oStartY = max(0, padT);

  int64_t inputPointX = abs(outputPointX - padL)
                  - abs(outputPointX - (gradInput.getSize(3) + padL - 1))
                  - outputPointX
                  + 2 * padL + gradInput.getSize(3) - 1
                  - oStartX + iStartX;

  int64_t inputPointY = abs(outputPointY - padT)
                  - abs(outputPointY - (gradInput.getSize(2) + padT - 1))
                  - outputPointY
                  + 2 * padT + gradInput.getSize(2) - 1
                  - oStartY + iStartY;

  Dtype valueToCopy = gradOutput[batch][plane][outputPointY][outputPointX];
  atomicAdd(&gradInput[batch][plane][inputPointY][inputPointX], valueToCopy);
}

#include "generic/SpatialReflectionPadding.cu"
#include "THCGenerateFloatTypes.h"
