#include "hip/hip_runtime.h"
#include "THCUNN.h"
#include "THCTensor.hpp"
#include "THCHalf.h"
#include "THCHalfAutoNumerics.cuh"
#include "THCAtomics.cuh"

#define START_IND(a,b,c) (int64_t)floor((float)(a * c) / b)
#define END_IND(a,b,c) (int64_t)ceil((float)((a + 1) * c) / b)
// #define START_IND(a,b,c) a * c / b
// #define END_IND(a,b,c)  (a + 1) * c / b + ((a + 1) * c % b > 0)?1:0


#define CUDA_MAX_THREADS 1024   // this is safe, in reality 256 is our limit

// 4d tensor B x D x H x W
// All kernels view batch dim B and feature dim D as collapsed.

/*
 * Description:
 *    this function adaptively average pools an input 4D tensor along dimensions 2 and 3
 *    4D input, 4D output
 */
 template <typename T>
__global__ void adaptiveaveragepool(T *input, T *output,
                        int64_t isizeH, int64_t isizeW,
                        int64_t osizeH, int64_t osizeW,
                        int64_t istrideD, int64_t istrideH, int64_t istrideW)
{
  // iterators on output pixels
  int64_t oh, ow;

  // select input/output plane based on thread/block ID
  int64_t o_plane = blockIdx.x;
  int64_t i_plane = o_plane;

  output = output + o_plane*osizeH*osizeW;
  input = input + i_plane*istrideD;

  int64_t ostartH = blockDim.y*blockIdx.y + threadIdx.y;
  int64_t oendH = osizeH;
  const int64_t ostepH = blockDim.y*gridDim.y;

  int64_t ostartW = threadIdx.x;
  int64_t oendW = osizeW;
  const int64_t ostepW = blockDim.x;

  // For all output pixels...
  for(oh = ostartH; oh < oendH; oh += ostepH) {

    int64_t istartH = START_IND(oh, osizeH, isizeH);
    int64_t iendH   = END_IND(oh, osizeH, isizeH);
    int64_t kH = iendH - istartH;

    for(ow = ostartW; ow < oendW; ow += ostepW) {

      int64_t istartW = START_IND(ow, osizeW, isizeW);
      int64_t iendW   = END_IND(ow, osizeW, isizeW);
      int64_t kW = iendW - istartW;

      // Compute the average pooling over corresponding input pixels
      T *ptr_input = input + istartH*istrideH + istartW*istrideW;
      T *ptr_output = output + oh*osizeW + ow;
      T sum = ScalarConvert<int, T>::to(0);
      int64_t ih, iw;
      for(ih = 0; ih < kH; ++ih) {
        for(iw = 0; iw < kW; ++iw) {
          T val = ptr_input[iw*istrideW];
          sum += val;
        }
        ptr_input += istrideH; // next input line
      }
      // Update output
      *ptr_output = sum / kH / kW;
    }
  }
}

/*
 * Description:
 *    this function computes the gradInput from gradOutput
 */
 template <typename T>
__global__ void adaptiveaveragegradinput(
  T *gradInput, T *gradOutput,
  int64_t isizeH, int64_t isizeW, int64_t osizeH, int64_t osizeW
)
{
  // iterators on input pixels
  int64_t ih, iw;

  // select input/output plane based on thread/block ID
  int64_t i_plane = blockIdx.x;
  int64_t o_plane = i_plane;

  gradOutput = gradOutput + o_plane*osizeH*osizeW;
  gradInput = gradInput + i_plane*isizeH*isizeW;

  int64_t istartH = blockDim.y*blockIdx.y + threadIdx.y;
  int64_t iendH = isizeH;
  int64_t istepH = blockDim.y*gridDim.y;

  int64_t istartW = threadIdx.x;
  int64_t iendW = isizeW;
  int64_t istepW = blockDim.x;

  // compute gradInput
  for(ih = istartH; ih < iendH; ih += istepH) {

    int64_t ostartH = START_IND(ih, isizeH, osizeH);
    int64_t oendH   = END_IND(ih, isizeH, osizeH);

    for(iw = istartW; iw < iendW; iw += istepW) {

      int64_t ostartW = START_IND(iw, isizeW, osizeW);
      int64_t oendW   = END_IND(iw, isizeW, osizeW);

      // Compute the gradients over corresponding output pixels
      T *ptr_gradInput = gradInput + ih*isizeW + iw;

      int64_t oh, ow;
      for(oh = ostartH; oh < oendH; ++oh) {
        int64_t kH = START_IND(oh, osizeH, isizeH) - END_IND(oh, osizeH, isizeH);
        for(ow = ostartW; ow < oendW; ++ow) {
          int64_t kW = START_IND(ow, osizeW, isizeW) - END_IND(ow, osizeW, isizeW);
          T grad_delta = gradOutput[ow + oh*osizeW] / kH / kW;
          *ptr_gradInput += grad_delta;
        }
      }
    }
  }
}

/*
 * Description:
 *    this function computes the gradInput from gradOutput
 *    (uses atomic add)
 */
 template <typename T>
__global__ void atomicadaptiveaveragegradinput(
  T *gradInput, T *gradOutput,
  int64_t isizeH, int64_t isizeW, int64_t osizeH, int64_t osizeW
)
{
  // iterators on output indices
  int64_t oh, ow;

  // select input/output plane based on thread/block ID
  int64_t o_plane = blockIdx.x;
  int64_t i_plane = o_plane;

  gradOutput = gradOutput + o_plane*osizeW*osizeH;
  gradInput = gradInput + i_plane*isizeW*isizeH;

  int64_t ostartH = blockDim.y*blockIdx.y + threadIdx.y;
  int64_t oendH = osizeH;
  int64_t ostepH = blockDim.y*gridDim.y;

  int64_t ostartW = threadIdx.x;
  int64_t oendW = osizeW;
  int64_t ostepW = blockDim.x;

  // For all output pixels...
  for(oh = ostartH; oh < oendH; oh += ostepH) {

    int64_t istartH = START_IND(oh, osizeH, isizeH);
    int64_t iendH   = END_IND(oh, osizeH, isizeH);
    int64_t kH = iendH - istartH;

    for(ow = ostartW; ow < oendW; ow += ostepW) {

      int64_t istartW = START_IND(ow, osizeW, isizeW);
      int64_t iendW   = END_IND(ow, osizeW, isizeW);
      int64_t kW = iendW - istartW;

      // Compute the gradients for over corresponding input pixels
      T *ptr_gradInput = gradInput + istartH*isizeW + istartW;
      T *ptr_gradOutput = gradOutput + oh*osizeW + ow;
      T grad_delta = *ptr_gradOutput / kW / kH;

      int64_t ih, iw;
      for(ih = 0; ih < kH; ++ih) {
        for(iw = 0; iw < kW; ++iw) {
          // atomic add since different threads could update same variable
          atomicAdd(&(ptr_gradInput[iw]), grad_delta);
        }
        ptr_gradInput += isizeW; // next input line
      }
    }
  }
}

#include "generic/SpatialAdaptiveAveragePooling.cu"
#include "THCGenerateFloatTypes.h"

#undef CUDA_MAX_THREADS
#undef START_IND
#undef END_IND
