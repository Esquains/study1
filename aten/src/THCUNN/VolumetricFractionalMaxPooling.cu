#include "hip/hip_runtime.h"
#include "THCUNN.h"
#include "common.h"
#include "THCDeviceTensor.cuh"
#include "THCDeviceTensorUtils.cuh"
#include "THCDeviceUtils.cuh"
#include "THCHalf.h"
#include "THCHalfAutoNumerics.cuh"
#include "THCAtomics.cuh"

#include <cfloat>

template <typename Dtype, typename Acctype>
__device__ inline int64_t getInterval(Acctype sample,
                                  int64_t index,
                                  int64_t inputSize,
                                  int64_t outputSize,
                                  int64_t poolSize) {
  Acctype alpha = (Acctype)(inputSize - poolSize) / (Acctype) (outputSize - 1);
  if (index == outputSize - 1) {
    return inputSize - poolSize;
  } else {
    return (int64_t) ((index + sample) * alpha) - (int64_t) (sample * alpha);
  }
}

// We template on poolSizeW to allow the innermost loop to be unrolled
template <int64_t PoolSizeTStatic, typename Dtype, typename Acctype>
__global__ void VolumetricFractionalMaxPooling_updateOutput(
  THCDeviceTensor<Dtype, 5> input,
  THCDeviceTensor<Dtype, 5> output,
  THCDeviceTensor<THCIndex_t, 5> indices,
  THCDeviceTensor<Dtype, 3> samples,
  int64_t poolSizeT, int64_t poolSizeW, int64_t poolSizeH) {

  // Output (h, w) point that this thread is responsible for
  int64_t ourOutputPoint = threadIdx.x + blockIdx.x * blockDim.x;
  int64_t plane = blockIdx.y;
  int64_t batch = blockIdx.z;

  // Each thread generates a specific output point
  if (ourOutputPoint < output.getSize(2) * output.getSize(3) * output.getSize(4)){
    int64_t outputT = ourOutputPoint % output.getSize(4);
    int64_t outputW = (ourOutputPoint / output.getSize(4)) % output.getSize(3);
    int64_t outputH = ourOutputPoint / (output.getSize(3)*output.getSize(4));

    int64_t poolT = getInterval<Dtype, Acctype>(ScalarConvert<Dtype, Acctype>::to(samples[batch][plane][0]), outputT,
                            input.getSize(4), output.getSize(4), poolSizeT);
    int64_t poolW = getInterval<Dtype, Acctype>(ScalarConvert<Dtype, Acctype>::to(samples[batch][plane][1]), outputW,
                            input.getSize(3), output.getSize(3), poolSizeW);
    int64_t poolH = getInterval<Dtype, Acctype>(ScalarConvert<Dtype, Acctype>::to(samples[batch][plane][2]), outputH,
                            input.getSize(2), output.getSize(2), poolSizeH);

    Dtype maxVal = THCNumerics<Dtype>::min();
    int64_t maxIndex = -1;

    for (int64_t h = poolH; h < poolH + poolSizeH; ++h) {
      for (int64_t w = poolW; w < poolW + poolSizeW; ++w) {
        if (PoolSizeTStatic == -1) {
          for (int64_t t = poolT; t < poolT + poolSizeT; ++t) {
            Dtype val = input[batch][plane][h][w][t];
            // for consistency with THNN, favor the first max
            if (val > maxVal) {
              maxIndex = h * input.getSize(3)*input.getSize(4) + w * input.getSize(4) + t;
              maxVal = val;
            }
          }
        } else {
#pragma unroll
          for (int64_t i = 0; i < PoolSizeTStatic; ++i) {
            int64_t t = i + poolT;
            Dtype val = input[batch][plane][h][w][t];
            // for consistency with THNN, favor the first max
            if (val > maxVal) {
              maxIndex = h * input.getSize(3)*input.getSize(4) + w * input.getSize(4) + t;
              maxVal = val;
            }
          }
        }
      }
    }

    assert(THCNumerics<Dtype>::ne(maxVal, THCNumerics<Dtype>::min()));
    assert(maxIndex != -1);

    // +1 for Lua index
    indices[batch][plane][outputH][outputW][outputT] = maxIndex + TH_INDEX_BASE;
    output[batch][plane][outputH][outputW][outputT] = maxVal;
  }
}

template <typename Dtype>
__global__ void VolumetricFractionalMaxPooling_updateGradInput(
  THCDeviceTensor<Dtype, 5> gradInput,
  THCDeviceTensor<Dtype, 5> gradOutput,
  THCDeviceTensor<THCIndex_t, 5> indices) {
  // Output (h, w) point that this thread is responsible for
  int64_t ourOutputPoint = threadIdx.x + blockIdx.x * blockDim.x;
  int64_t plane = blockIdx.y;
  int64_t batch = blockIdx.z;

  // Each thread generates a specific output point
  if (ourOutputPoint < gradOutput.getSize(2) * gradOutput.getSize(3) * gradOutput.getSize(4)) {
    int64_t outputT = ourOutputPoint % gradOutput.getSize(4);
    int64_t outputW = (ourOutputPoint / gradOutput.getSize(4)) % gradOutput.getSize(3);
    int64_t outputH = ourOutputPoint / (gradOutput.getSize(3)*gradOutput.getSize(4));

    int64_t index = indices[batch][plane][outputH][outputW][outputT] - TH_INDEX_BASE;
    assert(index >= 0);
    int64_t inputT = index % gradInput.getSize(4);
    int64_t inputW = (index / gradInput.getSize(4)) % gradInput.getSize(3);
    int64_t inputH = index / (gradInput.getSize(3) * gradInput.getSize(4));
    assert(inputH < gradInput.getSize(2));

    atomicAdd(gradInput[batch][plane][inputH][inputW][inputT].data(),
              gradOutput[batch][plane][outputH][outputW][outputT]);
  }
}

#include "generic/VolumetricFractionalMaxPooling.cu"
#include "THCGenerateFloatTypes.h"
