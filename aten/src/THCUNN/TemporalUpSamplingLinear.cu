#include "hip/hip_runtime.h"
// Adapted from interp.cpp from Caffe util by Pauline Luc
// Originally developed by George Papandreou
#include "THCUNN.h"
#include "THCTensor.hpp"
#include "common.h"
#include "linear_upsampling.h"
#include "THCDeviceTensor.cuh"
#include "THCDeviceTensorUtils.cuh"
#include "THCDeviceUtils.cuh"
#include "THCHalf.h"
#include "THCHalfAutoNumerics.cuh"
#include "THCAtomics.cuh"

template<typename Dtype, typename Acctype>
__global__ void caffe_gpu_interp2_kernel(const int64_t n,
    const Acctype rwidth, const bool align_corners,
    const THCDeviceTensor<Dtype, 3> data1, THCDeviceTensor<Dtype, 3> data2) {
  int64_t index = threadIdx.x + blockIdx.x * blockDim.x;
  const int64_t batchsize = data1.getSize(0);
  const int64_t channels = data1.getSize(1);
  const int64_t width1 = data1.getSize(2);
  const int64_t width2 = data2.getSize(2);

  if (index < n) {
    const int64_t w2 = index % width2;
    // special case: just copy
    if (width1 == width2) {
      const int64_t w1 = w2;
      for (int64_t n = 0; n < batchsize ; n++){
        for (int64_t c = 0; c < channels; ++c) {
          const Dtype val = data1[n][c][w1];
          data2[n][c][w2] = val;
        }
      }
      return;
    }
    //
    const Acctype w1r = linear_upsampling_compute_source_index<Acctype>(rwidth, w2, align_corners);
    const int64_t w1 = w1r;
    const int64_t w1p = (w1 < width1 - 1) ? 1 : 0;
    const Acctype w1lambda = w1r - w1;
    const Acctype w0lambda = Acctype(1) - w1lambda;
    //
    for (int64_t n = 0; n < batchsize ; n++){
        for (int64_t c = 0; c < channels; ++c) {
        const Acctype val = w0lambda * data1[n][c][w1]
                            + w1lambda * data1[n][c][w1+w1p];
        data2[n][c][w2] = ScalarConvert<Acctype, Dtype>::to(val);
      }
    }
  }
}

// Backward (adjoint) operation 1 <- 2 (accumulates)
template <typename Dtype, typename Acctype>
__global__ void caffe_gpu_interp2_kernel_backward(const int64_t n,
    const Acctype rwidth, const bool align_corners,
    THCDeviceTensor<Dtype, 3> data1, const THCDeviceTensor<Dtype, 3> data2){
  int64_t index = threadIdx.x + blockIdx.x * blockDim.x;
  const int64_t batchsize = data1.getSize(0);
  const int64_t channels = data1.getSize(1);
  const int64_t width1 = data1.getSize(2);
  const int64_t width2 = data2.getSize(2);
  if (index < n) {
    const int64_t w2 = index % width2;
    // special case: just copy
    if (width1 == width2) {
      const int64_t w1 = w2;
      for (int64_t n = 0; n < batchsize ; n++){
        for (int64_t c = 0; c < channels; ++c) {
          const Dtype val = data2[n][c][w1];
          data1[n][c][w2] += val;
        }
      }
      return;
    }
    //
    const Acctype w1r = linear_upsampling_compute_source_index<Acctype>(rwidth, w2, align_corners);
    const int64_t w1 = w1r;
    const int64_t w1p = (w1 < width1 - 1) ? 1 : 0;
    const Acctype w1lambda = w1r - w1;
    const Acctype w0lambda = Acctype(1) - w1lambda;
    //
    for (int64_t n = 0; n < batchsize ; n++){
      for (int64_t c = 0; c < channels; ++c) {
        const Dtype d2val = data2[n][c][w2];
        atomicAdd(data1[n][c][w1].data(),
                  ScalarConvert<Acctype, Dtype>::to(w0lambda * d2val));
        atomicAdd(data1[n][c][w1+w1p].data(),
                  ScalarConvert<Acctype, Dtype>::to(w1lambda * d2val));
      }
    }
  }
}


#include "generic/TemporalUpSamplingLinear.cu"
#include "THCGenerateFloatTypes.h"
