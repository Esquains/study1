#include "hip/hip_runtime.h"
#include "THCUNN.h"
#include "THCTensor.hpp"
#include "common.h"
#include "THCReduceApplyUtils.cuh"
#include "THCHalf.h"
#include "THCHalfAutoNumerics.cuh"

#include <thrust/functional.h>

#define MULTILABELMARGIN_THREADS 1024

template <typename Dtype, typename Acctype>
__global__ void cunn_MultiLabelMarginCriterion_updateOutput_kernel(Dtype *output,
                                                                   Dtype *input,
                                                                   THCIndex_t *target,
                                                                   Dtype *istarget,
                                                                   int64_t nframe,
                                                                   int64_t dim,
                                                                   int64_t sizeaverage)
{
  // Temporary sums (for mapreduce)
  __shared__ Acctype sums[MULTILABELMARGIN_THREADS];

  // vectors:
  int64_t k = blockIdx.x;
  Dtype *input_k = input + k*dim;
  THCIndex_t *target_k = target + k*dim;
  Dtype *output_k = output + k;
  Dtype *istarget_k = istarget + k*dim;

  // zero istarget
  for (int64_t d = threadIdx.x; d < dim; d += blockDim.x) {
    istarget_k[d] = ScalarConvert<int, Dtype>::to(0);
  }
  __syncthreads();

  // mark targets in istarget
  if (threadIdx.x == 0) {
    for (int64_t dt = 0; dt < dim; dt++) {
      int64_t target_idx = target_k[dt] - TH_INDEX_BASE;
      if (target_idx < 0) break;
      istarget_k[target_idx] = ScalarConvert<int, Dtype>::to(1);
    }
  }
  __syncthreads();

  // iterate over targets
  Acctype sum = 0;
  for (int64_t dt = 0; dt < dim; dt++) {
    // next target:
    int64_t target_idx = target_k[dt] - TH_INDEX_BASE;
    if (target_idx < 0) break;

    // current value for target
    Dtype input_target_k = input_k[target_idx];

    // compare to all inputs (multithreaded):
    for (int64_t d = threadIdx.x; d < dim; d += blockDim.x) {
      // contribute to loss only if not a target
      if (!ScalarConvert<Dtype, int64_t>::to(istarget_k[d])) {
        Dtype z = 1 - input_target_k + input_k[d];
        if (z > 0)
          sum += z;
      }
    }
  }

  // reduce
  Acctype totalSum = reduceBlock(sums, blockDim.x, sum, thrust::plus<Acctype>(), (Acctype)0);
  if (threadIdx.x == 0) {
    if (sizeaverage) {
      *output_k = ScalarConvert<Acctype, Dtype>::to((totalSum / dim) / nframe);
    } else {
      *output_k = ScalarConvert<Acctype, Dtype>::to(totalSum / dim);
    }
  }
}

template <typename Dtype, typename Acctype>
__global__ void cunn_MultiLabelMarginCriterion_updateGradInput_kernel(Dtype *gradInput,
                                                                      Dtype *gradOutput,
                                                                      Dtype *input,
                                                                      THCIndex_t *target,
                                                                      Dtype *istarget,
                                                                      int64_t nframe,
                                                                      int64_t dim,
                                                                      int64_t sizeaverage,
                                                                      int64_t reduce)
{
  // Temporary sums (for mapreduce)
  __shared__ Acctype sums[MULTILABELMARGIN_THREADS];

  // vectors:
  int64_t k = blockIdx.x;
  Dtype *input_k = input + k*dim;
  Dtype *gradInput_k = gradInput + k*dim;
  THCIndex_t *target_k = target + k*dim;
  Dtype *istarget_k = istarget + k*dim;

  Dtype *gradOutput_k = gradOutput;
  if (!reduce) {
    gradOutput_k += k;
  }

  // gain:
  Dtype g = ScalarConvert<Acctype, Dtype>::to( sizeaverage && reduce ? 1./((Acctype)(nframe*dim)) : 1./((Acctype)dim) );

  // zero gradients:
  for (int64_t d = threadIdx.x; d < dim; d += blockDim.x) {
    gradInput_k[d] = ScalarConvert<int, Dtype>::to(0);
  }
  __syncthreads();

  // iterate over targets
  for (int64_t dt = 0; dt < dim; dt++) {
    // next target:
    int64_t target_idx = (int64_t)target_k[dt] - TH_INDEX_BASE;
    if (target_idx < 0) break;

    // current value for target
    Dtype input_target_k = input_k[target_idx];

    // compare to all inputs (multithreaded):
    Acctype sum = 0;
    for (int64_t d = threadIdx.x; d < dim; d += blockDim.x) {
      // contribute to loss only if not a target
      if (!ScalarConvert<Dtype, int64_t>::to(istarget_k[d])) {
        Dtype z = 1 - input_target_k + input_k[d];
        if (z > 0) {
          sum -= g;
          gradInput_k[d] += g;
        }
      }
    }
    __syncthreads();

    // reduce sum
    Acctype totalSum = reduceBlock(sums, blockDim.x, sum, thrust::plus<Acctype>(), (Acctype)0);
    if (threadIdx.x == 0) {
      gradInput_k[target_idx] += ScalarConvert<Acctype, Dtype>::to(totalSum);
    }
  }

  for (int64_t d = threadIdx.x; d < dim; d += blockDim.x) {
    gradInput_k[d] *= *gradOutput_k;
  }
}

#include "generic/MultiLabelMarginCriterion.cu"
#include "THCGenerateFloatTypes.h"

#undef MULTILABELMARGIN_THREADS
