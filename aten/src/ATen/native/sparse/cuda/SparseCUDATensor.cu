#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/NativeFunctions.h>
#include <ATen/SparseTensorUtils.h>
#include <ATen/native/sparse/cuda/SparseCUDAApplyUtils.cuh>
#include <ATen/AccumulateType.h>
#include <ATen/cuda/CUDAApplyUtils.cuh>
#include <ATen/native/cuda/Loops.cuh>

#include <THC/THCThrustAllocator.cuh>
#include <THC/THCTensorSort.cuh>

#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <thrust/gather.h>
#include <thrust/generate.h>
#include <thrust/scan.h>
#include <thrust/sequence.h>
#include <thrust/sort.h>
#include <thrust/transform.h>
#include <thrust/unique.h>
#if CUDA_VERSION >= 7000
#include <thrust/system/cuda/execution_policy.h>
#endif

namespace at { namespace native {

using namespace at::sparse;

// --------------------------------------------------------------------
// see NOTE [Sparse Coalesce]
//
// coalesce sum
// --------------------------------------------------------------------
std::tuple<Tensor, Tensor, Tensor, Tensor, Tensor, int64_t> sparse_coalesce_common_cuda(const SparseTensor& self) {
  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  auto allocator = THCThrustAllocator(globalContext().lazyInitCUDA());
  auto policy = thrust::cuda::par(allocator).on(stream);

  // For indices, a simple sort + unique suffices
  // For values, we use a custom kernel for segmented reduction (can't use Thrust due to indirection).

  int64_t nnz = self._nnz();
  Tensor values = self._values();
  IntList sizes = self.sizes();
  int64_t sparse_dim = self.sparse_dim();
  int64_t dense_dim = self.dense_dim();

  // indices will be modified by Thrust, so we have to clone or use new storage
  // here.
  LongTensor indices1D = flatten_indices(self._indices(), self.sizes(), true);
  LongTensor origIndices = at::empty({nnz}, self._indices().options());
  LongTensor uniqueOffsets = at::empty({nnz}, self._indices().options());

  typedef thrust::device_ptr<int64_t> thrust_ptr;
  thrust_ptr indicesIter(indices1D.data<int64_t>());
  thrust_ptr origIndicesIter(origIndices.data<int64_t>());
  thrust_ptr uniqueOffsetsIter(uniqueOffsets.data<int64_t>());

  // Fill sortedOrigIndices with sequential indices, so that
  //  origIndicesIter = uniqueOffsetsIter = (0, 1, 2, ..., nnz)
  thrust::counting_iterator<int64_t> countIterI(TH_INDEX_BASE);
  thrust::counting_iterator<int64_t> countIterO(TH_INDEX_BASE);
  thrust::copy(policy, countIterI, countIterI + nnz, origIndicesIter);
  thrust::copy(policy, countIterO, countIterO + nnz, uniqueOffsetsIter);

  thrust::sort_by_key(policy,
    indicesIter, indicesIter + nnz,
    origIndicesIter, ThrustLTOp<int64_t>()
  );

  // after unique_by_key, uniqueOffsetsIter holds strided indices, where
  // difference of two indices = #same consecutive indices
  thrust::pair<thrust_ptr, thrust_ptr> newEnd = thrust::unique_by_key(policy,
    indicesIter, indicesIter + nnz,
    uniqueOffsetsIter
  );
  int64_t new_nnz = newEnd.first - indicesIter;

  indices1D.resize_({1, new_nnz});
  auto newValues_size = values.sizes().vec();
  newValues_size[0] = new_nnz;
  Tensor newValues = at::empty(newValues_size, values.options());

  // unflatten indices if necessary
  LongTensor newIndices;
  if (sparse_dim == 1) {
    newIndices = indices1D;
  } else {
    newIndices = at::empty({sparse_dim, new_nnz}, origIndices.options());
    if (TH_INDEX_BASE != 0) {
      indices1D.add_(-1);
    }
    for (int64_t d = sparse_dim - 1; d >= 0; d--) {
      // NB: Not a select, so I can preserve the outer dimension
      LongTensor indicesSlice = newIndices.narrow(0, d, 1);
      // Note for the porting guide: THCTensor_(copy) does NOT do normal
      // broadcasting logic; instead, it will blast the elements from one
      // to the other so long as the numel is the same
      indicesSlice.copy_(indices1D);
      indices1D.div_(self.size(d));
      indicesSlice.add_(indices1D, -self.size(d));
    }
    if (TH_INDEX_BASE != 0) {
      indices1D.add_(1); // "lol"
    }
  }

  return std::tuple<Tensor, Tensor, Tensor, Tensor, Tensor, int64_t>(uniqueOffsets, origIndices, newValues, newIndices, indices1D, new_nnz);
}

SparseTensor coalesce_sum_cuda(const SparseTensor& self) {
  int64_t nnz = self._nnz();

  if (self.is_coalesced()) {
    return self;
  }

  // NOTE: Since `coalesce` is not an in-place operation when `is_coalesced` is false,
  // we should keep the original tensor intact and do coalesce on a copy of the tensor
  if (nnz < 2) {
    SparseTensor out = self.clone();
    out._coalesced_(true);
    return out;
  }

  int64_t sparse_dim = self.sparse_dim();
  int64_t dense_dim = self.dense_dim();
  IntList sizes = self.sizes();

  Tensor uniqueOffsets, origIndices, newValues, newIndices, indices1D;
  int64_t new_nnz = 0;

  std::tie(uniqueOffsets, origIndices, newValues, newIndices, indices1D, new_nnz) = sparse_coalesce_common_cuda(self);

  Tensor values = self._values().contiguous();
  int64_t stride = at::prod_intlist(values.sizes().slice(1));

  // If there is no values to copy, save running the kernel.
  if (newValues.numel() > 0) {
    hipStream_t stream = at::cuda::getCurrentCUDAStream();
    auto allocator = THCThrustAllocator(globalContext().lazyInitCUDA());
    auto policy = thrust::cuda::par(allocator).on(stream);

    dim3 grid(THCCeilDiv(new_nnz, (int64_t) 4), THCCeilDiv(stride, (int64_t) 128));
    dim3 block(32, 4);
    AT_DISPATCH_ALL_TYPES_AND_HALF(values.type(), "coalesce_sum_cuda", [&] {
      apply::coalesce_sum_kernel<scalar_t><<<grid, block, 0, stream>>>(
        uniqueOffsets.data<int64_t>(),
        origIndices.data<int64_t>(),
        values.data<scalar_t>(),
        newValues.data<scalar_t>(),
        nnz,
        new_nnz,
        stride
      );
    });
  }

  SparseTensor out = at::_sparse_coo_tensor_with_dims_and_tensors(sparse_dim, dense_dim, sizes, newIndices, newValues, self.options());
  out._coalesced_(true);

  THCudaCheck(hipGetLastError());
  return out;
}

SparseTensor coalesce_sparse_cuda(const SparseTensor& self) {
  return coalesce_sum_cuda(self);
}

// --------------------------------------------------------------------
// see NOTE [Sparse Coalesce]
//
// coalesce max / min
// --------------------------------------------------------------------
std::tuple<SparseTensor, Tensor> coalesce_maxmin_common_cuda(const SparseTensor& self, CoalesceReductionType reduction_type) {
  int64_t nnz = self._nnz();
  LongTensor indices = self._indices();
  Tensor values = self._values().contiguous();
  // see NOTE [Reduction Indices at Coalesce]
  LongTensor reduction_indices;

  if (self.is_coalesced()) {
    reduction_indices = at::arange(0, nnz, indices.options()).reshape({nnz, 1}).repeat({1, values.stride(0)});
    return std::tuple<SparseTensor, Tensor>(self, reduction_indices);
  }

  if (nnz < 2) {
    reduction_indices = at::arange(0, nnz, indices.options()).reshape({nnz, 1}).repeat({1, values.stride(0)});
    // see NOTE [Coalesce SparseTensor]
    SparseTensor out = self.clone();
    out._coalesced_(true);
    return std::tuple<SparseTensor, Tensor>(out, reduction_indices);
  }

  int64_t sparse_dim = self.sparse_dim();
  int64_t dense_dim = self.dense_dim();
  IntList sizes = self.sizes();

  Tensor uniqueOffsets, origIndices, newValues, new_indices, indices1D;
  int64_t new_nnz = 0;

  std::tie(uniqueOffsets, origIndices, newValues, new_indices, indices1D, new_nnz) = sparse_coalesce_common_cuda(self);
  reduction_indices = at::empty({new_nnz, values.stride(0)}, new_indices.options());

  int64_t stride = at::prod_intlist(values.sizes().slice(1));

  // If there is no values to copy, save running the kernel.
  if (newValues.numel() > 0) {
    hipStream_t stream = at::cuda::getCurrentCUDAStream();
    auto allocator = THCThrustAllocator(globalContext().lazyInitCUDA());
    auto policy = thrust::cuda::par(allocator).on(stream);

    dim3 grid(THCCeilDiv(new_nnz, (int64_t) 4), THCCeilDiv(stride, (int64_t) 128));
    dim3 block(32, 4);
    AT_DISPATCH_ALL_TYPES_AND_HALF(values.type(), "coalesce_maxmin_common_cuda", [&] {
      if (reduction_type == CoalesceReductionType::MAX) {
        apply::coalesce_max_kernel<scalar_t><<<grid, block, 0, stream>>>(
          uniqueOffsets.data<int64_t>(),
          origIndices.data<int64_t>(),
          reduction_indices.data<int64_t>(),
          values.data<scalar_t>(),
          newValues.data<scalar_t>(),
          nnz,
          new_nnz,
          stride
        );
      }
      else if (reduction_type == CoalesceReductionType::MIN) {
        apply::coalesce_min_kernel<scalar_t><<<grid, block, 0, stream>>>(
          uniqueOffsets.data<int64_t>(),
          origIndices.data<int64_t>(),
          reduction_indices.data<int64_t>(),
          values.data<scalar_t>(),
          newValues.data<scalar_t>(),
          nnz,
          new_nnz,
          stride
        );
      }
      else {
        AT_ERROR("expected CoalesceReductionType MAX and MIN, but other type is found.");
      }
    });
  }

  SparseTensor out = at::_sparse_coo_tensor_with_dims_and_tensors(
    sparse_dim, dense_dim, sizes,
    new_indices, newValues, self.options()
  )._coalesced_(true);

  THCudaCheck(hipGetLastError());

  return std::tuple<SparseTensor, Tensor>(out, reduction_indices);
}

std::tuple<SparseTensor, Tensor> coalesce_max_cuda(const SparseTensor& self) {
  return coalesce_maxmin_common_cuda(self, CoalesceReductionType::MAX);
}

std::tuple<SparseTensor, Tensor> coalesce_min_cuda(const SparseTensor& self) {
  return coalesce_maxmin_common_cuda(self, CoalesceReductionType::MIN);
}

}} // namespace at::native
