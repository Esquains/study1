#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/Dispatch.h>
#include <ATen/native/Resize.h>
#include <ATen/cuda/HIPContext.h>

namespace at {
namespace native {

namespace {

template <typename input_t, typename output_t>
__global__ void convert_indices_from_coo_to_csr_cuda_kernel(output_t* data_out, const input_t* data_in, const int64_t size, const int64_t numel) {
  int64_t tid = blockDim.x * blockIdx.x + threadIdx.x;
  if (tid == 0) {
    for (int64_t i = 0; i <= data_in[0]; i++)
      data_out[i] = static_cast<output_t>(0);
  } else if (tid < numel) {
    for (int64_t i = data_in[tid - 1]; i < data_in[tid]; i++)
      data_out[i + 1] = static_cast<output_t>(tid);
  } else if (tid == numel) {
    for (int64_t i = data_in[numel - 1] + 1; i < size + 1; i++)
      data_out[i] = static_cast<output_t>(numel);
  }
}

template <typename input_t, typename output_t>
void convert_indices_from_coo_to_csr_cuda(const Tensor& result, const Tensor& input, const int64_t size) {
  int64_t numel = input.numel();
  const input_t* data_in = input.data_ptr<input_t>();
  output_t* data_out = result.data_ptr<output_t>();

  if (numel == 0) {
    result.zero_();
    return;
  }

  // Run (numel + 1) threads...
  int64_t THREADS = at::cuda::getCurrentDeviceProperties()->maxThreadsPerBlock;
  int64_t BLOCKS = (numel + THREADS) / THREADS;
  at::cuda::CUDAStream stream = at::cuda::getCurrentCUDAStream();
  convert_indices_from_coo_to_csr_cuda_kernel<<<BLOCKS, THREADS, 0, stream>>>(data_out, data_in, size, numel);
  C10_CUDA_KERNEL_LAUNCH_CHECK();
}

void dispatch(const Tensor& result, const Tensor& input, const int64_t size, const bool out_int32) {
  if (!out_int32) {
    AT_DISPATCH_INTEGRAL_TYPES(input.scalar_type(), "convert_indices_from_coo_to_csr_cuda", [&] {
      convert_indices_from_coo_to_csr_cuda<scalar_t, int64_t>(result, input, size);
    });
  } else {
    AT_DISPATCH_INTEGRAL_TYPES(input.scalar_type(), "convert_indices_from_coo_to_csr_cuda", [&] {
      convert_indices_from_coo_to_csr_cuda<scalar_t, int>(result, input, size);
    });
  }
}

} // namespace

TORCH_IMPL_FUNC(_convert_indices_from_coo_to_csr_structured_cuda) (
  const Tensor& self, const int64_t size, const bool out_int32, const Tensor& result
) {
  dispatch(result, self, size, out_int32);
}

} // namespace native
} // namespace at
