#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/InitialTensorOptions.h>
#include <ATen/NativeFunctions.h>
#include <ATen/cuda/CUDAApplyUtils.cuh>
#include <ATen/cuda/HIPContext.h>
#include <ATen/native/TensorFactories.h>
#include <ATen/native/cuda/Resize.cuh>
#include <c10/util/Exception.h>

#include <THC/THCGeneral.h>
#include <THC/THCThrustAllocator.cuh>
#include <THC/THCTensorRandom.h>
#include <thrust/device_ptr.h>
#include <thrust/sort.h>
#include <thrust/execution_policy.h>
#include <thrust/sequence.h>
#include <thrust/binary_search.h>

#include <algorithm>
#include <cstddef>
#include <cmath>

THCGenerator* THCRandom_getGenerator(THCState* state);
// This is to keep thread safety in hiprand with hiprandStateMtgp32
int const threadsPerBlock = 256;

namespace at {
namespace native {

Tensor& eye_out_cuda(Tensor& result, int64_t n) {
  return at::native::eye_out_cuda(result, n, /*m=*/-1);
}

Tensor& eye_out_cuda(Tensor& result, int64_t n, int64_t m) {
  TORCH_CHECK(n >= 0, "n must be greater or equal to 0, got ", n);

  if(m < 0) {
    m = n;
  }

  result.resize_({n, m});
  result.zero_();

  int64_t sz = std::min<int64_t>(n, m);
  int64_t stride = result.stride(0) + result.stride(1);

  Tensor diag = result.as_strided({sz}, {stride});
  diag.fill_(1);
  return result;
}

Tensor empty_cuda(IntArrayRef size, const TensorOptions& options) {
  AT_ASSERT(options.backend() == at::Backend::CUDA);
  AT_ASSERT(!options.is_variable());  // is_variable should have been 'unpacked'  // TODO: remove this when Variable and Tensor are merged
  TORCH_CHECK(!options.pinned_memory(), "Only dense CPU tensors can be pinned");
  check_size_nonnegative(size);

  auto* allocator = at::cuda::getCUDADeviceAllocator();
  int64_t nelements = prod_intlist(size);
  auto dtype = options.dtype();
  auto storage_impl = c10::make_intrusive<StorageImpl>(
    dtype,
    nelements,
    allocator->allocate(nelements * dtype.itemsize()),
    allocator,
    /*resizeable=*/true);

  auto tensor = detail::make_tensor<TensorImpl>(storage_impl, CUDATensorId());
  // Default TensorImpl has size [0]
  if (size.size() != 1 || size[0] != 0) {
    tensor.unsafeGetTensorImpl()->set_sizes_contiguous(size);
  }
  return tensor;
}

Tensor empty_strided_cuda(IntArrayRef size, IntArrayRef stride, const TensorOptions& options) {
  auto t = at::native::empty_cuda({0}, options);
  at::native::resize_impl_cuda_(t.unsafeGetTensorImpl(), size, stride);
  return t;
}

Tensor& randperm_out_cuda(Tensor& result, int64_t n, Generator* generator) {
  TORCH_CHECK(n >= 0, "n must be non-negative, got", n);
  TORCH_CHECK(at::scalar_tensor(n, result.options()).defined(),
  "n is too large for result tensor type: '", result.type().toString(), "'");

  result.resize_({n});

  if (result.scalar_type() == at::ScalarType::Half) {
    auto result_float = at::empty({n}, initialTensorOptions().device(Device(DeviceType::CUDA)));
    result.copy_(randperm_out_cuda(result_float, n, generator));
  } else {
    if (n < 30000) {  // For small inputs, we offload it to CPU instead.
      auto result_cpu = at::empty({n}, result.options().device(kCPU));
      randperm_out(result_cpu, n, generator);
      result.copy_(result_cpu);
    } else {
      // Generate random values for the keys array
      AT_DISPATCH_ALL_TYPES(
        result.scalar_type(), "randperm_out_cuda", [&] {
          auto keys = at::empty(result.sizes(), result.options()).random_(generator);

          auto result_data = thrust::device_ptr<scalar_t>(result.data<scalar_t>());
          auto keys_data = thrust::device_ptr<scalar_t>(keys.data<scalar_t>());

          auto state = globalContext().getTHCState();
          THCThrustAllocator thrustAlloc(state);
          auto policy = thrust::cuda::par(thrustAlloc).on(at::cuda::getCurrentCUDAStream());

          thrust::sequence(policy, result_data, result_data + n);

          // Use the sorted order of keys to rearrange the result array
          thrust::sort_by_key(policy, keys_data, keys_data + n, result_data);
        }
      );
    }
  }

  return result;
}

// ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~ triangle ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~

namespace {
// To find the max integer that does not exceed the root of an int64_t variable,
// we could use a loop to test one bit at a time, which takes up to 31
// iterations. This would give the accurate result, but is relatively slow and
// is an overkill for most cases where double's precision suffice.
//
// If we directly use sqrt to calculate the root, the convertion from int64_t
// to double would lose 11 bits precision.
//
// The following solution uses sqrt directly for most cases, and would only
// special handle it if there is indeed precision loss.
__device__
inline int64_t resolve_root_int(
    int64_t b, int64_t cX4, int64_t x, int32_t sign) {
  int64_t bXb_cX4 = b*b - cX4;
  // potential precision loss could occur here when casting int64_t (63 bits
  // precision) to double (52 bits precision)
  double sr = ::sqrt((double)bXb_cX4);
  int64_t res = ::__double2ll_rd((-b + sign * sr)/2);

  // have to cast double to int64_t, otherwise it would only compare up to the
  // precision of a double variable, ignoring the precision loss
  if (bXb_cX4 != (int64_t) (sr * sr)) {
    // handle precision loss by using binary search
    int64_t llsr = ::__double2ll_rd(sr);
    // Use the following math to reduce search space.
    // Suppose z is the accurate result of sqrt(bXb_cX4) without precision loss
    // let d = abs(bXb_cX4 - llsr * llsr), then we have:
    // z = sqrt(bXb_cX4) <= sqrt(llsr * llsr + d) <= llsr + sqrt(d)
    // z = sqrt(bXb_cX4) >= sqrt(llsr * llsr - d) >= llsr - sqrt(d)
    // Hence, it is sufficient to search range [llsr - sqrt(d), llsr + sqrt(d)).
    // And the true value of row would also be with in range,
    //            [res - sqrt(d), res + sqrt(d) + 1)
    // as the denominator would only reduce the precision penalty.
    int64_t diff =
      ::__double2ll_ru(::sqrt(::fabs((double)(bXb_cX4 - llsr * llsr))));
    // l never exceeds (could equal to) the target row index
    auto l = res > diff ? res - diff : 0;
    // r is always larger than the target row index
    auto r = res + diff + 1;

    // binary search for the correct answer
    x <<= 1; // the loop always compares with 2x, so do it once here
    while (l + 1 < r) {
      auto m = (l + r) >> 1;
      // for tril:
      //    b = 2f - 1, sign = 1, hence (2f + m - 1) * m / 2
      // for triu:
      //    b = -2f - 1, sign = -1, hence (2f - m + 1) * m / 2
      if (sign * (b + m) * m > x) {
        r = m;
      } else {
        l = m;
      }
    }
    res = l;
  }

  return res;
}

// f: the number of elements in the first row of the trapezoid.
// x: the index of the target coordinates ordered by row and then column.
//
// View the tril as a top trapezoid stacked on a bottom rectangle. Assume x
// corresponds to the coordinate (row, col) in the trapezoid, where the row and
// the col both start from 0, then we have:
//
//                   (f + f + row - 1) * row / 2 <= x                       [1]
//                 (f + f + row) * (row + 1) / 2  > x                       [2]
//
// Therefore, row is the maximum integer satisfying the following inequality:
//
//                       (row + 2f - 1)row <= 2x
//                  row^2 + (2f-1)row - 2x <= 0.                            [3]
//
// Based on ineuqality [3], we have the following coefficients for formula of
// root:
//                               a = 1
//                               b = 2f - 1
//                               c = -2x
// There are two roots, and we should use the largest integer that does not
// exceed the root on the right. Intuitively, it is because:
//  i)  the valid solution range of row is between two roots, as it is <= 0;
//  ii) as we count in more rows, the total # of elements should always
//      increase, hence so does the left-hand side row^2 + (2f-1)row - 2x.
//      Therefore, the valid range of row lies in between the nadir point and
//      the larger root on the right.
// Full proof can be derived from inequality [2]. So, we calculate the result
// coordinate as:
//
//                   row = floor((-b + sqrt(b^2 - 4c)) / 2)
//                   col = x - (f + f + row - 1) * row / 2
__device__
inline void get_coordinate_in_tril_trapezoid(
    int64_t f, int64_t x, int64_t & row, int64_t & col) {
  f <<= 1; // all statements use 2f, so only calculate it once here.
  auto b = f - 1;
  auto cX4 = - (x << 3); // 4 * c = 4 * (-2x) = -8x;
  row = resolve_root_int(b, cX4, x, 1);
  col = x - ((f + row - 1) * row >> 1);
}

// f: the number of elements in the first row of the bottom trapezoid.
// x: the index of the target coordinates ordered by row and then column.
//
// View the triu as a top rectangle stacked on a bottom trapezoid, where the
// trapezoid is upside down. Assume x corresponds to the coordinate (row, col)
// in the bottom trapezoid, where the row and the col start from 0, then we
// have:
//
//                   (f + f - row + 1) * row / 2 <= x                       [1]
//                 (f + f - row) * (row + 1) / 2  > x                       [2]
//
// Therefore, row is the maximum integer satisfying the following inequality:
//
//                       (-row + 2f + 1)row <= 2x
//                   row^2 - (2f+1)row + 2x >= 0.                           [3]
//
// Based on ineuqality [3], we have the following coefficients for formula of
// root:
//                               a = 1
//                               b = -1 - 2f
//                               c = 2x
// There are two roots, and we should use the largest integer that does not
// exceed the root on the left. Intuitively, it is because:
//  i)  the valid solution range of row is outside of the two roots, as it is <
//      > 0;
//  ii) as we count in more rows, the total # of elements should always
//      increase, hence so does the left-hand side row^2 - (2f+1)row + 2x.
//      Therefore, the valid range of row lies to the left of the smaller root
//      on the left.
// Full proof can be derived from inequality [2]. So, we calculate the result
// coordinate as:
//
//                   row = floor((-b - sqrt(b^2 - 4c)) / 2)
//                   col = x - (f + f - row + 1) * row / 2
__device__
inline void get_coordinate_in_triu_trapezoid(
    int64_t f, int64_t x, int64_t & row, int64_t & col) {
  f <<= 1; // all statements use 2f, so only calculate it once here.
  auto b = -1 - f;
  auto cX4 = x << 3; // 4 * c = 4 * (2x) = 8x;
  row = resolve_root_int(b, cX4, x, -1);
  col = x - ((f - row + 1) * row >> 1) + row;
}

} // namespace

template <typename scalar_t>
__global__
#ifdef __HIP_PLATFORM_HCC__
C10_LAUNCH_BOUNDS_1(512)
#endif
void tril_indices_kernel(scalar_t * tensor,
                         int64_t row_offset,
                         int64_t m_first_row,
                         int64_t col,
                         int64_t trapezoid_size,
                         int64_t tril_size) {
  int64_t linear_index = blockIdx.x * blockDim.x + threadIdx.x;

  if (linear_index < tril_size) {
    int64_t r, c;
    if (linear_index < trapezoid_size) {
      // the coordinate is within the top trapezoid
      get_coordinate_in_tril_trapezoid(m_first_row, linear_index, r, c);
    } else {
      // the coordinate falls in the bottom rectangle
      auto surplus = linear_index - trapezoid_size;
      // add the height of trapezoid: m_last_row (col) - m_first_row + 1
      r = surplus / col + col - m_first_row + 1;
      c = surplus % col;
    }
    r += row_offset;

    tensor[linear_index] = r;
    tensor[linear_index + tril_size] = c;
  }
}

// Some Large test cases for the fallback binary search path is disabled by
// default to speed up CI tests and to avoid OOM error. When modifying the
// implementation, please enable them in test/test_cuda.py and make sure they
// pass on your local server.
Tensor tril_indices_cuda(
    int64_t row, int64_t col, int64_t offset, const TensorOptions& options) {
  check_args(row, col, options);

  auto tril_size = get_tril_size(row, col, offset);
  auto tensor = empty_cuda({2, tril_size}, options);

  if (tril_size > 0) {
    auto m_first_row = offset > 0 ?
      std::min<int64_t>(col, 1 + offset) : // upper bounded by col
      row + offset > 0; // either 0 or 1
    auto trapezoid_row_offset = std::max<int64_t>(0, -offset);
    auto rectangle_row_offset = trapezoid_row_offset + col - m_first_row + 1;
    int64_t rectangle_size = 0;
    if (rectangle_row_offset < row) {
      rectangle_size = (row - rectangle_row_offset) * col;
    }

    dim3 dim_block = cuda::getApplyBlock();
    dim3 dim_grid;
    // using tril_size instead of tensor.numel(), as each thread takes care of
    // two elements in the tensor.
    TORCH_CHECK(
      cuda::getApplyGrid(tril_size, dim_grid, tensor.get_device()),
      "unable to get dim grid");

    AT_DISPATCH_ALL_TYPES_AND(at::ScalarType::Half, tensor.scalar_type(), "tril_indices_cuda", [&] {
      tril_indices_kernel<<<
          dim_grid, dim_block, 0, at::cuda::getCurrentCUDAStream()>>>(
        tensor.data<scalar_t>(),
        trapezoid_row_offset,
        m_first_row,
        col,
        tril_size - rectangle_size,
        tril_size);
    });
  }

  return tensor;
}

template <typename scalar_t>
__global__
void triu_indices_kernel(scalar_t * tensor,
                         int64_t col_offset,
                         int64_t m_first_row,
                         int64_t col,
                         int64_t rectangle_size,
                         int64_t triu_size) {
  int64_t linear_index = blockIdx.x * blockDim.x + threadIdx.x;

  if (linear_index < triu_size) {
    int64_t r, c;
    if (linear_index < rectangle_size) {
      // the coordinate is within the top rectangle
      r = linear_index / col;
      c = linear_index % col;
    } else {
      // the coordinate falls in the bottom trapezoid
      get_coordinate_in_triu_trapezoid(
        m_first_row, linear_index - rectangle_size, r, c);
      r += rectangle_size / col;
    }

    c += col_offset;
    tensor[linear_index] = r;
    tensor[linear_index + triu_size] = c;
  }
}

// Some Large test cases for the fallback binary search path is disabled by
// default to speed up CI tests and to avoid OOM error. When modifying the
// implementation, please enable them in test/test_cuda.py and make sure they
// pass on your local server.
Tensor triu_indices_cuda(
    int64_t row, int64_t col, int64_t offset, const TensorOptions& options) {
  check_args(row, col, options);

  auto triu_size = row * col - get_tril_size(row, col, offset - 1);
  auto tensor = empty_cuda({2, triu_size}, options);

  if (triu_size > 0) {
    // # of triu elements in the first row
    auto m_first_row = offset > 0 ?
      std::max<int64_t>(col - offset, 0) : // upper bounded by col
      col;

    // size of the top rectangle
    int64_t rectangle_size = 0;
    if (offset < 0) {
      rectangle_size = std::min<int64_t>(row, -offset) * col;
    }

    dim3 dim_block = cuda::getApplyBlock();
    dim3 dim_grid;

    // using triu_size instead of tensor.numel(), as each thread takes care of
    // two elements in the tensor.
    TORCH_CHECK(
      cuda::getApplyGrid(triu_size, dim_grid, tensor.get_device()),
      "unable to get dim grid");

    AT_DISPATCH_ALL_TYPES_AND(at::ScalarType::Half, tensor.scalar_type(), "triu_indices_cuda", [&] {
      triu_indices_kernel<<<
          dim_grid, dim_block, 0, at::cuda::getCurrentCUDAStream()>>>(
        tensor.data<scalar_t>(),
        std::max<int64_t>(0, offset),
        m_first_row,
        col,
        rectangle_size,
        triu_size);
    });
  }

  return tensor;
}

// ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~ choice ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~

__global__ void generate_samples(
  int64_t *samples,
  int64_t k,
  int64_t n,
  std::pair<uint64_t, uint64_t> seeds
){
  int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
  hiprandStatePhilox4_32_10_t state;
  hiprand_init(seeds.first, thread_id, seeds.second, &state);
  int64_t s = curand4(&state).x % (thread_id + k + 1);
  if (thread_id < n){
    samples[thread_id] = s;
  }
}

__global__ void generate_keys(
  float *keys,
  float *weights,
  int64_t n,
  std::pair<uint64_t, uint64_t> seeds
){
  int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
  hiprandStatePhilox4_32_10_t state;
  hiprand_init(seeds.first, thread_id, seeds.second, &state);
  float u = hiprand_uniform4(&state).x;
  if(thread_id < n){
    keys[thread_id] = weights[thread_id] > 0 ? (float) __powf(u, (float) 1 / weights[thread_id]):-1;
  }
}

__global__ void sampling_with_replacement_kernel(
  int64_t *samples,
  float *cdf,
  int64_t n,
  int64_t k,
  std::pair<uint64_t, uint64_t> seeds
){
  int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
  hiprandStatePhilox4_32_10_t state;
  hiprand_init(seeds.first, thread_id, seeds.second, &state);
  float u = hiprand_uniform4(&state).x;
  if(thread_id < k){
    auto ptr = thrust::lower_bound(thrust::device, cdf, cdf + n, u);
    samples[thread_id] = thrust::distance(cdf, ptr);
  }
}

__global__ void generate_reservoir(
  int64_t *indices,
  int64_t *samples,
  int64_t nb_iterations,
  int64_t k
){
  for(int i = 0; i < nb_iterations; i++){
    int64_t z = samples[i];
    if (z < k) {
      thrust::swap(indices[z], indices[i + k]);
    }
  }
}

Tensor reservoir_sampling_cuda(
  const Tensor& x,
  const Tensor& weights,
  int64_t k
){

  TORCH_CHECK(
    x.dim() > 0,
    "The input Tensor must have at least one dimension"
  );

  int n = x.size(0);

  TORCH_CHECK(
    n >= k,
    "Cannot take a larger sample than population when 'replace=False'"
  );

  auto options = x.options().dtype(at::kLong);
  dim3 threads(threadsPerBlock);

  THCState *state = at::globalContext().getTHCState();
  THCGenerator* gen = THCRandom_getGenerator(state);
  uint64_t offset = gen->state.philox_seed_offset.fetch_add(4);
  std::pair<uint64_t, uint64_t> next_philox_seed = std::make_pair(
                                                    gen->state.initial_seed,
                                                    offset
                                                  );

  if (weights.numel() == 0){ // Uniform Sampling
    Tensor indices_n = at::arange({n}, options);

    // This is a trick to speed up the reservoir sampling.
    // It makes the worst case be k = n / 2.
    int split, begin, end;
    if(2 * k < n){
      split = n - k;
      begin = n - k;
      end = n;
    } else {
      split = k;
      begin = 0;
      end = k;
    }

    int nb_iterations = std::min(k, n - k);
    dim3 blocks((nb_iterations + threadsPerBlock - 1)/threadsPerBlock);

    Tensor samples = at::arange({nb_iterations}, options);

    generate_samples<<<blocks, threads>>>(
      samples.data<int64_t>(),
      split,
      n,
      next_philox_seed
    );

    AT_CUDA_CHECK(hipGetLastError());

    // This must be done in a separeted kernel
    // since this algorithm isn't thread safe
    generate_reservoir<<<1, 1>>>(
      indices_n.data<int64_t>(),
      samples.data<int64_t>(),
      nb_iterations,
      split
    );

    AT_CUDA_CHECK(hipGetLastError());

    return x.index_select(
      0,
      indices_n.index_select(
        0,
        at::arange(begin, end, options)
      )
    );

  } else { // Weighted Sampling

    // If the weights are contiguous floating points, then
    // the next step won't generate a copy.
    Tensor weights_contiguous = weights.contiguous().to(at::kFloat);

    TORCH_CHECK(
      weights_contiguous.device() == x.device(),
      "The weights must share the same device as the inputs."
    );

    TORCH_CHECK(
      n == weights_contiguous.numel(),
      "The weights must have the same number of elements as the input's first dimension."
    );

    TORCH_CHECK(
      weights_contiguous.dim() == 1,
      "The weights must 1-dimensional."
    );

    TORCH_CHECK(
      weights_contiguous.nonzero().numel() >= k,
      "Cannot have less non-zero weights than the number of samples."
    );

    TORCH_CHECK(
      weights_contiguous.min().item().toLong() >= 0,
      "All the weights must be non-negative."
    );

    Tensor keys = at::empty({n}, weights_contiguous.options());
    dim3 all_blocks((n + threadsPerBlock - 1)/threadsPerBlock);

    generate_keys<<<all_blocks, threads>>>(
      keys.data<float>(),
      weights_contiguous.data<float>(),
      n,
      next_philox_seed
    );

    AT_CUDA_CHECK(hipGetLastError());

    return x.index_select(0, std::get<1>(keys.topk(k)));
  }
}

Tensor sampling_with_replacement_cuda(
  const Tensor& x,
  const Tensor& weights,
  int64_t k
){

  TORCH_CHECK(
    x.dim() > 0,
    "The input Tensor must have at least one dimension"
  );

  int n = x.size(0);
  Tensor samples;

  if (weights.numel() == 0){ // Uniform Sampling
    samples = at::randint(0, n, {k}, x.options().dtype(at::kLong));
  } else { // Weighted Sampling

    TORCH_CHECK(
      weights.min().item().toLong() >= 0,
      "All the weights must be non-negative."
    );


    TORCH_CHECK(
	    n == weights.numel(),
	    "The weights must have the same number of elements as the input's first dimension."
	  );

	  TORCH_CHECK(
	    weights.dim() == 1,
	    "The weights must 1-dimensional."
	  );

    THCState *state = at::globalContext().getTHCState();
    THCGenerator* gen = THCRandom_getGenerator(state);
    uint64_t offset = gen->state.philox_seed_offset.fetch_add(4);
    std::pair<uint64_t, uint64_t> next_philox_seed = std::make_pair(
                                                      gen->state.initial_seed,
                                                      offset
                                                    );


    samples = at::empty({k}, x.options().dtype(at::kLong));
    Tensor cdf = weights.cumsum(0).to(at::kFloat);

    TORCH_CHECK(
      cdf[-1].item().toFloat() > 0.0,
      "The sum of all the weights must be strictly greater than zero."
    );

    cdf /= cdf[-1];

    dim3 threads(threadsPerBlock);
    dim3 blocks((k + threadsPerBlock - 1)/threadsPerBlock);

    sampling_with_replacement_kernel<<<blocks, threads>>>(
      samples.data<int64_t>(),
      cdf.data<float>(),
      n,
      k,
      next_philox_seed
    );

    AT_CUDA_CHECK(hipGetLastError());
  }

  return x.index_select(0, samples);
}

Tensor choice_cuda(
  const Tensor& input,
  int64_t k,
  bool replace,
  const Tensor& weights
){
  if (replace){
    return sampling_with_replacement_cuda(input, weights, k);
  } else {
    return reservoir_sampling_cuda(input, weights, k);
  }
}

Tensor choice_cuda(
  const Tensor& input,
  int64_t k,
  bool replace
){
  at::Tensor weights = at::empty({0}, input.options().dtype(at::kFloat));
  return native::choice_cuda(input, k, replace, weights);
}

}} // namespace at::native
