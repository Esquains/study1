#include "hip/hip_runtime.h"
#include "ATen/ATen.h"
#include "ATen/NativeFunctions.h"

#include <THC/THCGeneral.h>
#include <THC/THCThrustAllocator.cuh>
#include <thrust/device_ptr.h>
#include <thrust/sort.h>
#include <thrust/execution_policy.h>
#include <thrust/sequence.h>

#include <algorithm>
#include <sstream>

namespace at {
namespace native {

Tensor& eye_out_cuda(Tensor& result, int64_t n, int64_t m) {
  if (n <= 0) {
    std::ostringstream oss;
    oss << "n must be greater than 0, got: " << n;
    std::runtime_error(oss.str());
  }
  if(m <= 0) {
    m = n;
  }

  result.resize_({n, m});
  result.zero_();

  int64_t sz = std::min<int64_t>(n, m);
  int64_t stride = result.stride(0) + result.stride(1);

  Tensor diag = result.as_strided({sz}, {stride});
  diag.fill_(1);
  return result;
}

Tensor& randperm_out_cuda(Tensor& result, int64_t n, Generator* generator) {
  if (n < 0) {
    std::ostringstream oss;
    oss << "n must be non-negative, got " << n;
    throw std::runtime_error(oss.str());
  }

  result.resize_({n});

  if (n < 30000) {  // For small inputs, we offload it to CPU instead.
    auto result_cpu = result.type().toBackend(kCPU).tensor({n});
    randperm_out(result_cpu, n, generator);
    result = result.type().copy(result_cpu);
  } else {
    // Generate random values for the keys array
    auto keys = result.type().tensor(result.sizes()).random_(generator);

    auto result_data = thrust::device_ptr<int64_t>(result.data<int64_t>());
    auto keys_data = thrust::device_ptr<int64_t>(keys.data<int64_t>());

    auto state = globalContext().getTHCState();
    THCThrustAllocator thrustAlloc(state);
    auto policy = thrust::cuda::par(thrustAlloc).on(THCState_getCurrentStream(state));

    thrust::sequence(policy, result_data, result_data + n);

    // Use the sorted order of keys to rearrange the result array
    thrust::sort_by_key(policy, keys_data, keys_data + n, result_data);
  }

  return result;
}

}} // namespace at::native
