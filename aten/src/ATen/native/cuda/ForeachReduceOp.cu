#include "hip/hip_runtime.h"
#define TORCH_ASSERT_ONLY_METHOD_OPERATORS
#include <ATen/Dispatch.h>
#include <ATen/AccumulateType.h>
#include <ATen/OpMathType.h>
#include <ATen/cuda/DeviceUtils.cuh>
#include <ATen/native/ForeachUtils.h>
#include <ATen/native/cuda/block_reduce.cuh>
#include <ATen/native/cuda/ForeachFunctors.cuh>
#include <ATen/native/cuda/MultiTensorApply.cuh>

#ifndef AT_PER_OPERATOR_HEADERS
#include <ATen/Functions.h>
#include <ATen/NativeFunctions.h>
#else
#include <ATen/ops/_foreach_norm_native.h>
#include <ATen/ops/_foreach_norm_per_tensor_native.h>

#include <ATen/ops/zeros.h>
#include <ATen/ops/empty.h>
#endif

#include <tuple>
#include <vector>


namespace at {
namespace native {

double convert_ord_to_double(const Scalar& ord) {
  double p;
  if (ord.isIntegral(false)) {
    p = ord.to<int64_t>();
  } else if (ord.isFloatingPoint()) {
    p = ord.to<double>();
  } else {
    AT_ERROR("foreach_tensor_norm_cuda expects ord to be integer or float");
  }
  return p;
}

template<typename T, int NormType, int depth=1, int r_args_depth=1, int res_arg_index=0>
struct LpNormFunctor {
  static_assert(NormType == 1 || NormType == 2, "foreach_norm supports only L1 and L2 norm");
  using opmath_t = typename at::opmath_type<T>;
  __device__ __forceinline__ void operator() (
      int chunk_size,
      TensorListMetadata<depth>& tl,
      opmath_t* output,
      const int max_chunks_per_tensor,
      const bool per_tensor
  ) {
    int tensor_loc = tl.block_to_tensor[blockIdx.x];
    int chunk_idx = tl.block_to_chunk[blockIdx.x];
    int n = tl.numel_for_tensor[tensor_loc];

    T* x = (T*)tl.addresses[0][tensor_loc];
    x += chunk_idx * chunk_size;
    n -= chunk_idx * chunk_size;

    __shared__ opmath_t s_vals[512];
    opmath_t vals[kILP];
    T r_x[kILP];
    for (int i = 0; i < kILP; i++) {
      vals[i] = opmath_t(0);
      r_x[i] = T(0);
    }

    if (n % kILP == 0 && (chunk_size & kILP) == 0 && is_aligned(x)) {
      for (int i_start = threadIdx.x; i_start * kILP < n && i_start * kILP < chunk_size; i_start += blockDim.x) {
        // load
        load_store(r_x, x, 0, i_start);
#pragma unroll
        for (int ii = 0; ii < kILP; ii++) {
          opmath_t next = static_cast<opmath_t>(r_x[ii]);
          vals[ii] += NormType == 1 ? ::abs(next)  : next * next;
        }
      }
    } else {
      for (int i_start = 0; i_start < n && i_start < chunk_size; i_start += blockDim.x * kILP) {
#pragma unroll
        for (int ii = 0; ii < kILP; ii++) {
          int i = i_start + threadIdx.x + ii * blockDim.x;
          if (i < n && i < chunk_size) {
            opmath_t next = static_cast<opmath_t>(x[i]);
            vals[ii] += NormType == 1 ? ::abs(next) : next * next;
          }
        }
      }
    }

    auto val = opmath_t(0);
    for (int i = 0; i < kILP; i++) {
      val += vals[i];
    }
    auto final = at::native::cuda_utils::BlockReduceSum(val, s_vals);

    if (threadIdx.x == 0) {
      if (per_tensor) {
        output[(tl.start_tensor_this_launch + tensor_loc) * max_chunks_per_tensor + chunk_idx] = final;
      } else {
        output[blockIdx.x] += final;
      }
    }
  }
};

template<typename T, int NormType, typename opmath_t = at::opmath_type<T>>
__global__ void lpnorm_cleanup(
    opmath_t* output,
    T* ret,
    int max_chunks_per_tensor,
    const bool per_tensor) {
  __shared__ opmath_t vals[512];
  if (!per_tensor) {
    if (blockIdx.x == 0) {
      opmath_t val = 0;
      if (threadIdx.x < 320) {
        val = output[threadIdx.x];
      }
      opmath_t final = at::native::cuda_utils::BlockReduceSum<opmath_t>(val, vals);
      if (threadIdx.x == 0) {
        *ret = NormType == 1 ? static_cast<T>(final) : static_cast<T>(::sqrt(final));
      }
    }
  } else {
    opmath_t* output_this_tensor = output + blockIdx.x*max_chunks_per_tensor;
    opmath_t val = 0;
    for (int i = threadIdx.x; i < max_chunks_per_tensor; i += blockDim.x) {
      val += output_this_tensor[i];
    }
    opmath_t final = at::native::cuda_utils::BlockReduceSum<opmath_t>(val, vals);
    if(threadIdx.x == 0) {
      ret[blockIdx.x] = NormType == 1 ? static_cast<T>(final) : static_cast<T>(::sqrt(final));
    }
  }
}

// note(mkozuki): Why excluding Int and Complex from fast path
// - Int: at::norm does not support.
// - Complex: __shfl_down_sync does not support complex and foreach does not support functions whose inputs dtypes and output dtype are different.
std::vector<Tensor> foreach_tensor_norm_per_tensor_cuda(TensorList tensors, const Scalar& ord) {
  const auto p = convert_ord_to_double(ord);
  check_foreach_api_restrictions(tensors);
  const bool has_int_or_complex = std::any_of(tensors.begin(), tensors.end(), [](const auto & t) {
      const auto scalar_type = t.scalar_type();
      return at::isIntegralType(scalar_type, /*includeBool*/true) || at::isComplexType(scalar_type);
  });
  if (!can_use_fast_route(tensors) ||
      has_int_or_complex ||
      !(p == static_cast<double>(1) || p == static_cast<double>(2))) {
    return foreach_tensor_norm_per_tensor_slow(tensors, ord);
  }

  const int ntensors = tensors.size();
  int max_chunks_per_tensor = -1;

  for (int t = 0; t < ntensors; t++) {
    int max_chunks_this_tensor = (tensors[t].numel() + kChunkSize - 1) / kChunkSize;
    if(max_chunks_this_tensor > max_chunks_per_tensor) {
      max_chunks_per_tensor = max_chunks_this_tensor;
    }
  }
  const auto options = tensors[0].options();
  auto output_per_tensor = at::zeros({ntensors*max_chunks_per_tensor}, options.dtype(toOpMathType(tensors[0].scalar_type())));
  auto ret_per_tensor = at::empty({ntensors}, options);
  auto tensor_lists = std::vector<std::vector<Tensor>>{tensors.vec()};
  constexpr bool per_tensor = true;

  if (p == static_cast<double>(1)) {
    AT_DISPATCH_FLOATING_TYPES_AND2(
      kHalf, kBFloat16, tensor_lists[0][0].scalar_type(), "foreach_tensor_norm_cuda", [&]() {
        using opmath_t = typename at::opmath_type<scalar_t>;
        multi_tensor_apply<1>(
          tensor_lists,
          LpNormFunctor<scalar_t, 1>(),
          output_per_tensor.data_ptr<opmath_t>(),
          max_chunks_per_tensor,
          per_tensor);
        C10_CUDA_KERNEL_LAUNCH_CHECK();
        const at::cuda::OptionalCUDAGuard device_guard(device_of(output_per_tensor));
        auto stream = at::cuda::getCurrentCUDAStream();
        lpnorm_cleanup<scalar_t, 1><<<ntensors, 512, 0, stream>>>(
          output_per_tensor.data_ptr<opmath_t>(),
          ret_per_tensor.data_ptr<scalar_t>(),
          max_chunks_per_tensor,
          per_tensor);
        C10_CUDA_KERNEL_LAUNCH_CHECK();
      });
  } else if (p == static_cast<double>(2)) {
    AT_DISPATCH_FLOATING_TYPES_AND2(
      kHalf, kBFloat16, tensor_lists[0][0].scalar_type(), "foreach_tensor_norm_cuda", [&]() {
        using opmath_t = typename at::opmath_type<scalar_t>;
        multi_tensor_apply<1>(
          tensor_lists,
          LpNormFunctor<scalar_t, 2>(),
          output_per_tensor.data_ptr<opmath_t>(),
          max_chunks_per_tensor,
          per_tensor);
        C10_CUDA_KERNEL_LAUNCH_CHECK();
        const at::cuda::OptionalCUDAGuard device_guard(device_of(output_per_tensor));
        auto stream = at::cuda::getCurrentCUDAStream();
        lpnorm_cleanup<scalar_t, 2><<<ntensors, 512, 0, stream>>>(
          output_per_tensor.data_ptr<opmath_t>(),
          ret_per_tensor.data_ptr<scalar_t>(),
          max_chunks_per_tensor,
          per_tensor);
        C10_CUDA_KERNEL_LAUNCH_CHECK();
      });
  } else {
    AT_ERROR("foreach_tensor_norm_cuda fast path got unexpected ord value: ", p);
  }

  std::vector<Tensor> result;
  result.reserve(ntensors);
  for (const auto& i : c10::irange(ntensors)) {
    result.emplace_back(ret_per_tensor[i]);
  }
  return result;
}

Tensor global_norm_cuda_impl(TensorList tensors, const Scalar& ord) {
  TORCH_CHECK((ord.isIntegral(false) || ord.isFloatingPoint()), "foreach_norm supports int and float ord");
  double p;
  if (ord.isIntegral(false)) {
    p = ord.to<int64_t>();
  }
  if (ord.isFloatingPoint()) {
    p = ord.to<double>();
  }
  check_foreach_api_restrictions(tensors);
  const bool has_int_or_complex = std::any_of(tensors.begin(), tensors.end(), [](const auto & t) {
      const auto scalar_type = t.scalar_type();
      return at::isIntegralType(scalar_type, /*includeBool*/true) || at::isComplexType(scalar_type);
  });
  if (!can_use_fast_route(tensors) ||
      has_int_or_complex ||
      !(p == static_cast<double>(1) || p == static_cast<double>(2))) {
    return foreach_tensor_norm_slow(tensors, ord);
  }

  const int num_tensors = tensors.size();
  int max_chunks_per_tensor = -1;

  for (const int & t : c10::irange(num_tensors)) {
    const int max_chunks_this_tensor = (tensors[0][t].numel() + kChunkSize - 1) / kChunkSize;
    if (max_chunks_this_tensor > max_chunks_per_tensor) {
      max_chunks_per_tensor = max_chunks_this_tensor;
    }
  }
  const auto options = tensors[0].options();
  auto output = at::zeros({320}, options.dtype(toOpMathType(tensors[0].scalar_type())));
  auto ret = at::empty({}, options);
  auto tensor_lists = std::vector<std::vector<Tensor>>{tensors.vec()};
  if (p == static_cast<double>(1)) {
    AT_DISPATCH_FLOATING_TYPES_AND2(
      kHalf, kBFloat16, tensor_lists[0][0].scalar_type(), "global_norm_cuda_impl",
      [&]() {
        using opmath_t = typename at::opmath_type<scalar_t>;
        multi_tensor_apply<1>(
          tensor_lists,
          LpNormFunctor<scalar_t, 1>(),
          output.data_ptr<opmath_t>(),
          max_chunks_per_tensor,
          false);
        C10_CUDA_KERNEL_LAUNCH_CHECK();
        const at::cuda::OptionalCUDAGuard device_guard(device_of(output));
        auto stream = at::cuda::getCurrentCUDAStream();
        lpnorm_cleanup<scalar_t, 1><<<num_tensors, 512, 0, stream>>>(
          output.data_ptr<scalar_t>(),
          ret.data_ptr<scalar_t>(),
          max_chunks_per_tensor,
          false);
        C10_CUDA_KERNEL_LAUNCH_CHECK();
      }
    );
  }
  return ret;
}

Tensor foreach_tensor_norm_cuda(TensorList tensors, const Scalar& ord) {
  const auto p = convert_ord_to_double(ord);
  check_foreach_api_restrictions(tensors);

  const bool has_int_or_complex = std::any_of(tensors.begin(), tensors.end(), [](const auto & t) {
      const auto scalar_type = t.scalar_type();
      return at::isIntegralType(scalar_type, /*includeBool*/true) || at::isComplexType(scalar_type);
  });
  if (!can_use_fast_route(tensors) ||
      has_int_or_complex ||
      !(p == static_cast<double>(1) || p == static_cast<double>(2))) {
    return foreach_tensor_norm_slow(tensors, ord);
  }

  const int ntensors = tensors.size();
  int max_chunks_per_tensor = -1;

  for (const auto & t : tensors) {
    const int max_chunks_this_tensor = (t.numel() + kChunkSize - 1) / kChunkSize;
    if (max_chunks_this_tensor > max_chunks_per_tensor) {
      max_chunks_per_tensor = max_chunks_this_tensor;
    }
  }

  const auto output_scalar_type = tensors[0].scalar_type();
  const auto options = tensors[0].options();
  auto output = at::zeros({320}, options.dtype(toOpMathType(output_scalar_type)));
  auto ret = at::empty({0}, options);
  auto tensor_lists = std::vector<std::vector<Tensor>>{tensors.vec()};
  constexpr bool per_tensor = false;

  if (p == static_cast<double>(1)) {
    AT_DISPATCH_FLOATING_TYPES_AND2(
      kHalf, kBFloat16, tensor_lists[0][0].scalar_type(), "foreach_tensor_norm_cuda", [&]() {
        using opmath_t = typename at::opmath_type<scalar_t>;
        multi_tensor_apply<1>(
          tensor_lists,
          LpNormFunctor<scalar_t, 1>(),
          output.data_ptr<opmath_t>(),
          max_chunks_per_tensor,
          per_tensor);
        C10_CUDA_KERNEL_LAUNCH_CHECK();
        const at::cuda::OptionalCUDAGuard device_guard(device_of(output));
        auto stream = at::cuda::getCurrentCUDAStream();
        lpnorm_cleanup<scalar_t, 1><<<ntensors, 512, 0, stream>>>(
          output.data_ptr<opmath_t>(),
          ret.data_ptr<scalar_t>(),
          max_chunks_per_tensor,
          per_tensor);
        C10_CUDA_KERNEL_LAUNCH_CHECK();
      });
  } else if (p == static_cast<double>(2)) {
    AT_DISPATCH_FLOATING_TYPES_AND2(
      kHalf, kBFloat16, tensor_lists[0][0].scalar_type(), "foreach_tensor_norm_cuda", [&]() {
        using opmath_t = typename at::opmath_type<scalar_t>;
        multi_tensor_apply<1>(
          tensor_lists,
          LpNormFunctor<scalar_t, 2>(),
          output.data_ptr<opmath_t>(),
          max_chunks_per_tensor,
          per_tensor);
        C10_CUDA_KERNEL_LAUNCH_CHECK();
        const at::cuda::OptionalCUDAGuard device_guard(device_of(output));
        auto stream = at::cuda::getCurrentCUDAStream();
        lpnorm_cleanup<scalar_t, 2><<<ntensors, 512, 0, stream>>>(
          output.data_ptr<opmath_t>(),
          ret.data_ptr<scalar_t>(),
          max_chunks_per_tensor,
          per_tensor);
        C10_CUDA_KERNEL_LAUNCH_CHECK();
      });
  } else {
    AT_ERROR("foreach_tensor_norm_cuda fast path got unexpected ord value: ", p);
  }
  return ret;
}

} // namespace native
} // namespace at
