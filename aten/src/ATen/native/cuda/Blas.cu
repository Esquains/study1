void THCudaBlas_Sgemv(THCState *state, char trans, int64_t m, int64_t n, float alpha, float *a, int64_t lda, float *x, int64_t incx, float beta, float *y, int64_t incy)
{
  at::cuda::blas::gemv<float>(at::cuda::getCurrentCUDAStream().stream(), trans, m, n, alpha, a, lda, x, incx, beta, y, incy);
}

void THCudaBlas_Dgemv(THCState *state, char trans, int64_t m, int64_t n, double alpha, double *a, int64_t lda, double *x, int64_t incx, double beta, double *y, int64_t incy)
{
  at::cuda::blas::gemv<double>(at::cuda::getCurrentCUDAStream().stream(), trans, m, n, alpha, a, lda, x, incx, beta, y, incy);
}

static void THCTensor_(addmvImpl)(THCState *state, THCTensor *r_, THCTensor *t, THCTensor *mat, THCTensor *vec, scalar_t beta, scalar_t alpha)
{
#if defined(THC_REAL_IS_FLOAT) || defined(THC_REAL_IS_DOUBLE)
  if(r_ != t)
  {
    THCTensor_(resizeAs)(state, r_, t);
    THCTensor_(copy)(state, r_, t);
  }

  auto r_stride = THTensor_strideLegacyNoScalars(r_, 0);

  if(mat->stride(0) == 1)
  {
#ifdef THC_REAL_IS_FLOAT
    THCudaBlas_Sgemv(state, 'n', mat->size(0), mat->size(1),
                    alpha, THCTensor_(data)(state, mat), mat->stride(1),
                    THCTensor_(data)(state, vec), vec_stride,
                    beta, THCTensor_(data)(state, r_), r_stride);
#elif defined(THC_REAL_IS_DOUBLE)
    THCudaBlas_Dgemv(state, 'n', mat->size(0), mat->size(1),
                    alpha, THCTensor_(data)(state, mat), mat->stride(1),
                    THCTensor_(data)(state, vec), vec_stride,
                    beta, THCTensor_(data)(state, r_), r_stride);
#endif
  }
  else if(mat->stride(1) == 1)
  {
#ifdef THC_REAL_IS_FLOAT
    THCudaBlas_Sgemv(state, 't',  mat->size(1), mat->size(0),
                    alpha, THCTensor_(data)(state, mat), mat->stride(0),
                    THCTensor_(data)(state, vec), vec_stride,
                    beta, THCTensor_(data)(state, r_), r_stride);
#elif defined(THC_REAL_IS_DOUBLE)
    THCudaBlas_Dgemv(state, 't',  mat->size(1), mat->size(0),
                     alpha, THCTensor_(data)(state, mat), mat->stride(0),
                     THCTensor_(data)(state, vec), vec_stride,
                     beta, THCTensor_(data)(state, r_), r_stride);
#endif
  }
  else
  {
    THCTensor *cmat = THCTensor_(newContiguous)(state, mat);

#ifdef THC_REAL_IS_FLOAT
    THCudaBlas_Sgemv(state, 't',  mat->size(1), mat->size(0),
                    alpha, THCTensor_(data)(state, cmat), cmat->stride(0),
                    THCTensor_(data)(state, vec), vec_stride,
                    beta, THCTensor_(data)(state, r_), r_stride);
#elif defined(THC_REAL_IS_DOUBLE)
    THCudaBlas_Dgemv(state, 't',  mat->size(1), mat->size(0),
                    alpha, THCTensor_(data)(state, cmat), cmat->stride(0),
                    THCTensor_(data)(state, vec), vec_stride,
                    beta, THCTensor_(data)(state, r_), r_stride);
#endif

    THCTensor_(free)(state, cmat);
  }

  // In hipblasSgemv, hipblasDgemv (x,0).mv(0) does not
  // handle beta, whereas hipblasSgemm, hipblasDgemm do for case where (x,0).mm(0,y).
  if (THTensor_sizeLegacyNoScalars(vec, 0) == 0 && mat->size(0) != 0) {
    if(THCNumerics<scalar_t>::eq(beta, ScalarConvert<int, scalar_t>::to(0))) {
      THCTensor_(zero)(state, r_);
    } else if(THCNumerics<scalar_t>::ne(beta, ScalarConvert<int, scalar_t>::to(1))) {
      THCTensor_(mul)(state, r_, r_, beta);
    }
  }

#elif defined(THC_REAL_IS_HALF) || defined(THC_REAL_IS_BFLOAT16)
    // Currently no Hgemv/SgemvEx in Cublas
    THCTensor *vecAsMatrix = THCTensor_(newWithTensor)(state, vec);
    THCTensor_(resize2d)(state, vecAsMatrix, vec_size, 1);

    THCTensor *tAsMatrix = THCTensor_(newWithTensor)(state, t);
    THCTensor_(resize2d)(state, tAsMatrix, THTensor_sizeLegacyNoScalars(tAsMatrix, 0), 1);

    THCTensor_(addmm)(state, r_, tAsMatrix, mat, vecAsMatrix, beta, alpha);

    // r_ will have answer as matrix, need to return a vector
    THCTensor_(resize1d)(state, r_, THTensor_sizeLegacyNoScalars(r_, 0));
    THCTensor_(free)(state, vecAsMatrix);
    THCTensor_(free)(state, tAsMatrix);
#endif
#else
  ERROR_ONLY_FP_TYPES("addmv");
#endif
}
