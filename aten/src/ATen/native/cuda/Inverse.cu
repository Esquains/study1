#include "ATen/Context.h"
#include "ATen/cuda/HIPContext.h"
#include "ATen/Dispatch.h"
#include "ATen/NativeFunctions.h"
#include "ATen/cuda/PinnedMemoryAllocator.h"
#include "ATen/cuda/CUDAApplyUtils.cuh"

#include "ATen/native/LinearAlgebraUtils.h"
#include "ATen/native/cuda/MiscUtils.h"
#include "ATen/native/Inverse.h"

#include "THC.h" // for USE_MAGMA

#ifdef USE_MAGMA
#include <magma.h>
#include <magma_types.h>
#endif

namespace at {
namespace native {

#ifdef USE_MAGMA
template<class scalar_t>
void magmaGetriBatched(
    magma_int_t n, scalar_t** dA_array, magma_int_t ldda,
    magma_int_t** ipiv_array, scalar_t** dinvA_array, magma_int_t lddia,
    magma_int_t* info_array, magma_int_t batchsize, magma_queue_t queue) {
  AT_ERROR("getri only takes float or double Tensors");
}

template<class scalar_t>
void magmaGetrfBatched(
    magma_int_t m, magma_int_t n, scalar_t** dA_array, magma_int_t ldda,
    magma_int_t** ipiv_array, magma_int_t* info_array, magma_int_t batchsize,
    magma_queue_t queue) {
  AT_ERROR("getrf only takes float or double Tensors");
}

template<>
void magmaGetriBatched<double>(
    magma_int_t n, double** dA_array, magma_int_t ldda,
    magma_int_t** ipiv_array, double** dinvA_array, magma_int_t lddia,
    magma_int_t* info_array, magma_int_t batchsize, magma_queue_t queue) {
    magma_dgetri_outofplace_batched(
        n, dA_array, ldda, ipiv_array, dinvA_array,
        lddia, info_array, batchsize, queue);
}

template<>
void magmaGetriBatched<float>(
    magma_int_t n, float** dA_array, magma_int_t ldda,
    magma_int_t** ipiv_array, float** dinvA_array, magma_int_t lddia,
    magma_int_t* info_array, magma_int_t batchsize, magma_queue_t queue) {
    magma_sgetri_outofplace_batched(
        n, dA_array, ldda, ipiv_array, dinvA_array,
        lddia, info_array, batchsize, queue);
}

template<>
void magmaGetrfBatched<double>(
    magma_int_t m, magma_int_t n, double** dA_array, magma_int_t ldda,
    magma_int_t** ipiv_array, magma_int_t* info_array, magma_int_t batchsize,
    magma_queue_t queue) {
    magma_dgetrf_batched(
    m, n, dA_array, ldda, ipiv_array,
    info_array, batchsize, queue);
}

template<>
void magmaGetrfBatched<float>(
    magma_int_t m, magma_int_t n, float** dA_array, magma_int_t ldda,
    magma_int_t** ipiv_array, magma_int_t* info_array, magma_int_t batchsize,
    magma_queue_t queue) {
    magma_sgetrf_batched(
    m, n, dA_array, ldda, ipiv_array,
    info_array, batchsize, queue);
}
#endif

#define ALLOCATE_ARRAY(name, type, size, dummy_tensor) \
  auto storage_##name = pin_memory<type>(size, dummy_tensor); \
  name = static_cast<type*>(storage_##name.data());

template <typename scalar_t>
static void applyInverse(Tensor &self, Tensor &self_inv, std::vector<int64_t>& infos) {
#ifndef USE_MAGMA
AT_ERROR("inverse: MAGMA library not found in "
    "compilation. Please rebuild with MAGMA.");
#else
  auto self_data = self.data<scalar_t>();
  auto self_mat_stride = matrixStride(self);
  auto self_inv_data = self_inv.data<scalar_t>();
  auto self_inv_mat_stride = matrixStride(self_inv);

  magma_int_t batch_size = magma_int_cast(batchCount(self), "batchCount");
  magma_int_t n = magma_int_cast(self.size(-2), "self.size(-2)");

  magma_int_t* info_array;
  magma_int_t* ipiv_data;
  magma_int_t** ipiv_array;
  scalar_t** self_array;
  scalar_t** self_inv_array;

  ALLOCATE_ARRAY(info_array, magma_int_t, batch_size, self);
  ALLOCATE_ARRAY(ipiv_data, magma_int_t, batch_size * n, self);
  ALLOCATE_ARRAY(ipiv_array, magma_int_t*, batch_size, self);
  ALLOCATE_ARRAY(self_array, scalar_t*, batch_size, self);
  ALLOCATE_ARRAY(self_inv_array, scalar_t*, batch_size, self_inv);

  // Set up the created arrays
  for (int64_t i = 0; i < batch_size; i++) {
    self_array[i] = &self_data[i * self_mat_stride];
    self_inv_array[i] = &self_inv_data[i * self_inv_mat_stride];
    ipiv_array[i] = &ipiv_data[i * n];
  }

  // Create queue for both getrf and getri per batch
  magma_queue_t inverse_magma_queue = createMagmaQueue(self);

  magmaGetrfBatched<scalar_t>(
    n, n, self_array, n, ipiv_array, info_array,
    batch_size, inverse_magma_queue);

  for (int64_t i = 0; i < batch_size; i++) {
    infos[i] = info_array[i];
  }

  // This is to pre-emptively stop computation if getrf fails
  checkErrors(infos, "inverse");

  magmaGetriBatched<scalar_t>(
    n, self_array, n, ipiv_array, self_inv_array,
    n, info_array, batch_size, inverse_magma_queue);

  for (int64_t i = 0; i < batch_size; i++) {
    infos[i] = info_array[i];
  }
#endif
}

Tensor _inverse_helper_cuda(const Tensor& self) {
  std::vector<int64_t> infos(batchCount(self), 0);
  auto self_working_copy = cloneBatchedColumnMajor(self);
  auto self_inv_working_copy = cloneBatchedColumnMajor(self);
  AT_DISPATCH_FLOATING_TYPES(self.type(), "inverse", [&]{
    applyInverse<scalar_t>(
      self_working_copy, self_inv_working_copy, infos);
  });
  checkErrors(infos, "inverse");
  return self_inv_working_copy;
}

} // namespace native
} // namespace at

#undef ALLOCATE_ARRAY
