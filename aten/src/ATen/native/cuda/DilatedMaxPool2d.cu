#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/AccumulateType.h>
#include <ATen/native/Pool.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/CUDAApplyUtils.cuh>
#include <ATen/cuda/detail/TensorInfo.cuh>
#include <ATen/cuda/detail/IndexUtils.cuh>
#include <ATen/cuda/detail/KernelUtils.h>
#include <THC/THCNumerics.cuh>
#include <c10/macros/Macros.h>
#include <ATen/native/cuda/LaunchUtils.h>

namespace at {
namespace native {
namespace {

__device__ inline int min(int a, int b) {
  return a <= b ? a : b;
}

// kernels borrowed from Caffe
template <typename scalar_t, typename accscalar_t>
__global__ void MaxPoolForwardNCHW(const int nthreads, const scalar_t* bottom_data,
    const int num, const int channels, const int height,
    const int width, const int pooled_height, const int pooled_width,
    const int kernel_h, const int kernel_w, const int stride_h,
    const int stride_w, const int pad_h, const int pad_w,
    const int dilation_h, const int dilation_w, scalar_t* top_data,
    int64_t* top_mask) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    int pw = index % pooled_width;
    int ph = (index / pooled_width) % pooled_height;
    int c = (index / pooled_width / pooled_height) % channels;
    int n = index / pooled_width / pooled_height / channels;
    int hstart = ph * stride_h - pad_h;
    int wstart = pw * stride_w - pad_w;
    int hend = min(hstart + (kernel_h - 1) * dilation_h + 1, height);
    int wend = min(wstart + (kernel_w - 1) * dilation_w + 1, width);
    while(hstart < 0)
      hstart += dilation_h;
    while(wstart < 0)
      wstart += dilation_w;
    accscalar_t maxval = at::numeric_limits<accscalar_t>::lower_bound(); // -Infinity
    int maxidx = hstart * width + wstart;
    bottom_data += (n * channels + c) * height * width;
    for (int h = hstart; h < hend; h += dilation_h) {
      for (int w = wstart; w < wend; w += dilation_w) {
        scalar_t val = bottom_data[h * width + w];
        if ((ScalarConvert<scalar_t, accscalar_t>::to(val) > maxval) || THCNumerics<scalar_t>::isnan(val)) {
          maxidx = h * width + w;
          maxval = ScalarConvert<scalar_t, accscalar_t>::to(val);
        }
      }
    }
    top_data[index] = ScalarConvert<scalar_t, accscalar_t>::to(maxval);
    top_mask[index] = maxidx;
  }
}

template <typename scalar_t, typename accscalar_t>
__global__ void MaxPoolForwardNHWC(const int nthreads, const scalar_t* bottom_data,
                                const int num, const int channels, const int height,
                                const int width, const int pooled_height, const int pooled_width,
                                const int kernel_h, const int kernel_w, const int stride_h,
                                const int stride_w, const int pad_h, const int pad_w,
                                const int dilation_h, const int dilation_w,
                                const int in_stride_c, const int in_stride_h, const int in_stride_w,
                                scalar_t* top_data, int64_t* top_mask) {

  extern __shared__ int smem[];
  scalar_t *out_cached = reinterpret_cast<scalar_t*>(smem);
  int cache_size = channels * blockDim.x;
  for (int i = threadIdx.x; i < cache_size; i+= blockDim.x) {
    out_cached[2 * i] = scalar_t(0.0);
    out_cached[2 * i + 1] = scalar_t(0.0);
  }
  __syncthreads();
  out_cached = &out_cached[2 * threadIdx.x * channels];
  CUDA_KERNEL_LOOP(index, nthreads) {
    int pw = index % pooled_width;
    int ph = (index / pooled_width) % pooled_height;
    int n = index / pooled_width / pooled_height;
    int hstart = ph * stride_h - pad_h;
    int wstart = pw * stride_w - pad_w;
    int hend = min(hstart + (kernel_h - 1) * dilation_h + 1, height);
    int wend = min(wstart + (kernel_w - 1) * dilation_w + 1, width);
    while(hstart < 0)
      hstart += dilation_h;
    while(wstart < 0)
      wstart += dilation_w;
    accscalar_t maxval = at::numeric_limits<accscalar_t>::lower_bound(); // -Infinity
    bottom_data += (n * channels * height * width);
    for (int h = hstart; h < hend; h += dilation_h) {
      for (int w = wstart; w < wend; w += dilation_w) {
        for (int c = 0; c < channels; c++)  {
          int idx_in = h * in_stride_h + w * in_stride_w + c;
          scalar_t val = bottom_data[idx_in];
          scalar_t maxval = out_cached[2 * c];
          if ((ScalarConvert<scalar_t, accscalar_t>::to(val) > maxval) || THCNumerics<scalar_t>::isnan(val)) {
            out_cached[2 * c] = ScalarConvert<scalar_t, accscalar_t>::to(val);
            out_cached[2 * c + 1] = idx_in;
          }
        }
      }
    }
    for (int c = 0; c < channels; c++) {
      top_data[index * channels + c] = out_cached[2 * c];
      top_mask[index * channels + c] = (out_cached[2 * c + 1] - c) / channels;
      out_cached[2 * c] = scalar_t(0.0);
      out_cached[2 * c + 1] = scalar_t(0.0);
    }
  }
}


  static const int BACKWARD_THREADS = 256;

template <typename scalar_t, typename accscalar_t>
#if defined (__HIP_PLATFORM_HCC__)
C10_LAUNCH_BOUNDS_2(BACKWARD_THREADS, 4)
#else
C10_LAUNCH_BOUNDS_2(BACKWARD_THREADS, 8)
#endif
__global__ void MaxPoolBackwardNCHW(const int nthreads, const scalar_t* top_diff,
    const int64_t* top_mask, const int num, const int channels,
    const int height, const int width, const int pooled_height,
    const int pooled_width, const int kernel_h, const int kernel_w,
    const int stride_h, const int stride_w, const int pad_h, const int pad_w,
    const int dilation_h, const int dilation_w,
    scalar_t* bottom_diff) {
    CUDA_KERNEL_LOOP(index, height*width) {
    int h = index/width;
    int w = index - h * width;
//get some templating performance benefits without actually templating
    int phstart, phend, pwstart, pwend;
    if (stride_h == 1) {
       phstart =
        (h + pad_h < ((kernel_h - 1) * dilation_h + 1)) ? 0 : (h + pad_h - ((kernel_h - 1) * dilation_h + 1))  + 1;
       phend = min((h + pad_h)  + 1, pooled_height);
    } else if (stride_h == 2) {
       phstart =
        (h + pad_h < ((kernel_h - 1) * dilation_h + 1)) ? 0 : (h + pad_h - ((kernel_h - 1) * dilation_h + 1)) / 2  + 1;
       phend = min((h + pad_h) / 2  + 1, pooled_height);
    } else {
       phstart =
        (h + pad_h < ((kernel_h - 1) * dilation_h + 1)) ? 0 : (h + pad_h - ((kernel_h - 1) * dilation_h + 1)) / stride_h  + 1;
       phend = min((h + pad_h) / stride_h  + 1, pooled_height);
    }
    if (stride_w == 1) {
        pwstart =
        (w + pad_w < ((kernel_w - 1) * dilation_w + 1)) ? 0 : (w + pad_w - ((kernel_w - 1) * dilation_w + 1)) + 1;
        pwend = min((w + pad_w) + 1, pooled_width);
    } else if (stride_w == 2) {
        pwstart =
        (w + pad_w < ((kernel_w - 1) * dilation_w + 1)) ? 0 : (w + pad_w - ((kernel_w - 1) * dilation_w + 1)) / 2 + 1;
        pwend = min((w + pad_w) / 2 + 1, pooled_width);
    } else {
        pwstart =
        (w + pad_w < ((kernel_w - 1) * dilation_w + 1)) ? 0 : (w + pad_w - ((kernel_w - 1) * dilation_w + 1)) / stride_w + 1;
        pwend = min((w + pad_w) / stride_w + 1, pooled_width);
    }
    for (int n = blockIdx.y; n < num; n += gridDim.y)
       for (int c = blockIdx.z; c < channels; c+= gridDim.z) {

        accscalar_t gradient = accscalar_t(0);
        int offset = (n * channels + c) * pooled_height * pooled_width;
        top_diff += offset;
        top_mask += offset;
//get some templating performance benefits without actually templating
        if ((phstart + 1 != phend) || (pwstart + 1 != pwend)) {
        for (int ph = phstart; ph < phend; ++ph) {
          for (int pw = pwstart; pw < pwend; ++pw) {
            int idx = ph * pooled_width + pw;
            if (top_mask[idx] == h * width + w) {
              gradient += ScalarConvert<scalar_t, accscalar_t>::to(top_diff[idx]);
            }
          }
        }
        } else {
            int idx = phstart * pooled_width + pwstart;
            if (top_mask[idx] == h * width + w) {
              gradient += ScalarConvert<scalar_t, accscalar_t>::to(top_diff[idx]);
            }
        }
        bottom_diff[(n*channels+c)*height*width+index] = ScalarConvert<accscalar_t, scalar_t>::to(gradient);
      }
  }
}

template <typename scalar_t, typename accscalar_t>
#if defined (__HIP_PLATFORM_HCC__)
C10_LAUNCH_BOUNDS_2(BACKWARD_THREADS, 4)
#else
C10_LAUNCH_BOUNDS_2(BACKWARD_THREADS, 8)
#endif
__global__ void MaxPoolBackwardNHWC(const int nthreads, const scalar_t* top_diff,
                                    const int64_t* top_mask, const int num, const int channels,
                                    const int height, const int width, const int pooled_height,
                                    const int pooled_width, const int kernel_h, const int kernel_w,
                                    const int stride_h, const int stride_w, const int pad_h, const int pad_w,
                                    const int dilation_h, const int dilation_w,
                                    const int out_stride_c, const int out_stride_h, const int out_stride_w,
                                    const int in_stride_c, const int in_stride_h, const int in_stride_w,
                                    scalar_t* bottom_diff) {
  extern __shared__ int smem[];
  scalar_t *out_cached = reinterpret_cast<scalar_t*>(smem);
  int cache_size = channels * blockDim.x;
  for (int i = threadIdx.x; i < cache_size; i+= blockDim.x) {
    out_cached[i] = scalar_t(0.0);
  }
  __syncthreads();
  out_cached = &out_cached[(threadIdx.y * blockDim.x + threadIdx.x) * channels];
  CUDA_KERNEL_LOOP(index, height*width) {
    int h = index/width;
    int w = index - h * width;
    int n = blockIdx.y * gridDim.y + threadIdx.y;
    int idx = n * channels * height * width + h * in_stride_h + w * in_stride_w;
    bottom_diff += idx;
//get some templating performance benefits without actually templating
    int phstart, phend, pwstart, pwend;
    if (stride_h == 1) {
      phstart =
          (h + pad_h < ((kernel_h - 1) * dilation_h + 1)) ? 0 : (h + pad_h - ((kernel_h - 1) * dilation_h + 1))  + 1;
      phend = min((h + pad_h)  + 1, pooled_height);
    } else if (stride_h == 2) {
      phstart =
          (h + pad_h < ((kernel_h - 1) * dilation_h + 1)) ? 0 : (h + pad_h - ((kernel_h - 1) * dilation_h + 1)) / 2  + 1;
      phend = min((h + pad_h) / 2  + 1, pooled_height);
    } else {
      phstart =
          (h + pad_h < ((kernel_h - 1) * dilation_h + 1)) ? 0 : (h + pad_h - ((kernel_h - 1) * dilation_h + 1)) / stride_h  + 1;
      phend = min((h + pad_h) / stride_h  + 1, pooled_height);
    }
    if (stride_w == 1) {
      pwstart =
          (w + pad_w < ((kernel_w - 1) * dilation_w + 1)) ? 0 : (w + pad_w - ((kernel_w - 1) * dilation_w + 1)) + 1;
      pwend = min((w + pad_w) + 1, pooled_width);
    } else if (stride_w == 2) {
      pwstart =
          (w + pad_w < ((kernel_w - 1) * dilation_w + 1)) ? 0 : (w + pad_w - ((kernel_w - 1) * dilation_w + 1)) / 2 + 1;
      pwend = min((w + pad_w) / 2 + 1, pooled_width);
    } else {
      pwstart =
          (w + pad_w < ((kernel_w - 1) * dilation_w + 1)) ? 0 : (w + pad_w - ((kernel_w - 1) * dilation_w + 1)) / stride_w + 1;
      pwend = min((w + pad_w) / stride_w + 1, pooled_width);
    }
    int offset = (n * channels * pooled_height * pooled_width);
    top_diff += offset;
    top_mask += offset;

    if ((phstart + 1 != phend) || (pwstart + 1 != pwend)) {
      for (int ph = phstart; ph < phend; ++ph) {
        for (int pw = pwstart; pw < pwend; ++pw) {
          for (int c = 0; c < channels; c++) {
            int idx = ph * out_stride_h + pw * out_stride_w + c;
            if (top_mask[idx] == index) {
              out_cached[c] += ScalarConvert<accscalar_t, scalar_t>::to(top_diff[idx]);
            }
          }
        }
      }
    } else {
      for (int c = 0; c < channels; c++) {
        int idx = phstart * out_stride_h + pwstart * out_stride_w + c;
        if (top_mask[idx] == index) {
          out_cached[c] += ScalarConvert<accscalar_t, scalar_t>::to(top_diff[idx]);
        }
      }
    }

    for (int c = 0; c < channels; c++) {
      bottom_diff[c] = ScalarConvert<accscalar_t, scalar_t>::to(out_cached[c]);
      out_cached[c] = scalar_t(0.0);
    }
  }
}

void max_pool2d_with_indices_out_cuda_template(
           Tensor& output,
           Tensor& indices,
           const Tensor& input_,
           IntArrayRef kernel_size,
           IntArrayRef stride,
           IntArrayRef padding,
           IntArrayRef dilation,
           bool ceil_mode)
{
  TensorArg output_arg{ output, "output", 1 };
  TensorArg indices_arg{ indices, "indices", 2 };
  TensorArg input_arg{ input_, "input_", 3 };

  checkAllSameGPU("max_pool2d_with_indices_out_cuda",
                  {output_arg, indices_arg, input_arg});

  // #20866, #22032: Guarantee this for the official C++ API?
  TORCH_CHECK((kernel_size.size() == 1 || kernel_size.size() == 2) &&
              (stride.empty() || stride.size() == 2) &&
              (padding.size() == 1 || padding.size() == 2) &&
              (dilation.size() == 1 || dilation.size() == 2),
    "max_pool2d_with_indices: internal error: all IntArrayRef sizes must be 2");

  TORCH_CHECK((input_.ndimension() == 3 || input_.ndimension() == 4),
    "non-empty 3D or 4D (batch mode) tensor expected for input");

  const int kH = safe_downcast<int, int64_t>(kernel_size[0]);
  const int kW = kernel_size.size() == 1 ? kH : safe_downcast<int, int64_t>(kernel_size[1]);

  const int dH = stride.empty() ? kH : safe_downcast<int, int64_t>(stride[0]);
  const int dW = stride.empty() ? kW : safe_downcast<int, int64_t>(stride[1]);

  const int padH = safe_downcast<int, int64_t>(padding[0]);
  const int padW = padding.size() == 1 ? padH : safe_downcast<int, int64_t>(padding[1]);

  const int dilationH = safe_downcast<int, int64_t>(dilation[0]);
  const int dilationW = dilation.size() == 1 ? dilationH : safe_downcast<int, int64_t>(dilation[1]);

  const auto memory_format = input_.suggest_memory_format();

  const int64_t nbatch = input_.ndimension() == 4 ? input_.size(-4) : 1;
  const int64_t nInputPlane = input_.size(-3);
  const int64_t inputHeight = input_.size(-2);
  const int64_t inputWidth = input_.size(-1);

  const int64_t outputWidth = pooling_output_shape<int64_t>(inputWidth, kW, padW, dW, dilationW, ceil_mode);
  const int64_t outputHeight = pooling_output_shape<int64_t>(inputHeight, kH, padH, dH, dilationH, ceil_mode);

  pool2d_shape_check(
    input_,
    kH, kW, dH, dW, padH, padW, dilationH, dilationW,
    nInputPlane,
    inputHeight, inputWidth,
    outputHeight, outputWidth);

  Tensor input = input_.contiguous(memory_format);

  const int64_t in_stride_c = input.stride(-3);
  const int64_t in_stride_h = input.stride(-2);
  const int64_t in_stride_w = input.stride(-1);

  output.resize_({nbatch, nInputPlane, outputHeight, outputWidth});
  indices.resize_({nbatch, nInputPlane, outputHeight, outputWidth});

  output.unsafeGetTensorImpl()->empty_tensor_restride(memory_format);
  indices.unsafeGetTensorImpl()->empty_tensor_restride(memory_format);

  const int num_threads = std::min(at::cuda::getCurrentDeviceProperties()->maxThreadsPerBlock,
                                   BACKWARD_THREADS);

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(input.scalar_type(),
    "max_pool2d_with_indices_out_cuda_frame",
    [&] {
      using accscalar_t = acc_type<scalar_t, true>;

      scalar_t *output_data = output.data<scalar_t>();
      scalar_t *input_data = input.data<scalar_t>();
      int64_t *indices_data = indices.data<int64_t>();

      if (memory_format == MemoryFormat::ChannelsLast) {
        const int count = safe_downcast<int, int64_t>(nbatch * outputHeight * outputWidth);
        int* maxThreadsDim = at::cuda::getCurrentDeviceProperties()->maxThreadsDim;
        int block_x = std::min<int>(maxThreadsDim[0], std::min<int>(lastPow2(count), at::cuda::warp_size()));
        const dim3 block(block_x);
        int grid_x = cuda::ATenCeilDiv(count, block_x);
        const dim3 grid(grid_x);

        MaxPoolForwardNHWC<scalar_t, scalar_t>
        <<<grid, block, 2 * nInputPlane * block_x * sizeof(scalar_t), at::cuda::getCurrentCUDAStream()>>>(
            count, input_data,
                nbatch, nInputPlane, inputHeight, inputWidth, outputHeight, outputWidth,
                kH, kW, dH, dW, padH, padW, dilationH, dilationW,
                in_stride_c, in_stride_h, in_stride_w,
                output_data, indices_data);
      } else {
        const int count = safe_downcast<int, int64_t>(output.numel());
        MaxPoolForwardNCHW<scalar_t, scalar_t>
            <<<cuda::ATenCeilDiv(count, num_threads), num_threads, 0, at::cuda::getCurrentCUDAStream()>>>(
            count, input_data,
                nbatch, nInputPlane, inputHeight, inputWidth, outputHeight, outputWidth,
                kH, kW, dH, dW, padH, padW, dilationH, dilationW,
                output_data, indices_data);
      }
    }
  );

  TORCH_CHECK(hipGetLastError() == hipSuccess,
     "max_pool2d_with_indices_out_cuda_frame failed with error code ",
     hipGetLastError());

  if(input.ndimension() == 3) {
    output.resize_({nInputPlane, outputHeight, outputWidth});
  }
}

void max_pool2d_with_indices_backward_out_cuda_template(
           Tensor& gradInput,
           const Tensor& gradOutput_,
           const Tensor& input_,
           const Tensor& indices,
           IntArrayRef kernel_size,
           IntArrayRef stride,
           IntArrayRef padding,
           IntArrayRef dilation,
           bool ceil_mode)
{
  TensorArg gradInput_arg{ gradInput, "gradInput", 1 };
  TensorArg gradOutput_arg{ gradOutput_, "gradOutput_", 2 };
  TensorArg input_arg{ input_, "input_", 3 };
  TensorArg indices_arg{ indices, "indices", 4 };

  checkAllSameGPU("max_pool2d_with_indices_out_cuda",
                  {gradInput_arg, gradOutput_arg, input_arg, indices_arg});

  // #20866, #22032: Guarantee this for the official C++ API?
  TORCH_CHECK((kernel_size.size() == 1 || kernel_size.size() == 2) &&
              (stride.empty() || stride.size() == 2) &&
              (padding.size() == 1 || padding.size() == 2) &&
              (dilation.size() == 1 || dilation.size() == 2),
    "max_pool2d_with_indices: internal error: all IntArrayRef sizes must be 2");

  TORCH_CHECK((input_.ndimension() == 3 || input_.ndimension() == 4),
    "non-empty 3D or 4D (batch mode) tensor expected for input");

  const int kH = safe_downcast<int, int64_t>(kernel_size[0]);
  const int kW = kernel_size.size() == 1 ? kH : safe_downcast<int, int64_t>(kernel_size[1]);

  const int dH = stride.empty() ? kH : safe_downcast<int, int64_t>(stride[0]);
  const int dW = stride.empty() ? kW : safe_downcast<int, int64_t>(stride[1]);

  const int padH = safe_downcast<int, int64_t>(padding[0]);
  const int padW = padding.size() == 1 ? padH : safe_downcast<int, int64_t>(padding[1]);

  const int dilationH = safe_downcast<int, int64_t>(dilation[0]);
  const int dilationW = dilation.size() == 1 ? dilationH : safe_downcast<int, int64_t>(dilation[1]);

  const auto memory_format = input_.suggest_memory_format();
  const Tensor input = input_.contiguous(memory_format);

  const int64_t nbatch = input.ndimension() == 4 ? input.size(-4) : 1;
  const int64_t nInputPlane = input.size(-3);
  const int64_t inputHeight = input.size(-2);
  const int64_t inputWidth = input.size(-1);

  const int64_t in_stride_c = input.stride(-3);
  const int64_t in_stride_h = input.stride(-2);
  const int64_t in_stride_w = input.stride(-1);

  const int64_t outputHeight = pooling_output_shape<int64_t>(inputHeight, kH, padH, dH, dilationH, ceil_mode);
  const int64_t outputWidth = pooling_output_shape<int64_t>(inputWidth, kW, padW, dW, dilationW, ceil_mode);

  max_pool2d_backward_shape_check(
    input_,
    gradOutput_,
    indices,
    nbatch,
    kH, kW, dH, dW, padH, padW, dilationH, dilationW,
    nInputPlane,
    inputHeight, inputWidth,
    outputHeight, outputWidth,
    /*cuda=*/ true);

  const Tensor gradOutput = gradOutput_.contiguous(memory_format);

  const int64_t out_stride_c = gradOutput.stride(-3);
  const int64_t out_stride_h = gradOutput.stride(-2);
  const int64_t out_stride_w = gradOutput.stride(-1);

  gradInput.resize_as_(input);
  gradInput.unsafeGetTensorImpl()->empty_tensor_restride(memory_format);

  int64_t count = input.numel();

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(input.scalar_type(),
    "max_pool2d_with_indices_out_cuda_frame",
    [&] {
      using accscalar_t = acc_type<scalar_t, true>;

      scalar_t *gradOutput_data = gradOutput.data<scalar_t>();
      scalar_t *gradInput_data = gradInput.data<scalar_t>();
      int64_t *indices_data = indices.data<int64_t>();

      int imgcount = inputWidth * inputHeight;

      if (memory_format == MemoryFormat::ChannelsLast) {
        int* maxThreadsDim = at::cuda::getCurrentDeviceProperties()->maxThreadsDim;
        int block_x = std::min<int>(maxThreadsDim[0], std::min<int>(lastPow2(imgcount), at::cuda::warp_size()));
        int block_y = std::min<int>(maxThreadsDim[1], std::min<int>(lastPow2(nbatch), BACKWARD_THREADS / block_x));
        const dim3 block(block_x, block_y);
        int grid_x = cuda::ATenCeilDiv(imgcount, block_x);
        int grid_y = (nbatch + block_y - 1) / block_y;
        const dim3 grid(grid_x, grid_y);

        MaxPoolBackwardNHWC<scalar_t, accscalar_t>
        <<<grid, block, nInputPlane * block_x * block_y * sizeof(scalar_t), at::cuda::getCurrentCUDAStream()>>>(
          count,
          gradOutput_data,
          indices_data,
          nbatch,
          nInputPlane, inputHeight, inputWidth, outputHeight, outputWidth,
          kH, kW, dH, dW, padH, padW, dilationH, dilationW,
          out_stride_c, out_stride_h, out_stride_w,
          in_stride_c, in_stride_h, in_stride_w,
          gradInput_data);
      } else {
        dim3 grid;
        const int blocks = (imgcount + BACKWARD_THREADS - 1) / BACKWARD_THREADS;
        grid.x = blocks;
        grid.y = nbatch;
        uint64_t maxGridY = at::cuda::getCurrentDeviceProperties()->maxGridSize[1];
        if (maxGridY < grid.y) grid.y = maxGridY;
        grid.z = nInputPlane;
        uint64_t maxGridZ = at::cuda::getCurrentDeviceProperties()->maxGridSize[2];
        if (maxGridZ < grid.z) grid.z = maxGridZ;

        MaxPoolBackwardNCHW<scalar_t, accscalar_t>
        <<<grid, BACKWARD_THREADS, 0, at::cuda::getCurrentCUDAStream()>>>(
          count,
          gradOutput_data,
          indices_data,
          nbatch,
          nInputPlane, inputHeight, inputWidth, outputHeight, outputWidth,
          kH, kW, dH, dW, padH, padW, dilationH, dilationW,
          gradInput_data);
      }
    }
  );

  TORCH_CHECK(hipGetLastError() == hipSuccess,
    "fractional_max_pool2d_backward_out_cuda failed with error code ",
    hipGetLastError());
}

} // namespace

std::tuple<Tensor&, Tensor&> max_pool2d_with_indices_out_cuda(
  Tensor& output,
  Tensor& indices,
  const Tensor& input,
  IntArrayRef kernel_size,
  IntArrayRef stride,
  IntArrayRef padding,
  IntArrayRef dilation,
  bool ceil_mode)
{
  max_pool2d_with_indices_out_cuda_template(
    output,
    indices,
    input,
    kernel_size,
    stride,
    padding,
    dilation,
    ceil_mode);
  return std::tuple<Tensor&, Tensor&>(output, indices);
}

std::tuple<Tensor, Tensor> max_pool2d_with_indices_cuda(
  const Tensor& input,
  IntArrayRef kernel_size,
  IntArrayRef stride,
  IntArrayRef padding,
  IntArrayRef dilation,
  bool ceil_mode)
{
  Tensor output = at::empty({0}, input.options());
  Tensor indices = at::empty({0}, input.options().dtype(kLong));
  max_pool2d_with_indices_out_cuda_template(
    output,
    indices,
    input,
    kernel_size,
    stride,
    padding,
    dilation,
    ceil_mode);
  return std::tuple<Tensor, Tensor>(output, indices);
}

Tensor& max_pool2d_with_indices_backward_out_cuda(
  Tensor& gradInput,
  const Tensor& gradOutput_,
  const Tensor& input,
  IntArrayRef kernel_size,
  IntArrayRef stride,
  IntArrayRef padding,
  IntArrayRef dilation,
  bool ceil_mode,
  const Tensor& indices)
{
  max_pool2d_with_indices_backward_out_cuda_template(
    gradInput,
    gradOutput_,
    input,
    indices,
    kernel_size,
    stride,
    padding,
    dilation,
    ceil_mode);
  return gradInput;
}

Tensor max_pool2d_with_indices_backward_cuda(
  const Tensor& gradOutput_,
  const Tensor& input,
  IntArrayRef kernel_size,
  IntArrayRef stride,
  IntArrayRef padding,
  IntArrayRef dilation,
  bool ceil_mode,
  const Tensor& indices)
{
  auto gradInput = at::zeros_like(input);
  max_pool2d_with_indices_backward_out_cuda_template(
    gradInput,
    gradOutput_,
    input,
    indices,
    kernel_size,
    stride,
    padding,
    dilation,
    ceil_mode);
  return gradInput;
}

} // at::native
} // at
