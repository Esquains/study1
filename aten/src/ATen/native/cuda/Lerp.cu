#include "hip/hip_runtime.h"
#define TORCH_ASSERT_NO_OPERATORS
#include <ATen/native/Lerp.h>
#include <ATen/Dispatch.h>
#include <ATen/TensorIterator.h>
#include <ATen/native/cuda/Loops.cuh>
#include <ATen/native/cuda/JitLoops.cuh>

namespace at {
namespace native {
namespace {

const char lerp_tensor_name[] = "lerp_kernel";
void lerp_tensor_kernel(at::TensorIteratorBase& iter) {
  auto dtype = iter.common_dtype();
  if(at::isComplexType(dtype)) {
#if AT_USE_JITERATOR()
  static const auto lerp_tensor_string = jiterator_stringify(
      T lerp_tensor_kernel(scalar_t self_val, scalar_t end_val, scalar_t weight_val) {
        template <typename T>
        return (std::abs(weight_val) < 0.5)
            ? self_val + weight_val * (end_val - self_val)
            : end_val -
                (end_val - self_val) * (static_cast<scalar_t>(1) - weight_val);
        }
      ); // lerp_tensor_string
  AT_DISPATCH_COMPLEX_TYPES(dtype, "lerp_cuda", [&] {
        jitted_gpu_kernel<
          /*name=*/ lerp_tensor_name,
          /*return_dtype=*/ scalar_t,
          /*common_dtype=*/ scalar_t,
          /*arity=*/ 3>(iter, lerp_tensor_string);
      });
#else
  AT_DISPATCH_COMPLEX_TYPES(dtype, "lerp_cuda", [&] {
      at::native::gpu_kernel(
        iter,
        [] GPU_LAMBDA(
            scalar_t self_val,
            scalar_t end_val,
            scalar_t weight_val) -> scalar_t {
          return (std:abs(weight_val) < 0.5)
              ? self_val + weight_val * (end_val - self_val)
              : end_val -
                  (end_val - self_val) * (static_cast<scalar_t>(1) - weight_val);
        });
      });
#endif
  } else {
  AT_DISPATCH_FLOATING_TYPES_AND2(
      at::ScalarType::Half, at::ScalarType::BFloat16,
      dtype, "lerp_cuda",
      [&] {
        at::native::gpu_kernel(
            iter,
            [] GPU_LAMBDA(
                scalar_t self_val,
                scalar_t end_val,
                scalar_t weight_val) -> scalar_t {
              return (std::abs(weight_val) < 0.5)
                  ? self_val + weight_val * (end_val - self_val)
                  : end_val -
                      (end_val - self_val) *
                          (static_cast<scalar_t>(1) - weight_val);
            });
      });
  }
}

const char lerp_scalar_name[] = "lerp_kernel";
void lerp_scalar_kernel(at::TensorIteratorBase& iter, const c10::Scalar& weight) {
  auto dtype = iter.common_dtype();
  if (at::isComplexType(dtype)) {
#if AT_USE_JITERATOR()
  static const auto lerp_scalar_string = jiterator_stringify(
      auto weight_val = weight.to<scalar_t>();
      template <typename T>
      T lerp_scalar_kernel(scalar_t self_val, scalar_t end_val) {
        return (std::abs(weight_val) < 0.5)
            ? self_val + weight_val * (end_val - self_val)
            : end_val -
                (end_val - self_val) * (static_cast<scalar_t>(1) - weight_val);
      } // lerp_scalar_string
  );
  AT_DISPATCH_COMPLEX_TYPES(dtype, "lerp_cuda", [&] {
      jitted_gpu_kernel<
        /*name=*/ lerp_scalar_name,
        /*return_dtype=*/ scalar_t,
        /*common_dtype=*/ scalar_t,
        /*arity=*/ 2>(iter, lerp_scalar_string);
  });
#else
  AT_DISPATCH_COMPLEX_TYPES(dtype, "lerp_cuda", [&] {
      auto weight_val = weight.to<scalar_t>();
      gpu_kernel(
        iter,
        [=] GPU_LAMBDA(scalar_t self_val, scalar_t end_val) {
          return (std::abs(weight_val) < 0.5)
              ? self_val + weight_val * (end_val - self_val)
              : end_val -
                  (end_val - self_val) * (static_cast<scalar_t>(1) - weight_val);
         });
  });
#endif
  } else {
  AT_DISPATCH_FLOATING_TYPES_AND2(
      at::ScalarType::Half, at::ScalarType::BFloat16,
      dtype, "lerp_cuda",
      [&]{
        auto weight_val = weight.to<scalar_t>();
        at::native::gpu_kernel(
            iter, [=] GPU_LAMBDA(scalar_t self_val, scalar_t end_val) {
              return (std::abs(weight_val) < 0.5)
                  ? self_val + weight_val * (end_val - self_val)
                  : end_val -
                      (end_val - self_val) * (static_cast<scalar_t>(1) - weight_val);
            });
      });
    }
}

} // anonymous namespace

REGISTER_DISPATCH(lerp_kernel_tensor_weight, &lerp_tensor_kernel);
REGISTER_DISPATCH(lerp_kernel_scalar_weight, &lerp_scalar_kernel);

} // namespace native
} // namespace at
