#include "hip/hip_runtime.h"
#define TORCH_ASSERT_NO_OPERATORS
#include <ATen/native/Lerp.h>
#include <ATen/Dispatch.h>
#include <ATen/TensorIterator.h>
#include <ATen/native/cuda/Loops.cuh>
#include <ATen/native/cuda/JitLoops.cuh>
#include <ATen/OpMathType.h>


namespace at {
namespace native {
namespace {

const char lerp_tensor_name[] = "lerp_tensor_kernel";
void lerp_tensor_kernel(at::TensorIteratorBase& iter) {
  auto dtype = iter.common_dtype();
  if(at::isComplexType(dtype)) {
#if AT_USE_JITERATOR()
  static const auto lerp_tensor_string = jiterator_stringify(
      template <typename T>
      T lerp_tensor_kernel(T self_val, T end_val, T weight_val) {
        return (std::abs(weight_val) < 0.5)
            ? self_val + weight_val * (end_val - self_val)
            : end_val -
                (end_val - self_val) * (static_cast<T>(1) - weight_val);
      }
  ); // lerp_tensor_string
  AT_DISPATCH_COMPLEX_TYPES_AND(kComplexHalf, dtype, "lerp_cuda", [&] {
        jitted_gpu_kernel<
          /*name=*/ lerp_tensor_name,
          /*return_dtype=*/ scalar_t,
          /*common_dtype=*/ scalar_t,
          /*arity=*/ 3>(iter, lerp_tensor_string);
      });
#else
  AT_DISPATCH_COMPLEX_TYPES_AND(kComplexHalf, dtype, "lerp_cuda", [&] {
      at::native::gpu_kernel(
        iter,
        [] GPU_LAMBDA(
            scalar_t self_val,
            scalar_t end_val,
            scalar_t weight_val) -> scalar_t {
          opmath_t self_val_f = self_val;
          opmath_t end_val_f = end_val;
          opmath_t weight_val_f = weight_val;
          return (std:abs(weight_val_f) < 0.5)
              ? self_val_f + weight_val_f * (end_val_f - self_val_f)
              : end_val_f -
                  (end_val_f - self_val_f) * (static_cast<opmath_t>(1) - weight_val_f);
        });
      });
#endif
  } else {
  AT_DISPATCH_FLOATING_TYPES_AND2(
      at::ScalarType::Half, at::ScalarType::BFloat16,
      dtype, "lerp_cuda",
      [&] {
        using opmath_t = at::opmath_type<scalar_t>;
        at::native::gpu_kernel(
            iter,
            [] GPU_LAMBDA(
                scalar_t self_val,
                scalar_t end_val,
                scalar_t weight_val) -> scalar_t {
              opmath_t self_val_f = self_val;
              opmath_t end_val_f = end_val;
              opmath_t weight_val_f = weight_val;
              // Conditional for better numeric. This has been discussed in
              // https://github.com/pytorch/pytorch/pull/18871
              return (std::abs(weight_val_f) < 0.5)
                  ? self_val_f + weight_val_f * (end_val_f - self_val_f)
                  : end_val_f -
                      (end_val_f - self_val_f) *
                          (opmath_t{1} - weight_val_f);
            });
      });
  }
}

void lerp_scalar_kernel(at::TensorIteratorBase& iter, const c10::Scalar& weight) {
  AT_DISPATCH_FLOATING_AND_COMPLEX_TYPES_AND3(
      kComplexHalf, at::ScalarType::Half, at::ScalarType::BFloat16,
      iter.common_dtype(), "lerp_cuda",
      [&]{
        using opmath_t = at::opmath_type<scalar_t>;
        auto weight_val = weight.to<opmath_t>();
        at::native::gpu_kernel(
            iter, [=] GPU_LAMBDA(scalar_t self_val, scalar_t end_val) {
              opmath_t self_val_f = self_val;
              opmath_t end_val_f = end_val;
              // Conditional for better numeric. This has been discussed in
              // https://github.com/pytorch/pytorch/pull/18871
              return (std::abs(weight_val) < 0.5)
                  ? self_val_f + weight_val * (end_val_f - self_val_f)
                  : end_val_f -
                      (end_val_f - self_val_f) * (opmath_t{1} - weight_val);
            });
      });
    }

} // anonymous namespace

REGISTER_DISPATCH(lerp_kernel_tensor_weight, &lerp_tensor_kernel);
REGISTER_DISPATCH(lerp_kernel_scalar_weight, &lerp_scalar_kernel);

} // namespace native
} // namespace at
