#include "hip/hip_runtime.h"
#define TORCH_ASSERT_NO_OPERATORS
#include <ATen/native/cuda/TensorTopK.h>
#include <ATen/core/TensorBase.h>
#include <ATen/ceil_div.h>
#include <ATen/Dispatch.h>
#include <ATen/cuda/detail/TensorInfo.cuh>
#include <ATen/cuda/detail/OffsetCalculator.cuh>
#include <ATen/cuda/ScanUtils.cuh>
#include <ATen/cuda/AsmUtils.cuh>
#include <ATen/cuda/DeviceUtils.cuh>
#include <ATen/native/cuda/SortingCommon.cuh>
#include <ATen/native/cuda/SortingRadixSelect.cuh>
#include <ATen/native/cuda/SortUtils.cuh>
#include <ATen/cuda/cub.cuh>
#include <c10/cuda/CUDACachingAllocator.h>

#include <c10/macros/Macros.h>

using namespace at::native;

namespace at {
namespace native {
namespace sbtopk {  // single_block_topk

template <typename T>
struct AddOp {
  __device__ __forceinline__ T operator()(T const &lhs, T const &rhs) {
    return (lhs + rhs);
  }
};

template <typename T, typename IndexType, int Dim, bool WithKthValues>
C10_LAUNCH_BOUNDS_1(1024)
__global__ void gatherTopK(at::cuda::detail::TensorInfo<T, IndexType> input,
                           IndexType inputSliceSize,
                           IndexType outputSliceSize, // aka `k`
                           bool largest,

                           IndexType numInputSlices,
                           IndexType inputWithinSliceStride,

                           at::cuda::detail::TensorInfo<T, IndexType> topK,
                           IndexType topKWithinSliceStride,

                           at::cuda::detail::TensorInfo<int64_t, IndexType> indices,
                           IndexType indicesWithinSliceStride,
                           T* kthValues) {
  // Indices are limited to integer fp precision, so counts can fit in
  // int32, regardless of IndexType
#if defined(USE_ROCM)
  __shared__ int smem[64];
#else
  __shared__ int smem[32]; // one per each warp, up to warp limit
#endif
  IndexType slice = getLinearBlockId<IndexType>();
  if (slice >= numInputSlices) {
    return;
  }

  // Find the start offset for our slice
  IndexType sliceStartIndex =
    at::cuda::detail::IndexToOffset<T, IndexType, Dim>::get(slice, input);
  IndexType topKSliceStartIndex =
    at::cuda::detail::IndexToOffset<T, IndexType, Dim>::get(slice, topK);
  IndexType indicesSliceStartIndex =
    at::cuda::detail::IndexToOffset<int64_t, IndexType, Dim>::get(slice, indices);

  T* inputSliceStart = &input.data[sliceStartIndex];
  T* topKSliceStart = &topK.data[topKSliceStartIndex];
  int64_t* indicesSliceStart = &indices.data[indicesSliceStartIndex];

  // Find the k-th highest element in our input
  T topKValue;
  if (WithKthValues){
    topKValue = kthValues[slice];
  } else {
    topKValue = static_cast<T>(0);
    radixSelect<T, typename TopKTypeConfig<T>::RadixType, IndexType>(
      inputSliceStart, outputSliceSize, largest,
      inputSliceSize, inputWithinSliceStride,
      smem, &topKValue);
  }
  const auto topKConverted = at::native::TopKTypeConfig<T>::convert(topKValue);

  // Every value that is strictly less/greater than `pattern`
  // (depending on sort dir) in sorted int format is in the top-K.
  // The top-K value itself might not be unique.
  //
  // Since there are a variable number of elements that we see that
  // are within the top-k, we don't know at what index to write out
  // the resulting values.
  // In order to get this, we perform an exclusive prefix sum of
  // `hasTopK`. This will return the resulting index into which we
  // need to write the result, if a thread has a result.

  // All threads need to participate in the loop and the prefix sum,
  // but not necessarily in the load; hence loop bounds being rounded
  // up to a multiple of the block dim.
  IndexType numIterations = round_up(inputSliceSize, (IndexType) blockDim.x);
  IndexType writeIndexStart = 0;

  for (IndexType i = threadIdx.x; i < numIterations; i += blockDim.x) {
    bool inRange = (i < inputSliceSize);
    T v =
      inRange ? doLdg(&inputSliceStart[i * inputWithinSliceStride]) : static_cast<T>(0);
    const auto convertedV = at::native::TopKTypeConfig<T>::convert(v);
    bool hasTopK;
    if (largest) {
      hasTopK = inRange && (convertedV > topKConverted);
    } else {
      hasTopK = inRange && (convertedV < topKConverted);
    }

    int index;
    int carry;
    at::cuda::exclusiveBinaryPrefixScan<int, true>(
        smem, hasTopK, &index, &carry, AddOp<int>());

    if (hasTopK) {
      int writeIndex = writeIndexStart + index;
      CUDA_KERNEL_ASSERT(writeIndex < outputSliceSize);

      IndexType topKOffset = writeIndex * topKWithinSliceStride;
      IndexType indexOffset = writeIndex * indicesWithinSliceStride;

      topKSliceStart[topKOffset] = v;
      indicesSliceStart[indexOffset] = i;
    }

    writeIndexStart += carry;
  }

  // We need to fill in the rest with actual == top-K values.
  // The number that we need is outputSliceSize -
  // writeIndexStart. There might be more than that number available,
  // in which case we have to choose the first seen set. We do this
  // via a prefix sum to calculate indices for writing results.
  CUDA_KERNEL_ASSERT(outputSliceSize >= writeIndexStart);
  IndexType topKRemaining = (outputSliceSize - writeIndexStart);

  for (IndexType i = threadIdx.x; i < numIterations; i += blockDim.x) {
    bool inRange = (i < inputSliceSize);
    T v =
      inRange ? doLdg(&inputSliceStart[i * inputWithinSliceStride]) : static_cast<T>(0);
    const auto convertedV = at::native::TopKTypeConfig<T>::convert(v);
    bool hasTopK = inRange && (convertedV == topKConverted);

    int index;
    int carry;
    at::cuda::exclusiveBinaryPrefixScan<int, true>(
        smem, hasTopK, &index, &carry, AddOp<int>());

    if (hasTopK && index < topKRemaining) {
      int writeIndex = writeIndexStart + index;
      CUDA_KERNEL_ASSERT(writeIndex < outputSliceSize);

      IndexType topKOffset = writeIndex * topKWithinSliceStride;
      IndexType indexOffset = writeIndex * indicesWithinSliceStride;

      topKSliceStart[topKOffset] = v;
      indicesSliceStart[indexOffset] = i;
    }

    if (carry >= topKRemaining) {
      break;
    }

    topKRemaining -= carry;
    writeIndexStart += carry;
  }

};

template <typename T, typename IndexType, int Dim>
void launch(
    at::cuda::detail::TensorInfo<T, IndexType> input,
    IndexType inputSliceSize,
    IndexType outputSliceSize, // aka `k`
    bool largest,

    IndexType numInputSlices,
    IndexType inputWithinSliceStride,

    at::cuda::detail::TensorInfo<T, IndexType> topK,
    IndexType topKWithinSliceStride,

    at::cuda::detail::TensorInfo<int64_t, IndexType> indices,
    IndexType indicesWithinSliceStride) {

    dim3 grid;
    TORCH_INTERNAL_ASSERT(getGridFromTiles(numInputSlices, grid), "Too many slices to sort");
    dim3 block(std::min(
        at::ceil_div((int64_t)inputSliceSize, (int64_t)C10_WARP_SIZE) * (int64_t)C10_WARP_SIZE, (int64_t)1024));
    gatherTopK<T, IndexType, Dim, /* WithKthValues= */false><<<grid, block, 0, c10::cuda::getCurrentCUDAStream()>>>(
        input,
        inputSliceSize,
        outputSliceSize,
        largest,
        numInputSlices,
        inputWithinSliceStride,
        topK,
        topKWithinSliceStride,
        indices,
        indicesWithinSliceStride,
        nullptr);
    C10_CUDA_KERNEL_LAUNCH_CHECK();
}
} // namespace at::native::sbtopk

namespace mbtopk {  // multi_block_topk

constexpr int BLOCK_THREADS = 128;
// in principle, we could write at most 255 into digit counter (in shared mem) with unsigned char type
// TODO tune this, maybe smaller

// Over what radix we are selecting values
constexpr int RADIX_BITS = 6; // digits are base-(2 ^ RADIX_BITS)
constexpr int RADIX_DIGITS = 1 << RADIX_BITS; // 2 ^ RADIX_BITS
constexpr int RADIX_MASK = (RADIX_DIGITS - 1);

// A stateful callback functor that maintains a running prefix to be applied
// during consecutive scan operations.
struct BlockPrefixCallbackOp {
  // Running prefix
  int running_total;
  // Constructor
  __device__ BlockPrefixCallbackOp(int running_total) : running_total(running_total) {}
  // Callback operator to be entered by the first warp of threads in the block.
  // Thread-0 is responsible for returning a value for seeding the block-wide scan.
  __device__ int operator()(int block_aggregate) {
    int old_prefix = running_total;
    running_total += block_aggregate;
    return old_prefix;
  }
};

template <typename T>
__global__ void fill(T* x, T value, int size) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  for (int i = idx; i < size; i += gridDim.x * blockDim.x) {
    x[i] = value;
  }
}

template <typename T, typename IndexType, typename Bitwise, int Dim, int RADIX_BITS>
C10_LAUNCH_BOUNDS_1(BLOCK_THREADS)
__global__ void radixFindKthValues(
    at::cuda::detail::TensorInfo<T, IndexType> input,
    IndexType inputSliceSize,
    IndexType* ksToFind, // aka `k`
    bool largest,

    IndexType numInputSlices,
    IndexType withinSliceStride,

    int current_bit,
    int items_per_thread,
    IndexType blocks_per_slice,
    Bitwise desiredMask,

    // outputs
    int* semaphores,
    Bitwise* desires,
    IndexType* counts,
    T* kthValues // only writes when current_bit reaches 0
) {
  constexpr int RADIX_DIGITS = 1 << RADIX_BITS; // 2 ^ RADIX_BITS
  constexpr int PACKING_RATIO = sizeof(int) / sizeof(unsigned char);
  constexpr int COUNTER_LANES = RADIX_DIGITS / PACKING_RATIO;

  int items_per_block = items_per_thread * BLOCK_THREADS;
  int tidx = threadIdx.x;
  IndexType block_idx = getLinearBlockId<IndexType>();
  IndexType slice_idx = block_idx / blocks_per_slice;
  IndexType blk_idx_in_slice = block_idx % blocks_per_slice;
  if (slice_idx >= numInputSlices) {
    return;
  }

  Bitwise desired = desires[slice_idx];
  IndexType kToFind = ksToFind[slice_idx];
  IndexType sliceStartIndex = at::cuda::detail::IndexToOffset<T, IndexType, Dim>::get(slice_idx, input);
  T* data = &input.data[sliceStartIndex];

  typedef hipcub::BlockScan<IndexType, BLOCK_THREADS> BlockScan;
  union __align__(16) TempStorage {
    unsigned char thread_counters[COUNTER_LANES][BLOCK_THREADS]
                                 [PACKING_RATIO]; // threads in a warp is guaranteed to access different banks
    uint32_t packed_thread_counters[COUNTER_LANES][BLOCK_THREADS];
    struct {
      IndexType digit_count_cumsum[RADIX_DIGITS];
      typename BlockScan::TempStorage temp_storage;
    } scan_storage;
  };
  __shared__ TempStorage temp_storage;
  __shared__ int s_block_counts[RADIX_DIGITS]; // only used when blk_per_slice == 1

  // reset temp_storage
  for (int i = 0; i < COUNTER_LANES; ++i) {
    temp_storage.packed_thread_counters[i][tidx] = 0;
  }
  __syncthreads();

  items_per_thread = (blk_idx_in_slice + 1 < blocks_per_slice)
      ? items_per_thread
      : at::ceil_div((int64_t)(inputSliceSize - blk_idx_in_slice * items_per_block), (int64_t)BLOCK_THREADS);

  // collect counts and store in shared memorey for each thread
  for (int i = 0; i < items_per_thread; ++i) {
    // Find the start offset for our slice
    IndexType idx = tidx + i * BLOCK_THREADS + blk_idx_in_slice * items_per_block;
    if (idx < inputSliceSize) {
      idx *= withinSliceStride;
      Bitwise val = TopKTypeConfig<T>::convert(doLdg(&data[idx]));
      bool hasVal = ((val & desiredMask) == (desired & desiredMask));
      Bitwise digit = at::cuda::Bitfield<Bitwise>::getBitfield(val, current_bit, RADIX_BITS);
      if (hasVal) {
        temp_storage.thread_counters[digit / PACKING_RATIO][tidx][digit % PACKING_RATIO]++; // threads in a warp is guaranteed to access different banks
      }
    }
  }

  __syncthreads();

  // extract counts and write count out
  for (int i = 0; i < (RADIX_DIGITS + BLOCK_THREADS - 1) / BLOCK_THREADS; ++i) {
    // every thread collects one overall digit count stored in shared mem for each thread
    int digit_count = 0;
    int digit = i * BLOCK_THREADS + tidx;
    if (digit < RADIX_DIGITS) {
      for (int j = 0, idx = tidx; j < BLOCK_THREADS; ++j, idx = (idx + 1) % BLOCK_THREADS) { // every thread access different bank
        digit_count += temp_storage.thread_counters[digit / PACKING_RATIO][idx][digit % PACKING_RATIO];
      }
      if (blocks_per_slice == 1) {
        s_block_counts[digit] = digit_count;
      } else {
        counts[block_idx * RADIX_DIGITS + digit] = digit_count;
      }
    }
  }

  __threadfence(); // make sure writes are globally visible
  __syncthreads();

  __shared__ bool s_is_last_block_done;
  __shared__ bool s_desired_found;

  if (tidx == 0) {
    if (blocks_per_slice == 1) {
      s_is_last_block_done = true;
    } else {
      int blocks_finished_old = atomicAdd(&semaphores[slice_idx], 1);
      s_is_last_block_done = (blocks_finished_old == blocks_per_slice - 1);
    }
    s_desired_found = false;
  }

  __syncthreads();

  // last block for each slice accumulate counts from blocks and update desired
  if (s_is_last_block_done) {
    // sum block counts
    BlockPrefixCallbackOp prefix_op(0);

    auto post_process = [&](int digit) {
      IndexType digit_count = 0;
      IndexType& digit_count_cumsum = digit_count;
      for (int blk = 0; blk < blocks_per_slice; ++blk) {
        if (blocks_per_slice == 1) {
          digit_count += s_block_counts[digit];
        } else {
          digit_count += counts[(slice_idx * blocks_per_slice + blk) * RADIX_DIGITS + digit];
        }
      }

      // Collectively compute the block-wide exclusive prefix sum
      BlockScan(temp_storage.scan_storage.temp_storage).InclusiveSum(digit_count, digit_count_cumsum, prefix_op);
      __syncthreads();
      temp_storage.scan_storage.digit_count_cumsum[digit] = digit_count_cumsum;
      __syncthreads();

      // update desired
      IndexType digit_count_cumsum_left;
      if (largest) {
        digit_count_cumsum_left =
            (digit == RADIX_DIGITS - 1) ? 0 : temp_storage.scan_storage.digit_count_cumsum[digit + 1];
      } else {
        digit_count_cumsum_left = (digit == 0) ? 0 : temp_storage.scan_storage.digit_count_cumsum[digit - 1];
      }
      if (digit_count_cumsum_left < kToFind && kToFind <= digit_count_cumsum) {
        desired = at::cuda::Bitfield<Bitwise>::setBitfield(desired, digit, current_bit, RADIX_BITS);
        if (current_bit > 0) {
          desires[slice_idx] = desired;
          ksToFind[slice_idx] = kToFind - digit_count_cumsum_left;
        } else {
          kthValues[slice_idx] = TopKTypeConfig<T>::deconvert(desired);
        }
        s_desired_found = true;
      }
      __syncthreads();
    };

    if (largest) {
      for (int digit = RADIX_DIGITS - tidx - 1; digit >= 0 && !s_desired_found; digit -= BLOCK_THREADS) {
        post_process(digit);
      }
    } else {
      for (int digit = tidx; digit < RADIX_DIGITS && !s_desired_found; digit += BLOCK_THREADS) {
        post_process(digit);
      }
    }

    if (tidx == 0) {
      semaphores[slice_idx] = 0;
    }
  }
};

template <typename T, typename IndexType, int Dim>
void launch(
    at::cuda::detail::TensorInfo<T, IndexType> input,
    IndexType inputSliceSize,
    IndexType outputSliceSize, // aka `k`
    bool largest,

    IndexType numInputSlices,
    IndexType inputWithinSliceStride,

    at::cuda::detail::TensorInfo<T, IndexType> topK,
    IndexType topKWithinSliceStride,

    at::cuda::detail::TensorInfo<int64_t, IndexType> indices,
    IndexType indicesWithinSliceStride) {

  // configure items_per_thread
  hipDeviceProp_t* prop = at::cuda::getCurrentDeviceProperties();
  int mpc = prop->multiProcessorCount;
  int shared_per_mp = prop->sharedMemPerMultiprocessor;
  int reserved_shared_per_block = 0;
#if defined(CUDA_VERSION) && CUDA_VERSION >= 11000
  reserved_shared_per_block = prop->reservedSharedMemPerBlock;
#endif
  constexpr int static_shared_per_block = RADIX_DIGITS * BLOCK_THREADS + RADIX_DIGITS * sizeof(int);
  int shared_per_block = static_shared_per_block + reserved_shared_per_block;
  int blocks_per_mp = std::min(shared_per_mp / shared_per_block, prop->maxBlocksPerMultiProcessor);
  int items_per_thread = at::ceil_div((int64_t)(inputSliceSize * numInputSlices), (int64_t)(mpc * blocks_per_mp * BLOCK_THREADS));
  items_per_thread = std::max(4, std::min(items_per_thread, 64)); // clamp to (4, 64)
  int items_per_block = items_per_thread * BLOCK_THREADS;

  using Bitwise = typename TopKTypeConfig<T>::RadixType;
  int64_t blocks_per_slice = at::ceil_div((int64_t)inputSliceSize, (int64_t)items_per_block);
  int64_t num_blocks = numInputSlices * blocks_per_slice;

  // temporary storage
  auto& allocator = *c10::cuda::CUDACachingAllocator::get();

  auto kthValues_buffer = allocator.allocate(numInputSlices * sizeof(T));
  T* kthValues = reinterpret_cast<T*>(kthValues_buffer.get());
  AT_CUDA_CHECK(hipMemsetAsync(kthValues, 0, numInputSlices * sizeof(T), c10::cuda::getCurrentCUDAStream()));

  auto semaphores_buffer = allocator.allocate(numInputSlices * sizeof(int));
  int* semaphores = reinterpret_cast<int*>(semaphores_buffer.get());
  AT_CUDA_CHECK(hipMemsetAsync(semaphores, 0, numInputSlices * sizeof(int), c10::cuda::getCurrentCUDAStream()));

  auto ksToFind_buffer = allocator.allocate(numInputSlices * sizeof(IndexType));
  IndexType* ksToFind = reinterpret_cast<IndexType*>(ksToFind_buffer.get());
  fill<IndexType><<<std::min((numInputSlices + 511) / 512, (IndexType)65535), 512, 0, c10::cuda::getCurrentCUDAStream()>>>(
          ksToFind, outputSliceSize, numInputSlices);
  C10_CUDA_KERNEL_LAUNCH_CHECK();

  auto desired_buffer = allocator.allocate(numInputSlices * sizeof(Bitwise));
  Bitwise* desired = reinterpret_cast<Bitwise*>(desired_buffer.get());

  auto counts_buffer = allocator.allocate(num_blocks * RADIX_DIGITS * sizeof(IndexType));
  IndexType* counts = reinterpret_cast<IndexType*>(counts_buffer.get());

  Bitwise desiredMask = 0;
  dim3 grid;
  TORCH_INTERNAL_ASSERT(getGridFromTiles(num_blocks, grid), "Too many slices to sort");
  dim3 block(BLOCK_THREADS);

#define RUN_K(BIT)                                             \
  radixFindKthValues<T, IndexType, Bitwise, Dim, BIT>   \
      <<<grid, block, 0, c10::cuda::getCurrentCUDAStream()>>>( \
          input,                                               \
          inputSliceSize,                                      \
          ksToFind,                                            \
          largest,                                             \
          numInputSlices,                                      \
          inputWithinSliceStride,                              \
          current_bit,                                         \
          items_per_thread,                                     \
          blocks_per_slice,                                    \
          desiredMask,                                         \
          semaphores,                                          \
          desired,                                             \
          counts,                                              \
          kthValues);                                          \
  C10_CUDA_KERNEL_LAUNCH_CHECK();

#define RUN_BIT()                                              \
  if (radix_bits == 6) {                                       \
    RUN_K(6);                                                  \
  } else if (radix_bits == 4) {                                \
    RUN_K(4);                                                  \
  } else if (radix_bits == 2) {                                \
    RUN_K(2);                                                  \
  } else {                                                     \
    TORCH_INTERNAL_ASSERT(                                     \
        false, "RADIX_BIT ", radix_bits, " is not supported"); \
  }

  int current_bit = sizeof(T) * 8 - RADIX_BITS;
  int radix_bits = RADIX_BITS;
  for (; current_bit > 0; current_bit -= RADIX_BITS) {
    RUN_BIT();
    desiredMask = at::cuda::Bitfield<Bitwise>::setBitfield(
        desiredMask, RADIX_MASK, current_bit, RADIX_BITS);
  }
  radix_bits = current_bit + RADIX_BITS;
  current_bit = 0;
  RUN_BIT();

#undef RUN_BIT
#undef RUN_K

  // Find topk values based on kth value
  {
    dim3 grid;
    TORCH_INTERNAL_ASSERT(getGridFromTiles(numInputSlices, grid), "Too many slices to sort");
    dim3 block(std::min(at::ceil_div((int64_t)inputSliceSize, (int64_t)C10_WARP_SIZE) * (int64_t)C10_WARP_SIZE, (int64_t)1024));
    sbtopk::gatherTopK<T, IndexType, Dim, /* WithKthValues= */true><<<grid, block, 0, c10::cuda::getCurrentCUDAStream()>>>(
            input,
            inputSliceSize,
            outputSliceSize,
            largest,
            numInputSlices,
            inputWithinSliceStride,
            topK,
            topKWithinSliceStride,
            indices,
            indicesWithinSliceStride,
            kthValues);
    C10_CUDA_KERNEL_LAUNCH_CHECK();
  }
}

} // namespace at::native::mbtopk

bool should_use_multiblock(int64_t num_slices, int64_t slice_size) {
  // This heuristics is based on the experiment in https://github.com/pytorch/pytorch/pull/71081
  return (num_slices < 1000 && slice_size >= 20000) || (num_slices >= 1000 && slice_size >= 800);
}

void launch_gather_topk_kernel(
    const TensorBase& self, int64_t k, int64_t dim, bool largest,
    const TensorBase& values, const TensorBase& indices) {
  int numDims = self.dim();
  numDims = numDims == 0 ? 1 : numDims;
  TORCH_CHECK(numDims <= MAX_DIMS, "input tensor has too many dimensions");
  int64_t sliceSize = self.dim() == 0 ? 1 : self.size(dim);

  auto input = self.contiguous();
  // static_cast is required to ensure that the correct type (INDEX_T)
  // is provided to the kernel for the arguments.
#define RUN_K(INDEX_T, DIM, LAUNCH_FUNCTION_NAME)                       \
  LAUNCH_FUNCTION_NAME<scalar_t, INDEX_T, DIM>(                         \
      inputInfo,                                                        \
      static_cast<INDEX_T>(sliceSize),                                  \
      static_cast<INDEX_T>(k),                                          \
      largest,                                                          \
      static_cast<INDEX_T>(numInputSlices),                             \
      /* The actual dimension that the k-selection is running in */     \
      /* may have changed from collapseDims() */                        \
      static_cast<INDEX_T>(inputInfo.strides[collapseInputDim]),        \
      topKInfo,                                                         \
      static_cast<INDEX_T>(topKInfo.strides[collapseTopKDim]),          \
      indicesInfo,                                                      \
      static_cast<INDEX_T>(indicesInfo.strides[collapseIndicesDim]));

#define RUN_MB(INDEX_T, DIM)                                            \
  if (should_use_multiblock(numInputSlices, sliceSize)) {               \
    RUN_K(INDEX_T, DIM, mbtopk::launch);                                \
  } else {                                                              \
    RUN_K(INDEX_T, DIM, sbtopk::launch);                                \
  }

#define RUN_DIM(INDEX_T)                        \
  if (allDims == 1) {                           \
    RUN_MB(INDEX_T, 1);                         \
  } else if (allDims == 2) {                    \
    RUN_MB(INDEX_T, 2);                         \
  } else if (allDims == 3) {                    \
    RUN_MB(INDEX_T, 3);                         \
  } else {                                      \
    RUN_MB(INDEX_T, -1);                        \
  }

#define RUN_T(INDEX_T)                                                  \
  AT_DISPATCH_ALL_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, input.scalar_type(), "topk_out_cuda", [&] { \
    at::cuda::detail::TensorInfo<scalar_t, INDEX_T> inputInfo =           \
      at::cuda::detail::getTensorInfo<scalar_t, INDEX_T>(input);          \
    at::cuda::detail::TensorInfo<scalar_t, INDEX_T> topKInfo =            \
      at::cuda::detail::getTensorInfo<scalar_t, INDEX_T>(values);         \
    at::cuda::detail::TensorInfo<int64_t, INDEX_T> indicesInfo =          \
      at::cuda::detail::getTensorInfo<int64_t, INDEX_T>(indices);         \
    /* tensorInfoLegacyIfScalar*/                                         \
    if (!input.dim()) {                                                   \
      inputInfo.dims = 1;                                                 \
      inputInfo.sizes[0] = 1;                                             \
      inputInfo.strides[0] = 1;                                           \
      topKInfo.dims = 1;                                                  \
      topKInfo.sizes[0] = 1;                                              \
      topKInfo.strides[0] = 1;                                            \
      indicesInfo.dims = 1;                                               \
      indicesInfo.sizes[0] = 1;                                           \
      indicesInfo.strides[0] = 1;                                         \
    }                                                                     \
    /* We use these structures solely to find the offset to */            \
    /* each slice we are operating on */                                  \
    inputInfo.sizes[dim] = 1;                                             \
    topKInfo.sizes[dim] = 1;                                              \
    indicesInfo.sizes[dim] = 1;                                           \
    /* stash the stride of dim because it can be accidentally collapsed */ \
    auto strideTopK = topKInfo.strides[dim];                              \
    auto strideIndices = indicesInfo.strides[dim];                        \
    /* Collapse all other dims */                                         \
    int collapseInputDim = inputInfo.collapseDims(dim);                   \
    int collapseTopKDim = topKInfo.collapseDims(dim);                     \
    int collapseIndicesDim = indicesInfo.collapseDims(dim);               \
    /* restore stride in case it was collapsed */                         \
    topKInfo.strides[collapseTopKDim] = strideTopK;                       \
    indicesInfo.strides[collapseIndicesDim] = strideIndices;              \
    int64_t numInputSlices = 1;                                           \
    for (int i = 0; i < inputInfo.dims; ++i) {                            \
      numInputSlices *= inputInfo.sizes[i];                               \
    }                                                                     \
                                                                          \
    /* This is used as a template parameter to calculate indices. */      \
    /* We only specialize it if all collapsed dim sizes are the */        \
    /* same; otherwise, we use -1 which is the specialization */          \
    /* parameter for arbitrary dimensions */                              \
    int allDims = inputInfo.dims;                                         \
    if (topKInfo.dims != allDims || indicesInfo.dims != allDims) {        \
      allDims = -1;                                                       \
    }                                                                     \
                                                                          \
    RUN_DIM(INDEX_T);                                                     \
  });

  // the below is safe with 0-dimensional tensors because it is based on
  // TensorInfo which implicitly expands to 1-dimensional.
  if (input.numel() > 0) {
    // Based on required index size, run the algorithm with the
    // appropriate index type
    if (at::cuda::detail::canUse32BitIndexMath(input) &&
        at::cuda::detail::canUse32BitIndexMath(values) &&
        at::cuda::detail::canUse32BitIndexMath(indices)) {
      RUN_T(uint32_t);
    } else {
      RUN_T(uint64_t);
    }
  }
#undef RUN_T
#undef RUN_DIM
#undef RUN_K
}

} // at::native
} // at
