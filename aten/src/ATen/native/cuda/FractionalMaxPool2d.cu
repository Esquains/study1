#include "hip/hip_runtime.h"
#include "ATen/ATen.h"
#include "ATen/AccumulateType.h"
#include "ATen/cuda/CUDAApplyUtils.cuh"
#include "ATen/cuda/HIPContext.h"
#include "ATen/cuda/detail/IndexUtils.cuh"
#include "ATen/cuda/detail/TensorInfo.cuh"
#include "ATen/cuda/detail/KernelUtils.h"
#include "ATen/NativeFunctions.h"
#include "ATen/TensorUtils.h"
#include "ATen/Utils.h"
#include "c10/util/Exception.h"

#include <algorithm>
#include <cfloat>
#include <cmath>

namespace at {
namespace native {

using namespace at::cuda::detail;

namespace {

template <typename scalar_t, typename accscalar_t>
__device__ inline int get_interval(accscalar_t sample,
  int index, int inputSize, int outputSize, int poolSize) {
  accscalar_t alpha = (accscalar_t)(inputSize - poolSize) / (accscalar_t) (outputSize - 1);
  if (index == outputSize - 1) {
    return inputSize - poolSize;
  } else {
    return (int) ((index + sample) * alpha) - (int) (sample * alpha);
  }
}

template <typename scalar_t, typename index_t>
__device__ inline scalar_t* get_ref_by_coord(
  TensorInfo<scalar_t, index_t> tensor_info,
  int ndims, index_t batch, index_t plane,
  index_t h, index_t w) {
  index_t offset = plane * tensor_info.strides[ndims - 3] +
    h * tensor_info.strides[ndims - 2] + w * tensor_info.strides[ndims - 1];
  if(ndims == 4) {
    offset += batch * tensor_info.strides[0];
  }
  return tensor_info.data + offset;
}

template <typename scalar_t>
__global__ void fractional_max_pool2d_out_frame(
  TensorInfo<scalar_t, int> input,
  TensorInfo<scalar_t, int> output,
  TensorInfo<int64_t, int> indices,
  TensorInfo<scalar_t, int> samples,
  IntList pool_size,
  int PoolSizeWStatic) {

  using accscalar_t = at::acc_type<scalar_t, true>;

  int poolSizeH = pool_size[0];
  int poolSizeW = pool_size[1];
  int ourOutputPoint = threadIdx.x + blockIdx.x * blockDim.x;
  int plane = blockIdx.y;
  int batch = blockIdx.z;

  int ndims = output.dims;

  // Each thread generates a specific output point
  if (ourOutputPoint < output.sizes[ndims - 2] * output.sizes[ndims - 1]) {
    int outputW = ourOutputPoint % output.sizes[ndims - 1];
    int outputH = ourOutputPoint / output.sizes[ndims - 1];

    int poolW = get_interval<scalar_t, accscalar_t>(
      static_cast<accscalar_t>(*(samples.data + batch * samples.strides[0] +
        plane * samples.strides[1])),
        outputW, input.sizes[ndims - 1], output.sizes[ndims - 1], poolSizeW);
    int poolH = get_interval<scalar_t, accscalar_t>(
      static_cast<accscalar_t>(*(samples.data + batch * samples.strides[0] +
        plane * samples.strides[1] + samples.strides[2])),
        outputH, input.sizes[ndims - 2], output.sizes[ndims - 2], poolSizeH);

    scalar_t maxVal = at::numeric_limits<scalar_t>::lowest();
    int maxIndex = -1;

    for (int h = poolH; h < poolH + poolSizeH; ++h) {
      if (PoolSizeWStatic == -1) {
        for (int w = poolW; w < poolW + poolSizeW; ++w) {
          scalar_t val = *get_ref_by_coord<scalar_t, int>(
            input, ndims, batch, plane, h, w);
          // for consistency with THNN, favor the first max
          if (val > maxVal) {
            maxIndex = h * input.sizes[ndims - 1] + w;
            maxVal = val;
          }
        }
      } else {
#pragma unroll
        for (int i = 0; i < PoolSizeWStatic; ++i) {
          int w = i + poolW;
          scalar_t val = *get_ref_by_coord<scalar_t, int>(
            input, ndims, batch, plane, h, w);
          // for consistency with THNN, favor the first max
          if (val > maxVal) {
            maxIndex = h * input.sizes[ndims - 1] + w;
            maxVal = val;
          }
        }
      }
    }

    assert(maxVal != at::numeric_limits<scalar_t>::lowest());
    assert(maxIndex != -1);

    auto indices_ref = get_ref_by_coord(indices, ndims,
      batch, plane, outputH, outputW);
    auto output_ref = get_ref_by_coord(output, ndims,
      batch, plane, outputH, outputW);
    *indices_ref = maxIndex;
    *output_ref = maxVal;
  }
}

template <typename scalar_t>
__global__ void fractional_max_pool2d_backward_out_frame(
  TensorInfo<scalar_t, int> gradInput,
  TensorInfo<scalar_t, int> gradOutput,
  TensorInfo<int64_t, int> indices) {
  // Output (h, w) point that this thread is responsible for
  int ourOutputPoint = threadIdx.x + blockIdx.x * blockDim.x;
  int plane = blockIdx.y;
  int batch = blockIdx.z;

  int ndims = gradOutput.dims;

  // Each thread generates a specific output point
  if (ourOutputPoint < gradOutput.sizes[ndims - 2] *
    gradOutput.sizes[ndims - 1]) {
    int outputW = ourOutputPoint % gradOutput.sizes[ndims - 1];
    int outputH = ourOutputPoint / gradOutput.sizes[ndims - 1];

    int index = *get_ref_by_coord<int64_t, int>(
      indices, ndims, batch, plane, outputH, outputW);
    assert(index >= 0);
    int inputW = index % gradInput.sizes[ndims - 1];
    int inputH = index / gradInput.sizes[ndims - 1];
    assert(inputH < gradInput.sizes[ndims - 2]);

    atomicAdd(
      get_ref_by_coord<scalar_t, int>(
        gradInput, ndims, batch, plane, inputH, inputW),
      *get_ref_by_coord<scalar_t, int>(
        gradOutput, ndims, batch, plane, inputH, inputW));
  }
}

void fractional_max_pool2d_out_cuda_template(
  Tensor & output,
  Tensor& indices,
  const Tensor& input,
  IntList pool_size,
  IntList output_size,
  const Tensor& randomSamples) {
  int planeDim = 0;
  int dimh = 1;
  int dimw = 2;
  int64_t numBatch = 1;

  int ndims = input.ndimension();
  for (int64_t i = 0; i < ndims; i++) {
     AT_CHECK(input.size(i) > 0,
       "fractional_max_pool2d(): expected input to have non-empty spatial dimensions, "
       "but input has sizes ", input.sizes(), " with dimension ", i, " being "
       "empty");
   }

   AT_CHECK((ndims == 3 || ndims == 4),
     "non-empty 3D or 4D (batch mode) tensor expected for input");

  if (ndims == 4) {
    numBatch = input.size(0);
    planeDim++;
    dimh++;
    dimw++;
  }

  /* sizes */
  int64_t numPlanes = input.size(planeDim);
  int64_t inputH = input.size(dimh);
  int64_t inputW = input.size(dimw);

  int64_t outputH = output_size[0];
  int64_t outputW = output_size[1];
  int64_t poolSizeH = pool_size[0];
  int64_t poolSizeW = pool_size[1];

  AT_CHECK(outputH + poolSizeH - 1 <= inputH,
             "fractional_max_pool2d(): pool_size height ", poolSizeH,
             " too large relative to input height ", inputH);
  AT_CHECK(outputW + poolSizeW - 1 <= inputW,
           "pool_size width ", poolSizeW,
           " too large relative to input width ", inputW);

  if (ndims == 3) {
    /* resize output */
    output.resize_({numPlanes, outputH, outputW});
    /* indices will contain the locations for each output point */
    indices.resize_({numPlanes, outputH, outputW});
  } else {
    output.resize_({numBatch, numPlanes, outputH, outputW});
    indices.resize_({numBatch, numPlanes, outputH, outputW});
  }

  // block is limited to 4 warps
  // grid handles overflow per each plane
  int outputPlaneSize = output.size(ndims - 2) *
    output.size(ndims - 1);
  dim3 grid((outputPlaneSize + 127) / 128,
            input.size(ndims - 3),
            ndims == 3 ? 1 : input.size(0));
  dim3 block(outputPlaneSize > 128 ? 128 : outputPlaneSize);

  int POOL_W = (poolSizeW <= 7 && poolSizeW >= 2) ? poolSizeW : -1;

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(input.type(),
    "fractional_max_pool2d_out_frame",
    [&] {
      fractional_max_pool2d_out_frame<scalar_t>
        <<<grid, block, 0, at::cuda::getCurrentCUDAStream()>>>(
          getTensorInfo<scalar_t, int>(input),
          getTensorInfo<scalar_t, int>(output),
          getTensorInfo<int64_t, int>(indices),
          getTensorInfo<scalar_t, int>(randomSamples),
          pool_size,
          POOL_W);
        }
      );
}

void fractional_max_pool2d_backward_out_cuda_template(
  Tensor& gradInput,
  const Tensor& gradOutput,
  const Tensor& input,
  IntList pool_size,
  IntList output_size,
  const Tensor& indices)
{
  int dimh = 1;
  int dimw = 2;

  int64_t ndims = input.ndimension();
  if (ndims == 4) {
    dimh++;
    dimw++;
  }

  /* sizes */
  int64_t inputH = input.size(dimh);
  int64_t inputW = input.size(dimw);

  int64_t outputH = output_size[0];
  int64_t outputW = output_size[1];
  int64_t poolSizeH = pool_size[0];
  int64_t poolSizeW = pool_size[1];

  AT_CHECK(outputH == gradOutput.size(dimh),
           "fractional_max_pool2d(): gradOutput height unexpected");
  AT_CHECK(outputW == gradOutput.size(dimw),
           "fractional_max_pool2d(): gradOutput width unexpected");

  /* resize */
  gradInput = at::zeros_like(input);

  /* backprop */
  // block is limited to 4 warps
  // grid handles overflow per each plane
  int outputPlaneSize = gradOutput.size(ndims - 2) *
    gradOutput.size(ndims - 1);
  dim3 grid((outputPlaneSize + 127) / 128,
            gradInput.size(ndims - 3),
            gradInput.size(ndims - 4));
  dim3 block(outputPlaneSize > 128 ? 128 : outputPlaneSize);

AT_DISPATCH_FLOATING_TYPES_AND_HALF(gradOutput.type(),
  "fractional_max_pool2d_backward_out_frame",
  [&] {
    fractional_max_pool2d_backward_out_frame<scalar_t>
      <<<grid, block, 0, at::cuda::getCurrentCUDAStream()>>>(
        getTensorInfo<scalar_t, int>(gradInput),
        getTensorInfo<scalar_t, int>(gradOutput),
        getTensorInfo<int64_t, int>(indices));
      }
    );
}

}// namespace

std::tuple<Tensor&, Tensor&> fractional_max_pool2d_out_cuda(
  at::Tensor& output,
  at::Tensor& indices,
  at::Tensor const& input,
  IntList pool_size,
  IntList output_size,
  at::Tensor const& randomSamples)
{
  fractional_max_pool2d_out_cuda_template(
    output,
    indices,
    input,
    pool_size,
    output_size,
    randomSamples);
  return std::tuple<Tensor&, Tensor&>(output, indices);
}

std::tuple<Tensor, Tensor> fractional_max_pool2d_cuda(
  at::Tensor const& input,
  IntList pool_size,
  IntList output_size,
  at::Tensor const& randomSamples)
{
  Tensor output = at::empty({0}, input.options());
  Tensor indices = at::empty({0}, input.options().dtype(kLong));
  fractional_max_pool2d_out_cuda_template(
    output,
    indices,
    input,
    pool_size,
    output_size,
    randomSamples);
  return std::tuple<Tensor, Tensor>(output, indices);
}

Tensor& fractional_max_pool2d_backward_out_cuda(
  at::Tensor& gradInput,
  at::Tensor const& gradOutput_,
  at::Tensor const& input,
  IntList pool_size,
  IntList output_size,
  at::Tensor const& indices)
{
  gradInput.resize_as_(input);
  fractional_max_pool2d_backward_out_cuda_template(
    gradInput,
    gradOutput_,
    input,
    pool_size,
    output_size,
    indices);
  return gradInput;
}

Tensor fractional_max_pool2d_backward_cuda(
  at::Tensor const& gradOutput_,
  at::Tensor const& input,
  IntList pool_size,
  IntList output_size,
  at::Tensor const& indices)
{
  Tensor gradInput = at::zeros_like(input);
  fractional_max_pool2d_backward_out_cuda_template(
    gradInput,
    gradOutput_,
    input,
    pool_size,
    output_size,
    indices);
  return gradInput;
}

}// at::native
}// at
