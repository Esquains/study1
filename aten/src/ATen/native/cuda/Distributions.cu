#include "hip/hip_runtime.h"
#include "ATen/Dispatch.h"
#include "ATen/NativeFunctions.h"
#include "ATen/cuda/CUDAApplyUtils.cuh"

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand/hiprand_kernel.h>
#include <utility>
#include <functional>
#include <nvfunctional>

#include "ATen/SharedDist.cu"
#include "ATen/native/Distributions.cpp"

#include <TH/THAtomic.h>

#include <THC/THCGeneral.h>
#include <THC/THCTensorRandom.h>
#include <THC/THCGenerator.h>
#include <THC/THCApply.cuh>
#include <THC/THCNumerics.cuh>

#include <cstdint>
#include <utility>

THCGenerator* THCRandom_getGenerator(THCState* state);

namespace {
std::pair<uint64_t, uint64_t> next_philox_seed(at::Generator* gen) {
  auto gen_ = THCRandom_getGenerator(at::globalContext().thc_state);
  uint64_t offset = THAtomicAddLong(&gen_->state.philox_seed_offset, 1);
  return std::make_pair(gen_->state.initial_seed, offset);
}

template <typename scalar_t>
void poisson_cuda_kernel(
    at::Tensor& ret,
    const at::Tensor& lambda,
    std::pair<uint64_t, uint64_t> seeds) {
  at::cuda::CUDA_tensor_apply2<scalar_t, float>(
      ret,
      lambda,
      [seeds] __device__(
          scalar_t & ret_val, const float& lambda, bool early_exit) {
        hiprandStatePhilox4_32_10_t state;
        hiprand_init(
            seeds.first,
            blockIdx.x * blockDim.x + threadIdx.x,
            seeds.second,
            &state);
        ret_val = scalar_cast<scalar_t>(hiprand_poisson(&state, lambda));
      });
}

  template <typename scalar>
  struct GammaOpCUDA {
    static void apply(Tensor& ret, const Tensor& alpha, std::pair<uint64_t, uint64_t> seeds) {
      at::cuda::CUDA_tensor_apply2<scalar, float>(ret, alpha,
        [seeds] __device__ (scalar& ret_val, const float& alpha, bool early_exit) {
          hiprandStatePhilox4_32_10_t state;
          hiprand_init(seeds.first, blockIdx.x * blockDim.x + threadIdx.x, seeds.second, &state);
          baseSampler<float> standard_uniform([&state] __device__ () {
            return hiprand_uniform(&state);
          });
          baseSampler<float> standard_normal([&state] __device__ () {
            return hiprand_normal(&state);
          });
          auto sample = scalar_cast<scalar>(sample_gamma<float>(alpha, standard_uniform, standard_normal));
          ret_val = ::max(THCNumerics<scalar>::min(), (scalar) sample);
        }
      );
    }
  };

  template <typename scalar>
  struct PoissonOpCUDA {
    static void apply(Tensor& ret, const Tensor& lambda, std::pair<uint64_t, uint64_t> seeds) {
      at::cuda::CUDA_tensor_apply2<scalar, float>(ret, lambda,
        [seeds] __device__ (scalar& ret_val, const float& lambda, bool early_exit) {
          hiprandStatePhilox4_32_10_t state;
          hiprand_init(seeds.first, blockIdx.x * blockDim.x + threadIdx.x, seeds.second, &state);
          ret_val = scalar_cast<scalar>(hiprand_poisson(&state, lambda));
        }
      );
    }
  };

} // namespace

namespace at { namespace native {
Tensor _s_poisson_cuda(const Tensor& lambda, Generator* gen) {
  Tensor ret = lambda.type().tensor(lambda.sizes());
  auto lambda_ = lambda.toType(ScalarType::Float);
  AT_DISPATCH_FLOATING_TYPES(ret.type(), "poisson", [&] {
     poisson_cuda_kernel<scalar_t>(ret, lambda_, next_philox_seed(gen));
   });
  return ret;
}

Tensor _s_gamma_cuda(const Tensor& alpha, Generator* gen) {
  Tensor ret = alpha.type().tensor(alpha.sizes());
  auto alpha_ = alpha.toType(ScalarType::Float);
  dispatch_floating_types<void, dist::GammaOpCUDA>(ret.type(), "gamma", ret, alpha_, dist::next_philox_seed(gen));
  return ret;
}

}} // namespace at::native
