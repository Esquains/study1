#include "ATen/Context.h"
#include "ATen/cuda/HIPContext.h"
#include "ATen/Dispatch.h"
#include "ATen/NativeFunctions.h"
#include "ATen/cuda/PinnedMemoryAllocator.h"
#include "ATen/cuda/CUDAApplyUtils.cuh"

#include "ATen/native/LinearAlgebraUtils.h"
#include "ATen/native/Gesv.h"

#include "THC.h" // for USE_MAGMA

#ifdef USE_MAGMA
#include <magma.h>
#include <magma_types.h>
#endif

namespace at {
namespace native {

#ifdef USE_MAGMA

template<class scalar_t>
void magmaGesv(
    int64_t n, int64_t nrhs, scalar_t* A_data, int64_t lda,
    int* ipiv, scalar_t* B_data, int64_t ldb, int* info) {
  AT_ERROR("magma: gesv only takes float or double Tensors");
}

template<>
void magmaGesv<float>(
    int64_t n, int64_t nrhs, float* A_data, int64_t lda,
    int* ipiv, float* B_data, int64_t ldb, int* info) {
  magma_sgesv_gpu(n, nrhs, A_data, lda, ipiv, B_data, ldb, info);
}

template<>
void magmaGesv<double>(
    int64_t n, int64_t nrhs, double* A_data, int64_t lda,
    int* ipiv, double* B_data, int64_t ldb, int* info) {
  magma_dgesv_gpu(n, nrhs, A_data, lda, ipiv, B_data, ldb, info);
}

template<class scalar_t>
void magmaGesvBatched(
    magma_int_t n, magma_int_t nrhs, scalar_t** dA_array, magma_int_t ldda,
    magma_int_t** dipiv_array, scalar_t** dB_array, magma_int_t lddb,
    magma_int_t* dinfo_array, magma_int_t batch_count, magma_queue_t queue) {
  AT_ERROR("gesv only takes float or double Tensors");
}

template<>
void magmaGesvBatched<float>(
    magma_int_t n, magma_int_t nrhs, float** dA_array, magma_int_t ldda,
    magma_int_t** dipiv_array, float** dB_array, magma_int_t lddb,
    magma_int_t* dinfo_array, magma_int_t batch_count, magma_queue_t queue) {
  magma_sgesv_batched(
      n, nrhs, dA_array, ldda, dipiv_array,
      dB_array, lddb, dinfo_array, batch_count, queue);
}

template<>
void magmaGesvBatched<double>(
    magma_int_t n, magma_int_t nrhs, double** dA_array, magma_int_t ldda,
    magma_int_t** dipiv_array, double** dB_array, magma_int_t lddb,
    magma_int_t* dinfo_array, magma_int_t batch_count, magma_queue_t queue) {
  magma_dgesv_batched(
      n, nrhs, dA_array, ldda, dipiv_array,
      dB_array, lddb, dinfo_array, batch_count, queue);
}

static magma_queue_t createMagmaQueue(const Tensor& tensor) {
  auto& context = tensor.type().get_context();
  magma_queue_t magma_queue;
  magma_queue_create_from_cuda(
      tensor.get_device(),
      at::cuda::getCurrentCUDAStream(),
      THCState_getCurrentBlasHandle(context.getTHCState()),
      THCState_getCurrentSparseHandle(context.getTHCState()),
      &magma_queue);
  return magma_queue;
}

static inline magma_int_t magma_int_cast(int64_t value, const char* varname) {
  auto result = static_cast<magma_int_t>(value);
  if (static_cast<int64_t>(result) != value) {
    AT_ERROR("magma: The value of %s (%lld) is too large to fit into a magma_int_t (%llu bytes)",
             varname, (long long)value, sizeof(magma_int_t));
  }
  return result;
}
#endif

// Creates an array of size elements of type T, backed by pinned memory
// wrapped in a Storage
template<class T>
static inline std::unique_ptr<Storage> pin_memory(int64_t size, Tensor dummy) {
  int64_t adjusted_size = size * sizeof(T);
  auto* allocator = cuda::getPinnedMemoryAllocator();
  auto& backend = dummy.type().toBackend(kCPU).toScalarType(kByte);
  return backend.storageWithAllocator(adjusted_size, allocator);
}

static inline bool isTransposeContiguous(Tensor& self) {
  return self.dim() == 2 &&
         self.stride(0) == 1 &&
         self.stride(1) == self.size(0);
}

#define ALLOCATE_ARRAY(name, type, size, dummy_tensor) \
  auto storage_##name = pin_memory<type>(size, dummy_tensor); \
  name = reinterpret_cast<type*>(storage_##name->data());

template <typename scalar_t>
static void applyGesv(Tensor& b, Tensor& A, std::vector<int64_t> infos) {
#ifndef USE_MAGMA
AT_ERROR("gesv: MAGMA library not found in "
    "compilation. Please rebuild with MAGMA.");
#else
  auto A_data = A.data<scalar_t>();
  auto b_data = b.data<scalar_t>();
  auto A_mat_stride = matrixStride(A);
  auto b_mat_stride = matrixStride(b);

  magma_int_t batch_size = magma_int_cast(batchCount(A), "batchCount");
  magma_int_t n = magma_int_cast(A.size(-2), "A.size(-2)");
  magma_int_t nrhs = magma_int_cast(b.size(-1), "b.size(-1)");

  magma_int_t* info_array;
  magma_int_t* ipiv_data;
  magma_int_t** ipiv_array;
  scalar_t** A_array;
  scalar_t** b_array;

  ALLOCATE_ARRAY(info_array, magma_int_t, batch_size, b);
  ALLOCATE_ARRAY(ipiv_data, magma_int_t, batch_size * n, b);
  ALLOCATE_ARRAY(ipiv_array, magma_int_t*, batch_size, b);
  ALLOCATE_ARRAY(A_array, scalar_t*, batch_size, b);
  ALLOCATE_ARRAY(b_array, scalar_t*, batch_size, b);

  // Set up the created arrays
  for (int64_t i = 0; i < batch_size; i++) {
    A_array[i] = &A_data[i * A_mat_stride];
    b_array[i] = &b_data[i * b_mat_stride];
    ipiv_array[i] = &ipiv_data[i * n];
  }

  magmaGesvBatched<scalar_t>(
      n, nrhs, A_array, n, ipiv_array, b_array, n,
      info_array, batch_size, createMagmaQueue(b));

  for (int64_t i = 0; i < batch_size; i++) {
    infos[i] = info_array[i];
  }
#endif
}

std::tuple<Tensor,Tensor> _gesv_helper_cuda(const Tensor& self, const Tensor& A) {
  std::vector<int64_t> infos(batchCount(A), 0);
  auto A_working_copy = cloneBatchedColumnMajor(A);
  auto b_working_copy = cloneBatchedColumnMajor(self);
  AT_DISPATCH_FLOATING_TYPES(self.type(), "gesv", [&]{
    applyGesv<scalar_t>(b_working_copy, A_working_copy, infos);
  });
  checkErrors(infos);
  return std::tuple<Tensor,Tensor>(b_working_copy, A_working_copy);
}

std::tuple<Tensor&,Tensor&> _gesv_single_out_cuda(Tensor& sol, Tensor& lu,
    const Tensor& self, const Tensor& A) {
#ifndef USE_MAGMA
AT_ERROR("gesv: MAGMA library not found in "
    "compilation. Please rebuild with MAGMA.");
#else
  int64_t bx = self.size(0);
  int64_t by = (self.dim() == 1) ? 1 : self.size(1);
  int64_t ax = A.size(0);
  int64_t ay = A.size(1);
  int info;
  int* ipiv;

  // init to column major format
  // Note: `sol` and `lu` need to be contiguous since we pass
  // sol.data() and lu.data() to Magma
  if (&self == &sol) {
    // eg. torch.gesv(b, A, out=(b, A))
    if (isTransposeContiguous(sol)) {
      sol.t_();
    } else {
      auto Bc = self.view({bx, by}).t().clone();
      sol.resize_({by, bx}).copy_(Bc);
    }
  } else if (sol.numel() == self.numel() &&
             isTransposeContiguous(sol)) {
    // allow reuse
    sol.t_().copy_(self.view({bx, by}).t());
  } else {
    sol.resize_({by, bx}).copy_(self.view({bx, by}).t());
  }

  if (&A == &lu) {
    if (isTransposeContiguous(lu)) {
      lu.t_();
    } else {
      auto Ac = A.t().clone();
      lu.copy_(Ac);
    }
  } else if (lu.numel() == A.numel() &&
             isTransposeContiguous(lu)) {
    lu.t_().copy_(A.t());
  } else {
    lu.resize_({ay, ax}).copy_(A.t());
  }

  AT_DISPATCH_FLOATING_TYPES(self.type(), "gesv", [&]{
      auto A_ptr = lu.data<scalar_t>();
      auto b_ptr = sol.data<scalar_t>();
      ALLOCATE_ARRAY(ipiv, int, bx, sol);
      magmaGesv<scalar_t>(bx, by, A_ptr, bx, ipiv, b_ptr, bx, &info);
  });

  sol.t_();
  lu.t_();

  checkErrors({info});
  return std::tuple<Tensor&,Tensor&>(sol, lu);
#endif
}
}}  // namespace at::native

#undef ALLOCATE_ARRAY
