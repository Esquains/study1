#include "hip/hip_runtime.h"
#include "ATen/ATen.h"
 #include "ATen/NativeFunctions.h"
 #include "ATen/TensorUtils.h"

  #include "ATen/cuda/HIPContext.h"
 #include "c10/util/Exception.h"

  namespace at {
 namespace native {

  template <typename T>
 __host__ __device__ __forceinline__ T ceilDiv(T a, T b) {
   return (a + b - 1) / b;
 }

  template <typename T>
 __global__ void MaxUnpooling2d_forward_kernel(
     const int64_t numInputElements,
     const T* input,
     const int64_t* indices,
     const int64_t numBatch,
     const int64_t numChannels,
     const int64_t inputHeight,
     const int64_t inputWidth,
     const int64_t outputHeight,
     const int64_t outputWidth,
     T* output) {
   for (int linearIndex = blockIdx.x * blockDim.x + threadIdx.x;
        linearIndex < numInputElements;
        linearIndex += blockDim.x * gridDim.x) {
     int c = (linearIndex / inputWidth / inputHeight) % numChannels;
     int n = linearIndex / inputWidth / inputHeight / numChannels;
     output += (n * numChannels + c) * outputHeight * outputWidth;
     int maxind = indices[linearIndex];
     output[maxind] = input[linearIndex];
   }
 }
 template <typename T>
 __global__ void MaxUnpooling3d_forward_kernel(
     const T* input,
     const int64_t* indices,
     const int64_t batchSize,
     const int64_t inputSlices,
     const int64_t inputTime,
     const int64_t inputHeight,
     const int64_t inputWidth,
     const int64_t oT,
     const int64_t oW,
     const int64_t oH,
     const int64_t dT,
     const int64_t dW,
     const int64_t dH,
     const int64_t pT,
     const int64_t pW,
     const int64_t pH,
     const int64_t offsetZ,
     T* output) {
   int64_t iColumn = blockIdx.x * blockDim.x + threadIdx.x;
   int64_t iRow = blockIdx.y * blockDim.y + threadIdx.y;
   int64_t iFrame = (blockIdx.z + offsetZ) % inputTime; // intput frame/time
   int64_t slice = (blockIdx.z + offsetZ) / inputTime; // intput slice/feature
   if (iRow < inputHeight && iColumn < inputWidth) {
     int64_t newIndex = slice * (inputTime * inputHeight * inputWidth) +
         iFrame * (inputHeight * inputWidth) + iRow * inputWidth + iColumn;
     T val = input[newIndex];
     int64_t index = indices[newIndex];
     output[slice * oT * oH * oW + index] = val;
   }
 }

  template <typename T>
 __global__ void MaxUnpooling2d_backward_kernel(
     const int64_t numInputElements,
     const T* input,
     const int64_t* indices,
     const int64_t numBatch,
     const int64_t numChannels,
     const int64_t inputHeight,
     const int64_t inputWidth,
     const int64_t outputHeight,
     const int64_t outputWidth,
     T* output) {
   for (int linearIndex = blockIdx.x * blockDim.x + threadIdx.x;
        linearIndex < numInputElements;
        linearIndex += blockDim.x * gridDim.x) {
     int c = (linearIndex / inputWidth / inputHeight) % numChannels;
     int n = linearIndex / inputWidth / inputHeight / numChannels;
     input += (n * numChannels + c) * outputHeight * outputWidth;
     int maxind = indices[linearIndex];
     output[linearIndex] = input[maxind];
   }
 }

  template <typename T>
 __global__ void MaxUnpooling3d_backward_kernel(
   T* gradOutputData,
   int64_t oT, int64_t oH, int64_t oW,
   int64_t* indices,
   T* gradInput,
   int64_t dT, int64_t dH, int64_t dW,
   int64_t padT, int64_t padH, int64_t padW, int offsetZ,
   int64_t grad_input_size_0,
   int64_t grad_input_size_1,
   int64_t grad_input_size_2,
   int64_t grad_input_size_3,
   int64_t grad_input_size_4,
   int64_t indices_size_0,
   int64_t indices_size_1,
   int64_t indices_size_2,
   int64_t indices_size_3,
   int64_t indices_size_4
   )
 {
   int iColumn = blockIdx.x * blockDim.x + threadIdx.x;
   int iRow    = blockIdx.y * blockDim.y + threadIdx.y;
   int iFrame  = (blockIdx.z + offsetZ) % grad_input_size_1; // output frame/time
   int slice   = (blockIdx.z + offsetZ) / grad_input_size_1; // output slice/feature

    if (iRow < grad_input_size_3 && iColumn < grad_input_size_4)
   {
     // int64_t index = indices[slice][iFrame][iRow][iColumn];
     int64_t indices_index = slice * (indices_size_2 * indices_size_3 * indices_size_4)
       + iFrame * (indices_size_3 * indices_size_4) + iRow * (indices_size_4) + iColumn;
     int64_t index = indices[indices_index];

      T grad_val = gradOutputData[slice*oT*oH*oW + index];

      int64_t grad_input_index = slice * (grad_input_size_2 * grad_input_size_3 * grad_input_size_4)
       + iFrame * (grad_input_size_3 * grad_input_size_4) + iRow * (grad_input_size_4) + iColumn;
     gradInput[grad_input_index] = grad_val;
     // gradInput[slice][iFrame][iRow][iColumn] = grad_val;
   }
 }

  at::Tensor& MaxUnpooling2d_forward_out_cuda(
     Tensor& output,
     const Tensor& self,
     const Tensor& indices,
     IntList output_size) {
   TensorArg output_arg{ output, "output", 1 },
           self_arg{ self, "self", 2 },
           indices_arg{ indices, "indices", 3 };
   checkAllSameGPU("MaxUnpooling2d_forward_cuda_out", {output_arg, self_arg, indices_arg});

    AT_CHECK(self.sizes() == indices.sizes(), "input shape must match indices shape");
   for (int64_t i = 0; i < self.ndimension(); i++) {
     AT_CHECK(
         self.size(i) > 0,
         "input must be nonempty, but input has sizes: ",
         self.sizes());
   }
   AT_CHECK(
       self.ndimension() == 4,
       "Input to MaxUnpooling2d should be a NCHW Tensor",
       self.sizes());
   AT_CHECK(
       output_size.size() == 2,
       "There should be exactly two elements (height, width) in output_size");

    auto numBatch = self.size(0);
   auto numChannels = self.size(1);
   auto inputHeight = self.size(2);
   auto inputWidth = self.size(3);
   auto outputHeight = output_size[0];
   auto outputWidth = output_size[1];

    dim3 block(512);
   dim3 grid((output.numel() + 512 - 1) / 512);

    AT_DISPATCH_ALL_TYPES_AND_HALF(
       self.type(), "MaxUnpooling2d_forward_kernel", ([&] {
         MaxUnpooling2d_forward_kernel<<<
             grid,
             block,cle
             0,
             at::cuda::getCurrentCUDAStream()>>>(
             self.numel(),
             self.data<scalar_t>(),
             indices.data<int64_t>(),
             numBatch,
             numChannels,
             inputHeight,
             inputWidth,
             outputHeight,
             outputWidth,
             output.data<scalar_t>());
       }));
   AT_CHECK(
       hipGetLastError() == hipSuccess,
       "RoiPooling2d_forward_kernel failed with error code ",
       hipGetLastError());
   return output;
 }

 at::Tensor& MaxUnpooling2d_backward_out_cuda(
   Tensor& grad_input,
   const Tensor& grad_output,
   const Tensor& self,
   const Tensor& indices,
   IntList output_size
 )
 {
   TensorArg grad_input_arg{ grad_input, "grad_input", 1 },
           grad_output_arg{ grad_output, "grad_output", 2 },
           self_arg{ self, "self", 3 },
           indices_arg{indices, "indices", 4};
   checkAllSameGPU("MaxUnpooling2d_backward_out_cuda", {grad_input_arg, grad_output_arg, self_arg, indices_arg});
   for (int64_t i = 0; i < self.ndimension(); i++) {
     AT_CHECK(
         self.size(i) > 0,
         "input must be nonempty, but input has sizes: ",
         self.sizes());
   }
   AT_CHECK(self.ndimension() == 4, "Input to MaxUnpooling2d should be a NCHW 4d Tensor, instead got: ",
     self
   );
   AT_CHECK(self.sizes() == indices.sizes(), "Input should have same shape as indices");

   AT_CHECK(output_size.size() == 2, "output_size must have two elements");
   int64_t oheight = output_size[0];
   int64_t owidth = output_size[1];

   int dimw = 3;
   int dimh = 2;

   int64_t nInputCols = self.size(dimw);
   int64_t nInputRows = self.size(dimh);
   int64_t nInputPlane = self.size(1);
   int64_t batchSize = self.size(0);

   if (oheight != grad_output.size(dimh) ||
        owidth != grad_output.size(dimw)) {
      AT_ERROR(
          "Inconsistent gradOutput size",
          oheight,
          ", output width= ",
          owidth,
          ", gradOutput: ",
          grad_output.size(dimh),
          "x",
          grad_output.size(dimw));
    }
    auto input_contiguous = self.contiguous();
    auto indices_contiguous = indices.contiguous();
    auto grad_output_contiguous = grad_output.contiguous();
    grad_input.resize_as_(input_contiguous);
    grad_input.zero_();

   int count = input_contiguous.numel();


   dim3 block(512);
   dim3 grid((count + 512 - 1) / 512);
   AT_DISPATCH_ALL_TYPES_AND_HALF(
       input_contiguous.type(), "MaxUnpooling2d_backward_kernel", ([&] {
         MaxUnpooling2d_backward_kernel<<<
             grid,
             block,
             0,
             at::cuda::getCurrentCUDAStream()>>>(
             count,
             grad_output_contiguous.data<scalar_t>(),
             indices_contiguous.data<int64_t>(),
             batchSize,
             nInputPlane,
             nInputRows,
             nInputCols,
             oheight,
             owidth,
             grad_input.data<scalar_t>());
       }));
   AT_CHECK(
       hipGetLastError() == hipSuccess,
       "MaxUnpooling2d_backward_kernel failed with error code ",
       hipGetLastError());
   return grad_input;
 }
 at::Tensor MaxUnpooling2d_backward_cuda(
   const Tensor& grad_output,
   const Tensor& self,
   const Tensor& indices,
   IntList output_size
 )
 {
   AT_CHECK(
     self.ndimension() == 4,
     "Input to MaxUnpooling2d should be a NCHW Tensor",
     self.sizes());
   AT_CHECK(
       output_size.size() == 2,
       "There should be exactly two elements (height, width) in output_size");
   for (int64_t i = 0; i < self.ndimension(); i++) {
     AT_CHECK(
         self.size(i) > 0,
         "input must be nonempty, but input has sizes: ",
         self.sizes());
   }
   auto grad_input = at::zeros_like(self);
   MaxUnpooling2d_backward_out_cuda(
     grad_input, grad_output, self, indices, output_size);
   return grad_input;
 }

 at::Tensor MaxUnpooling2d_forward_cuda(
     const Tensor& self,
     const Tensor& indices,
     IntList output_size) {
   AT_CHECK(
       self.ndimension() == 4,
       "Input to MaxUnpooling2d should be a NCHW Tensor",
       self.sizes());
   AT_CHECK(
       output_size.size() == 2,
       "There should be exactly two elements (height, width) in output_size");

   auto output = at::zeros(
       {self.size(0), self.size(1), output_size[0], output_size[1]},
       self.options());
   MaxUnpooling2d_forward_out_cuda(output, self, indices, output_size);
   return output;
 }

 void MaxUnpooling3d_shape_check(
     const Tensor& input,
     const Tensor& gradOutput,
     const Tensor& indices,
     IntList output_size,
     IntList stride,
     IntList padding,
     bool check_grad) {
   // is_empty check
   for (int64_t i = 0; i < input.ndimension(); i++) {
     AT_CHECK(
         input.size(i) > 0,
         "input must be nonempty, but input has sizes: ",
         input.sizes());
   }
   AT_CHECK(input.sizes() == indices.sizes(), "Input should have same shape as indices");
   AT_CHECK(
       stride[0] > 0 && stride[1] > 0 && stride[2] > 0,
       "stride should be never greater than zero, but got stride: ",
       stride);

   int dimw = 4;
   int dimh = 3;
   int dimt = 2;
   int dimn = 1;

   int nslices = input.size(dimn);
   if (check_grad) {
     if (output_size[0] != gradOutput.size(dimt) ||
         output_size[1] != gradOutput.size(dimh) ||
         output_size[2] != gradOutput.size(dimw)) {
       AT_ERROR(
           "Inconsistent gradOutput size. output_size[1]= %d, output_size[1] = %d, output_size[2] = %d, gradOutput: %dx%dx%d",
           output_size[0],
           output_size[1],
           output_size[2],
           gradOutput[0],
           gradOutput[1],
           gradOutput[2]);
     }
     AT_CHECK(gradOutput.ndimension() == 5 && gradOutput.size(dimn) == nslices);
   }
 }
 at::Tensor& MaxUnpooling3d_forward_out_cuda(
     Tensor& output,
     const Tensor& self,
     const Tensor& indices,
     IntList output_size,
     IntList stride,
     IntList padding) {
   AT_CHECK(
       self.ndimension() == 5,
       "Input to MaxUnpooling2d should be a NCDHW Tensor",
       self.sizes());
   AT_CHECK(
       output_size.size() == 3,
       "There should be exactly three elements (depth, height, width) in output_size");

   auto batchSize = self.size(0);
   auto inputSlices = self.size(1);
   auto inputTime = self.size(2);
   auto inputHeight = self.size(3);
   auto inputWidth = self.size(4);
   auto outputTime = output_size[0];
   auto outputHeight = output_size[1];
   auto outputWidth = output_size[2];

   auto dT = stride[0];
   auto dH = stride[1];
   auto dW = stride[2];

   auto padT = padding[0];
   auto padH = padding[1];
   auto padW = padding[2];

   MaxUnpooling3d_shape_check(
       self, at::empty({}), indices, output_size, stride, padding, false);
   TensorArg output_arg{ output, "output", 1 },
           self_arg{ self, "self", 2 },
           indices_arg{ indices, "indices", 3 };
   checkAllSameGPU("MaxUnpooling3d_forward_out_cuda", {output_arg, self_arg, indices_arg});

   AT_CHECK(output.is_contiguous(), "output must be contiguous");

   int totalZ = inputTime * inputSlices * batchSize;
   int offsetZ = 0;
   dim3 block(32, 8);

   while (totalZ > 0) {
     dim3 grid(
         ceilDiv(inputWidth, static_cast<int64_t>(block.x)),
         ceilDiv(inputHeight, static_cast<int64_t>(block.y)),
         totalZ > 65535 ? 65535 : totalZ);
     AT_DISPATCH_ALL_TYPES_AND_HALF(
         self.type(), "MaxUnpooling3d_forward_kernel", ([&] {
           MaxUnpooling3d_forward_kernel<<<
               grid,
               block,
               0,
               at::cuda::getCurrentCUDAStream()>>>(
               self.contiguous().data<scalar_t>(),
               indices.contiguous().data<int64_t>(),
               batchSize,
               inputSlices,
               inputTime,
               inputHeight,
               inputWidth,
               outputTime,
               outputHeight,
               outputWidth,
               dT,
               dH,
               dW,
               padT,
               padH,
               padW,
               offsetZ,
               output.data<scalar_t>());
         }));
     AT_CHECK(
         hipGetLastError() == hipSuccess,
         "RoiPooling3d_forward_kernel failed with error code ",
         hipGetLastError());
     totalZ -= 65535;
     offsetZ += 65535;
   }
   return output;
 }
 at::Tensor MaxUnpooling3d_forward_cuda(
     const Tensor& self,
     const Tensor& indices,
     IntList output_size,
     IntList stride,
     IntList padding) {
       AT_CHECK(
           self.ndimension() == 5,
           "Input to MaxUnpooling2d should be a NCDHW Tensor",
           self.sizes());
       AT_CHECK(
           output_size.size() == 3,
           "There should be exactly three elements (depth, height, width) in output_size");

   auto output = at::zeros(
       {self.size(1),
        self.size(2),
        output_size[0],
        output_size[1],
        output_size[2]},
       self.options());
   MaxUnpooling3d_forward_out_cuda(output, self, indices, output_size, stride, padding);
   return output;
 }

 at::Tensor& MaxUnpooling3d_backward_out_cuda(
     Tensor& grad_input,
     const Tensor& grad_output,
     const Tensor& self,
     const Tensor& indices,
     IntList output_size,
     IntList stride,
     IntList padding) {

   int batchSize = 0;
   int inputSlices = 0;
   int inputTime = 0;
   int64_t inputHeight = 0;
   int64_t inputWidth = 0;
   AT_CHECK(output_size.size() == 3, "output_size must have three elements");
   AT_CHECK(stride.size() == 3, "stride must have three elements");
   AT_CHECK(padding.size() == 3, "padding must have three elements");

   AT_CHECK(self.ndimension() == 5, "self must be a 5d NCDHW tensor");

   MaxUnpooling3d_shape_check(
       self, grad_output, indices, output_size, stride, padding, true);
   TensorArg self_arg{self, "self", 1}, indices_arg{indices, "indices", 2},
       grad_output_arg{grad_output, "grad_output", 3},
       grad_input_arg{grad_input, "grad_input", 4};
   checkAllSameGPU(
       "MaxUnpooling3d_backward_out_cuda",
       {self_arg, indices_arg, grad_output_arg, grad_input_arg});

   batchSize = self.size(0);
   inputSlices = self.size(1);
   inputTime = self.size(2);
   inputHeight = self.size(3);
   inputWidth = self.size(4);

   auto input_contiguous = self.contiguous();
   grad_input.resize_as_(input_contiguous);
   grad_input.zero_();
   auto indices_contiguous = indices.contiguous();
   auto grad_output_contiguous = grad_output.contiguous();

   int totalZ = inputTime * inputSlices * batchSize;
   int offsetZ = 0;

   int64_t output_size_0 = output_size[0];
   int64_t output_size_1 = output_size[1];
   int64_t output_size_2 = output_size[2];

   int64_t stride_0 = stride[0];
   int64_t stride_1 = stride[1];
   int64_t stride_2 = stride[2];

   int64_t padding_0 = padding[0];
   int64_t padding_1 = padding[1];
   int64_t padding_2 = padding[2];

   dim3 block(32, 8);
   while (totalZ > 0) {
     dim3 grid(
         ceilDiv(inputWidth, static_cast<int64_t>(block.x)),
         ceilDiv(inputHeight, static_cast<int64_t>(block.y)),
         totalZ > 65535 ? 65535 : totalZ);
     std::cout << "grad_input.sizes(): " << grad_input.sizes() << "\n";
     AT_DISPATCH_ALL_TYPES_AND_HALF(
         input_contiguous.type(), "MaxUnpooling3d_backward_kernel", ([&] {
           MaxUnpooling3d_backward_kernel<<<
               grid,
               block,
               0,
               at::cuda::getCurrentCUDAStream()>>>(
               grad_output.data<scalar_t>(),
               output_size_0,
               output_size_1,
               output_size_2,
               indices_contiguous.data<int64_t>(),
               grad_input.data<scalar_t>(),
               stride_0,
               stride_1,
               stride_2,
               padding_0,
               padding_1,
               padding_2,
               offsetZ,
               grad_input.size(0),
               grad_input.size(1),
               grad_input.size(2),
               grad_input.size(3),
               grad_input.size(4),
               indices.size(0),
               indices.size(1),
               indices.size(2),
               indices.size(3),
               indices.size(4)
             );
         }));
     AT_CHECK(
         hipGetLastError() == hipSuccess,
         "RoiPooling3d_forward_kernel failed with error code ",
         hipGetLastError());
     totalZ -= 65535;
     offsetZ += 65535;
   }
   return grad_input;
 }

 at::Tensor MaxUnpooling3d_backward_cuda(
     const Tensor& grad_output,
     const Tensor& self,
     const Tensor& indices,
     IntList output_size,
     IntList stride,
     IntList padding) {
   auto grad_input = at::zeros_like(self);
   MaxUnpooling3d_backward_out_cuda(
       grad_input, grad_output, self, indices, output_size, stride, padding);
   return grad_input;
 }

 } // namespace native
 } // namespace at
