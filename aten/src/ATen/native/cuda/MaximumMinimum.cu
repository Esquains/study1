#include "hip/hip_runtime.h"
#include <ATen/AccumulateType.h>
#include <ATen/Dispatch.h>
#include <ATen/native/BinaryOps.h>
#include <ATen/native/DispatchStub.h>
#include <ATen/native/TensorIterator.h>
#include <ATen/native/cuda/Loops.cuh>

// NOTE: CUDA on Windows requires that the enclosing function
// of a __device__ lambda not have internal linkage.

namespace at { namespace native {

void maximum_kernel_cuda(TensorIterator& iter) {
  if (isIntegralType(iter.dtype(), /*includeBool=*/ true)) {
    AT_DISPATCH_INTEGRAL_TYPES_AND(at::ScalarType::Bool, iter.dtype(), "maximum_cuda", [&] {
      gpu_kernel(iter, [] GPU_LAMBDA (scalar_t a, scalar_t b) -> scalar_t {
        return a >= b ? a : b;
      });
    });
  } else if (isFloatingType(iter.dtype())) {
    AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, iter.input_dtype(), "maximum_cuda", [&]() {
      gpu_kernel(iter, [] GPU_LAMBDA (scalar_t a, scalar_t b) -> scalar_t {
        // isnan(half) breaks the Windows build. We explicitly cast half to float.
        using acc_type = typename AccumulateType<scalar_t, /*is_cuda=*/true>::type;
        if (::isnan(static_cast<acc_type>(a))) {
          return a;
        }
        if (::isnan(static_cast<acc_type>(b))) {
          return b;
        }
        return a >= b ? a : b;
      });
    });
  } else {
    AT_DISPATCH_COMPLEX_TYPES(iter.dtype(), "maximum_cuda", [&] {
      gpu_kernel(iter, [] GPU_LAMBDA (scalar_t a, scalar_t b) -> scalar_t {
        if (::isnan(a.real()) || ::isnan(a.imag())) {
          return a;
        }
        if (::isnan(b.real()) || ::isnan(b.imag())) {
          return b;
        }
        if (b.real() >= a.real() && b.imag() >= a.imag()) {
          return b;
        }
        return a;
      });
    });
  }
}

void minimum_kernel_cuda(TensorIterator& iter) {
  if (isIntegralType(iter.dtype(), /*includeBool=*/ true)) {
    AT_DISPATCH_INTEGRAL_TYPES_AND(at::ScalarType::Bool, iter.dtype(), "minimum_cuda", [&] {
      gpu_kernel(iter, [] GPU_LAMBDA (scalar_t a, scalar_t b) -> scalar_t {
        return a <= b ? a : b;
      });
    });
  } else if (isFloatingType(iter.dtype())) {
    AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, iter.input_dtype(), "minimum_cuda", [&]() {
      gpu_kernel(iter, [] GPU_LAMBDA (scalar_t a, scalar_t b) -> scalar_t {
        // isnan(half) breaks the Windows build. We explicitly cast half to float.
        using acc_type = typename AccumulateType<scalar_t, /*is_cuda=*/true>::type;
        if (::isnan(static_cast<acc_type>(a))) {
          return a;
        }
        if (::isnan(static_cast<acc_type>(b))) {
          return b;
        }
        return a <= b ? a : b;
      });
    });
  } else {
    AT_DISPATCH_COMPLEX_TYPES(iter.dtype(), "minimum_cuda", [&] {
      gpu_kernel(iter, [] GPU_LAMBDA (scalar_t a, scalar_t b) -> scalar_t {
        if (::isnan(a.real()) || ::isnan(a.imag())) {
          return a;
        }
        if (::isnan(b.real()) || ::isnan(b.imag())) {
          return b;
        }
        if (b.real() <= a.real() && b.imag() <= a.imag()) {
          return b;
        }
        return a;
      });
    });
  }
}

REGISTER_DISPATCH(maximum_stub, &maximum_kernel_cuda);
REGISTER_DISPATCH(minimum_stub, &minimum_kernel_cuda);

}} // namespace at::native
