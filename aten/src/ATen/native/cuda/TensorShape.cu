#include "hip/hip_runtime.h"
#define TORCH_ASSERT_ONLY_METHOD_OPERATORS
#include <ATen/Dispatch.h>
#include <ATen/core/Tensor.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/native/Resize.h>

#ifndef AT_PER_OPERATOR_HEADERS
#include <ATen/Functions.h>
#include <ATen/NativeFunctions.h>
#else
#include <ATen/ops/empty.h>
#include <ATen/ops/split_with_sizes_copy_native.h>
#endif

namespace at::native {

namespace detail {

// NOTE [CUDA fast path for split_with_sizes_copy.out]
// split_with_sizes_copy.out for contiguous operands has the following
// properties:
// - Each src split consists of multiple chunks that are separated by a fixed
// stride. The number of chunks and the strides are the same across all src
// splits.
// - Each dst split is the concatenation of the chunks in its corresponding src
// splits.
// - The sizes of chunks vary across splits.
// - A (src, dst) chunk pair is not guaranteed to have the
// same alignment.
//
// The following strategies are employed to optimize for this workload:
// - The entire workload is fused into a single kernel to maximize I/O
// throughput and minimize wave quantization.
// - To account for both small and large chunk sizes, a "jagged grid" is used.
// Each chunk is processed by one or more blocks depending on its size.
// - Within each chunk, the region in which writes can be vectorized is
// identified. Within this region, writes are always vectorized and reads are
// oppurtunistically vectorized.
static constexpr int64_t BLOCK_SIZE = 128;
static constexpr int64_t BYTES_PER_THREAD = 16;
static constexpr int64_t TILE_SIZE = BYTES_PER_THREAD * BLOCK_SIZE;

static __host__ __device__ inline int64_t div_up(int64_t a, int64_t b) {
  return (a + b - 1) / b;
}

static __host__ __device__ inline int64_t minInt64(int64_t a, int64_t b) {
  return a < b ? a : b;
}

static __host__ __device__ inline int64_t maxInt64(int64_t a, int64_t b) {
  return a < b ? b : a;
}

template <typename T>
__device__ inline void stream_load128(uint4& val, const T* addr) {
  uint64_t low, high;
#if defined(USE_ROCM) || (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ < 800))
  low = reinterpret_cast<const uint64_t*>(addr)[0];
  high = reinterpret_cast<const uint64_t*>(addr)[1];
#else
  asm("ld.global.nc.v2.u64 {%0, %1}, [%2];"
      : "=l"(low), "=l"(high)
      : "l"(addr));
#endif
  reinterpret_cast<uint64_t*>(&val)[0] = low;
  reinterpret_cast<uint64_t*>(&val)[1] = high;
}

template <typename T>
__device__ inline void stream_store128(T* addr, const uint4& val) {
  uint64_t low, high;
  low = reinterpret_cast<const uint64_t*>(&val)[0];
  high = reinterpret_cast<const uint64_t*>(&val)[1];
#if defined(USE_ROCM) || (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ < 800))
  reinterpret_cast<uint64_t*>(addr)[0] = low;
  reinterpret_cast<uint64_t*>(addr)[1] = high;
#else
  asm("st.global.cs.v2.u64 [%0], {%1, %2};" : : "l"(addr), "l"(low), "l"(high));
#endif
}

template <typename T>
static __device__ inline bool is_aligned(const void* addr) {
  return reinterpret_cast<uintptr_t>(addr) % sizeof(T) == 0;
}

template <typename T>
static __device__ inline void load128(uint4& val, const char* addr) {
  for (size_t i = 0; i < detail::BYTES_PER_THREAD / sizeof(T); ++i) {
    reinterpret_cast<T*>(&val)[i] = reinterpret_cast<const T*>(addr)[i];
  }
}

template <>
__device__ inline void load128<uint4>(uint4& val, const char* addr) {
  stream_load128(val, addr);
}

static __device__ inline void load128(uint4& val, const char* addr) {
  if (is_aligned<uint4>(addr)) {
    load128<uint4>(val, addr);
  } else if (is_aligned<int64_t>(addr)) {
    load128<uint64_t>(val, addr);
  } else if (is_aligned<uint32_t>(addr)) {
    load128<uint32_t>(val, addr);
  } else {
    load128<uint8_t>(val, addr);
  }
}

static __device__ __inline__ void get_aligned_region(
    char* ptr,
    const int64_t chunk_size,
    const int64_t alignment,
    int64_t& align_off,
    int64_t& aligned_size) {
  const int64_t ptr_val = reinterpret_cast<uintptr_t>(ptr);
  align_off = detail::div_up(ptr_val, alignment) * alignment - ptr_val;
  aligned_size = (chunk_size - align_off) / alignment * alignment;
}

__device__ inline uint4 get_zero_uint4() {
  uint4 zero;
  reinterpret_cast<uint64_t*>(&zero)[0] = 0;
  reinterpret_cast<uint64_t*>(&zero)[1] = 0;
  return zero;
}

static __device__ __inline__ void copy_chunk(
    char* dst,
    const char* src,
    int64_t chunk_size,
    int64_t thread_idx,
    int64_t num_threads) {
  if (chunk_size < num_threads) {
    if (thread_idx < chunk_size) {
      dst[thread_idx] = src[thread_idx];
    }
    return;
  }

  // Identify the region in which writes are guaranteed to be 128-bit aligned
  int64_t align_off, aligned_size;
  get_aligned_region(
      dst, chunk_size, detail::BYTES_PER_THREAD, align_off, aligned_size);

  for (int64_t off = align_off + thread_idx * detail::BYTES_PER_THREAD;
       off < align_off + aligned_size;
       off += num_threads * detail::BYTES_PER_THREAD) {
    uint4 val;
    // Oppurtunistically vectorize reads
    load128(val, &src[off]);
    stream_store128(&dst[off], val);
  }

  // Handle unaligned regions
  if (thread_idx < align_off && thread_idx < chunk_size) {
    dst[thread_idx] = src[thread_idx];
  }
  if (align_off + aligned_size + thread_idx < chunk_size) {
    dst[align_off + aligned_size + thread_idx] =
        src[align_off + aligned_size + thread_idx];
  }
}

static __global__ void split_with_sizes_copy_out_contiguous_no_cast_kernel(
    char** dst_base_addrs,
    char** src_base_addrs,
    int64_t* split_chunk_sizes,
    int64_t* block_idx_to_split_idx,
    int64_t* blocks_cumsums,
    int64_t src_stride,
    int64_t num_chunks) {
  const int64_t split_idx = block_idx_to_split_idx[blockIdx.x];
  const int64_t split_blocks =
      blocks_cumsums[split_idx + 1] - blocks_cumsums[split_idx];
  const int64_t split_threads = split_blocks * blockDim.x;
  const int64_t split_thread_idx =
      (blockIdx.x - blocks_cumsums[split_idx]) * blockDim.x + threadIdx.x;
  const int64_t split_chunk_size = split_chunk_sizes[split_idx];

  char* dst_base_addr = dst_base_addrs[split_idx];
  char* src_base_addr = src_base_addrs[split_idx];

  for (int64_t i = blockIdx.y; i < num_chunks; i += gridDim.y) {
    copy_chunk(
        dst_base_addr + i * split_chunk_size,
        src_base_addr + i * src_stride,
        split_chunk_size,
        split_thread_idx,
        split_threads);
  }
}

// Calculate the base addr for each split.
static inline std::vector<int64_t> get_split_base_addrs(
    const at::Tensor& tensor,
    at::IntArrayRef split_sizes,
    int64_t dim) {
  const auto* data_ptr = static_cast<char*>(tensor.data_ptr());
  const auto strides = tensor.strides();
  const auto element_sz = tensor.element_size();
  int64_t off = 0;
  std::vector<int64_t> split_base_addrs;
  split_base_addrs.reserve(split_sizes.size());
  for (const auto& split_size : split_sizes) {
    split_base_addrs.push_back(reinterpret_cast<int64_t>(data_ptr + off));
    off += split_size * strides[dim] * element_sz;
  }
  return split_base_addrs;
}

static inline std::vector<int64_t> get_dst_addrs(at::TensorList out) {
  std::vector<int64_t> addrs;
  addrs.reserve(out.size());
  for (const auto& tensor : out) {
    addrs.push_back(reinterpret_cast<int64_t>(tensor.data_ptr()));
  }
  return addrs;
}

// Calculate the chunk size for each split in bytes.
static inline std::vector<int64_t> get_split_chunk_sizes(
    const at::Tensor& tensor,
    at::IntArrayRef split_sizes,
    int64_t dim) {
  const auto stride = tensor.stride(dim);
  const auto element_sz = tensor.element_size();
  std::vector<int64_t> split_chunk_sizes;
  split_chunk_sizes.reserve(split_sizes.size());
  for (const auto& split_size : split_sizes) {
    split_chunk_sizes.push_back(split_size * stride * element_sz);
  }
  return split_chunk_sizes;
}

// Calculate the chunk stride in bytes. This is the same for all splits.
static inline int64_t get_chunk_stride(const at::Tensor& tensor, int64_t dim) {
  int64_t stride = 1;
  for (int64_t d = dim; d < tensor.dim(); ++d) {
    stride *= tensor.sizes()[d];
  }
  return stride * tensor.element_size();
}

// Calculate the number of chunks. This is the same for all splits.
static inline int64_t get_num_chunks(const at::Tensor& tensor, int64_t dim) {
  int64_t num_chunks = tensor.numel();
  for (int64_t d = dim; d < tensor.dim(); ++d) {
    num_chunks /= tensor.sizes()[d];
  }
  return num_chunks;
}

// Pack multiple std::vector<int64_t> into a single cuda tensor.
std::pair<at::Tensor, std::vector<int64_t*>> pack_vecs(
    std::vector<const std::vector<int64_t>*> vecs,
    const at::Device& device) {
  int64_t numel = 0;
  for (const auto* vec : vecs) {
    numel += vec->size();
  }

  auto packed = at::empty(
      {numel}, at::TensorOptions().dtype(at::kLong).pinned_memory(true));
  size_t offset = 0;
  for (const auto* vec : vecs) {
    memcpy(
        packed.data_ptr<int64_t>() + offset,
        vec->data(),
        sizeof(int64_t) * vec->size());
    offset += vec->size();
  }
  packed = packed.to(device, /*non_blocking=*/true);

  std::vector<int64_t*> ptrs;
  ptrs.reserve(vecs.size());
  offset = 0;
  for (const auto* vec : vecs) {
    ptrs.push_back(packed.data_ptr<int64_t>() + offset);
    offset += vec->size();
  }
  return std::make_pair(std::move(packed), std::move(ptrs));
}

static __device__ __inline__ void copy_chunk_with_pad(
  char* dst,
  const char* src,
  int64_t max_chunk_size,
  int64_t actual_chunk_size,
  int64_t thread_idx,
  int64_t num_threads
) {
  if (max_chunk_size < num_threads) {
    int64_t val = 0;
    if (thread_idx < actual_chunk_size) {
      val = src[thread_idx];
    }
    if(thread_idx < max_chunk_size) {
      dst[thread_idx] = val;
    }
    return;
  }
  uint4 zero = get_zero_uint4();
  int64_t align_off, aligned_size;
  get_aligned_region(dst, actual_chunk_size, BYTES_PER_THREAD, align_off, aligned_size);
  int64_t align_end = align_off + aligned_size;
  for (
    int64_t i = align_off + thread_idx * BYTES_PER_THREAD;
    i < align_end;
    i += num_threads * BYTES_PER_THREAD
  ) {
    uint4 val = zero;
    if(is_aligned<uint4>(src + i)) {
      stream_load128(val, src + i);
    } else {
      for (size_t j = 0; j < BYTES_PER_THREAD; ++j) {
        reinterpret_cast<char*>(&val)[j] = src[i + j];
      }
    }
    stream_store128(&dst[i], val);
  }
  if(thread_idx < align_off && thread_idx < max_chunk_size) {
    char val = (char) 0;
    if (thread_idx < actual_chunk_size) {
      val = src[thread_idx];
    }
    dst[thread_idx] = val;
  }
  if(align_end + thread_idx < max_chunk_size) {
    char val = (char) 0;
    if (align_end + thread_idx < actual_chunk_size) {
      val = src[align_end + thread_idx];
    }
    dst[align_end + thread_idx] = val;
  }
}

static __global__ void chunk_cat_cuda_kernel(
  char** src,
  char* dst,
  int64_t* block_idx_to_tensor_idx,
  int64_t* block_idx_to_start_tensor_bytes,
  int64_t* start_block_idx_per_tensor_chunk,
  int64_t* actual_tensor_sizes,
  int64_t* pad_tensor_chunk_sizes,
  int64_t* num_blocks_per_tensor_chunk,
  int64_t slice_size,
  int64_t chunk_size) {
  const int64_t slice_idx = blockIdx.z;
  const int64_t chunk_idx = blockIdx.y;
  const int64_t tensor_idx = block_idx_to_tensor_idx[blockIdx.x];
  const int64_t tile_idx = blockIdx.x - start_block_idx_per_tensor_chunk[tensor_idx];
  // Number of threads for the `tensor_idx`-th tensor chunk.
  const int64_t num_threads = num_blocks_per_tensor_chunk[tensor_idx] * BLOCK_SIZE;
  const int64_t thread_idx = tile_idx * BLOCK_SIZE + threadIdx.x;
  const char* src_addr = src[tensor_idx]
      + slice_idx * actual_tensor_sizes[tensor_idx]
      + chunk_idx * pad_tensor_chunk_sizes[tensor_idx];
  char* dst_addr = dst
      + slice_idx * slice_size
      + chunk_idx  * chunk_size
      + block_idx_to_start_tensor_bytes[tensor_idx];
  const int64_t actual_copy_size = minInt64(
    pad_tensor_chunk_sizes[tensor_idx],
    maxInt64(0, actual_tensor_sizes[tensor_idx]-chunk_idx * pad_tensor_chunk_sizes[tensor_idx])
  );
  copy_chunk_with_pad(
    dst_addr,
    src_addr,
    pad_tensor_chunk_sizes[tensor_idx],
    actual_copy_size,
    thread_idx,
    num_threads
  );
}

void assert_leading_dimension_matches(TensorList tensors, uint64_t dim) {
  const auto num_tensors = tensors.size();
  TORCH_CHECK(
    num_tensors > 0,
    "assert_leading_dimension_matches() has invalid args: should have at least 1 input tensors"
  );
  std::vector<c10::SymInt> leading_dim_sizes;
  for (const auto i : c10::irange(dim)) {
    leading_dim_sizes.push_back(tensors[0].size(i));
  }
  for (const auto i : c10::irange(num_tensors)) {
    at::Tensor tensor = tensors[i];
    TORCH_CHECK(tensor.numel() > 0, "assert_leading_dimension_matches() error: tensor should have at least 1 element");
    auto sizes = tensor.sizes();
    TORCH_CHECK(sizes.size() >= dim, "assert_leading_dimension_matches() error: invalid dim");
    for(const auto j : c10::irange(dim)) {
      TORCH_CHECK(
        tensor.size(j) == leading_dim_sizes[j],
        "chunk_cat_cuda() has invalid args: tensors should have same sizes in the first dim dimensions"
      );
    }
  }
}
} // namespace detail

// See [CUDA fast path for split_with_sizes_copy.out]
void split_with_sizes_copy_out_cuda_contiguous_no_cast(
    const at::Tensor& self,
    at::IntArrayRef split_sizes,
    int64_t dim,
    at::TensorList out) {
  const auto device = self.device();
  const auto src_base_addrs =
      detail::get_split_base_addrs(self, split_sizes, dim);
  const auto dst_base_addrs = detail::get_dst_addrs(out);
  const auto src_stride = detail::get_chunk_stride(self, dim);
  const auto split_chunk_sizes =
      detail::get_split_chunk_sizes(self, split_sizes, dim);
  const auto num_chunks = detail::get_num_chunks(self, dim);

  // Calculate the number of blocks required for the first chunk across all
  // splits, assuming each thread only processes BYTES_PER_THREAD bytes.
  int64_t num_blocks = 0;
  for (const auto& split_chunk_size : split_chunk_sizes) {
    num_blocks += detail::div_up(
        split_chunk_size, detail::BLOCK_SIZE * detail::BYTES_PER_THREAD);
  }

  // Calculate the maximum number of blocks to launch. Only consider
  // maxThreadsPerMultiProcessor as a limiting factor as the kernel uses no
  // shared memory and little registers. Over-subscribe the SMs to hide I/O
  // latency.
  const auto num_sms =
      at::cuda::getCurrentDeviceProperties()->multiProcessorCount;
  const auto max_threads_per_sm =
      at::cuda::getCurrentDeviceProperties()->maxThreadsPerMultiProcessor;
  const int64_t max_blocks =
      num_sms * max_threads_per_sm / detail::BLOCK_SIZE * 2.0;

  // Make each thread process BYTES_PER_THREAD * iter_factor bytes to regulate
  // block size. Spread iter_factor evenly between chunks_per_block and
  // iters_per_chunk.
  int64_t iter_factor = detail::div_up(num_blocks * num_chunks, max_blocks);
  int64_t chunks_per_block = std::ceil(std::sqrt(iter_factor));
  chunks_per_block = std::min(chunks_per_block, num_chunks);
  const int64_t iters_per_chunk = detail::div_up(iter_factor, chunks_per_block);

  // Launch a logically jagged grid of shape
  // (chunk_size*, num_splits, num_chunks / chunks_per_block)
  // backed by a physical grid of shape
  // (sum(chunk_size), num_chunks / chunks_per_block).
  // A block can find its split_idx via block_idx_to_split_idx.
  std::vector<int64_t> block_idx_to_split_idx;
  std::vector<int64_t> blocks_cumsums{0};
  block_idx_to_split_idx.reserve(num_blocks);
  for (size_t split_idx = 0; split_idx < split_sizes.size(); ++split_idx) {
    const auto blocks = detail::div_up(
        split_chunk_sizes[split_idx],
        detail::BLOCK_SIZE * detail::BYTES_PER_THREAD * iters_per_chunk);
    block_idx_to_split_idx.insert(
        block_idx_to_split_idx.end(), blocks, split_idx);
    blocks_cumsums.push_back(blocks_cumsums.back() + blocks);
  }

  dim3 blocks(blocks_cumsums.back(), num_chunks / chunks_per_block, 1);
  dim3 threads(detail::BLOCK_SIZE, 1, 1);

  auto [_, ptrs] = detail::pack_vecs(
      {&dst_base_addrs,
       &src_base_addrs,
       &split_chunk_sizes,
       &block_idx_to_split_idx,
       &blocks_cumsums},
      device);

  detail::split_with_sizes_copy_out_contiguous_no_cast_kernel<<<
      blocks,
      threads,
      0,
      at::cuda::getCurrentCUDAStream()>>>(
      /*dst_base_addrs=*/reinterpret_cast<char**>(ptrs[0]),
      /*src_base_addrs=*/reinterpret_cast<char**>(ptrs[1]),
      /*split_chunk_sizes=*/ptrs[2],
      /*block_idx_to_split_idx=*/ptrs[3],
      /*blocks_cumsums=*/ptrs[4],
      src_stride,
      num_chunks);
  C10_CUDA_KERNEL_LAUNCH_CHECK();
}

void split_with_sizes_copy_out_cuda(
    const Tensor& self,
    IntArrayRef split_sizes,
    int64_t dim,
    TensorList out) {
  bool contiguous_no_cast = self.is_non_overlapping_and_dense();
  for (const auto& t : out) {
    contiguous_no_cast &= t.is_non_overlapping_and_dense();
    contiguous_no_cast &= (t.dtype() == self.dtype());
  }
  if (contiguous_no_cast) {
    // Perform equivalent checks performed by the composite impl
    if (dim < 0) {
      dim = at::maybe_wrap_dim(dim, self.dim());
    }
    TORCH_CHECK(
        self.dim() != 0, "split expects at least a 1-dimensional tensor")

    const int64_t dim_size = self.size(dim);
    int64_t split_sizes_sum = 0;
    for (const auto i : c10::irange(split_sizes.size())) {
      TORCH_CHECK(
          split_sizes[i] >= 0,
          "split_with_sizes expects split_sizes have only non-negative ",
          "entries, but got split_sizes=",
          split_sizes[i]);
      split_sizes_sum += split_sizes[i];
    }
    TORCH_CHECK(
        split_sizes_sum == dim_size,
        "split_with_sizes expects split_sizes to sum exactly to ",
        dim_size,
        " (input tensor's size at dimension ",
        dim,
        "), ",
        "but got split_sizes=",
        split_sizes);

    TORCH_CHECK(
        out.size() == split_sizes.size(),
        "split_with_sizes_copy_out() expected an out= argument of size ",
        split_sizes.size(),
        ", got size ",
        out.size());

    auto out_shape = self.sizes().vec();
    for (const auto i : c10::irange(split_sizes.size())) {
      out_shape[dim] = split_sizes[i];
      if (resize_output_check(out[i], out_shape)) {
        out[i].resize_(out_shape);
      }
      TORCH_CHECK(
          out[i].dtype() == self.dtype(),
          "Expected out tensor to have dtype ",
          self.dtype(),
          ", but got ",
          out[i].dtype(),
          " instead");
      TORCH_CHECK(
          out[i].device() == self.device(),
          "Expected out tensor to have device ",
          self.device(),
          ", but got ",
          out[i].device(),
          " instead");
    }
    split_with_sizes_copy_out_cuda_contiguous_no_cast(
        self, split_sizes, dim, out);
  } else {
    at::native::split_with_sizes_copy_out(self, split_sizes, dim, out);
  }
}

Tensor chunk_cat_cuda(
  TensorList tensors,
  int64_t dim,
  int64_t num_chunks
) {
  TORCH_CHECK(!tensors.empty(),
           "chunk_cat expects a non-empty TensorList");
  auto num_tensors = tensors.size();
  const auto device = tensors[0].device();
  dim = at::maybe_wrap_dim(dim, tensors[0].dim());
  detail::assert_leading_dimension_matches(tensors, (uint64_t)dim);
  int64_t leading_dim = 1;
  if (dim > 0) {
    leading_dim = c10::multiply_integers(tensors[0].sizes().slice(0, dim));
  }
  std::vector<int64_t> pad_tensor_chunk_sizes;
  std::vector<int64_t> num_blocks_per_tensor_chunk;
  std::vector<int64_t> start_block_idx_per_tensor_chunk{0};
  std::vector<int64_t> actual_tensor_sizes;
  std::vector<int64_t> block_idx_to_start_tensor_bytes{0};
  std::vector<int64_t> srcs;
  pad_tensor_chunk_sizes.reserve(num_tensors);
  num_blocks_per_tensor_chunk.reserve(num_tensors);
  start_block_idx_per_tensor_chunk.reserve(num_tensors + 1);
  actual_tensor_sizes.reserve(num_tensors);
  block_idx_to_start_tensor_bytes.reserve(num_tensors);
  srcs.reserve(num_tensors);
  //  block_idx_to_tensor_idx cannot be reserved since the number of blocks is data dependent.
  std::vector<int64_t> block_idx_to_tensor_idx;
  int64_t chunk_size = 0;
  for (const auto i : c10::irange(num_tensors)) {
    at::Tensor tensor = tensors[i];
    srcs.push_back(reinterpret_cast<int64_t>(tensor.data_ptr()));
    TORCH_CHECK(
      tensor.is_cuda() && tensor.is_non_overlapping_and_dense(),
      "chunk_cat_cuda() error: invalid input tensor"
    );
    auto sizes = tensor.sizes();
    const int64_t size_along_dim = sizes[dim];
    int64_t trailing_numel = 1;
    if(sizes.size() > (uint64_t)dim + 1) {
      trailing_numel = c10::multiply_integers(sizes.slice(dim+1, sizes.size()-dim-1));
    }
    const int64_t pad_size_along_dim = detail::div_up(size_along_dim, num_chunks) * num_chunks;
    const int64_t pad_tensor_chunk_size = pad_size_along_dim * trailing_numel * tensor.element_size() / num_chunks;
    pad_tensor_chunk_sizes.push_back(pad_tensor_chunk_size);
    chunk_size += pad_tensor_chunk_size;
    // Number of blocks required to process this tensor chunk.
    const int64_t num_blocks = detail::div_up(pad_tensor_chunk_size, detail::TILE_SIZE);
    num_blocks_per_tensor_chunk.push_back(num_blocks);
    start_block_idx_per_tensor_chunk.push_back(start_block_idx_per_tensor_chunk.back() + num_blocks);
    block_idx_to_tensor_idx.insert(block_idx_to_tensor_idx.end(), num_blocks, i);
    actual_tensor_sizes.push_back(size_along_dim * trailing_numel * tensor.element_size());
    block_idx_to_start_tensor_bytes.push_back(block_idx_to_start_tensor_bytes.back() + pad_tensor_chunk_size);
  }
  const int64_t trailing_dim = chunk_size / tensors[0].element_size();
  Tensor out = tensors[0].new_empty({leading_dim * num_chunks * trailing_dim});
  const int64_t num_blocks_per_chunk = start_block_idx_per_tensor_chunk.back();
  const int64_t slice_size = num_chunks * chunk_size;
  auto packed = detail::pack_vecs(
    {&srcs,
     &block_idx_to_tensor_idx,
     &block_idx_to_start_tensor_bytes,
     &start_block_idx_per_tensor_chunk,
     &actual_tensor_sizes,
     &pad_tensor_chunk_sizes,
     &num_blocks_per_tensor_chunk},
     device);
  dim3 blocks(num_blocks_per_chunk, num_chunks, leading_dim);
  dim3 threads(detail::BLOCK_SIZE, 1, 1);
  detail::chunk_cat_cuda_kernel<<<blocks, threads, 0, at::cuda::getCurrentCUDAStream()>>>(
    /*srcs=*/reinterpret_cast<char**>(packed.second[0]),
    reinterpret_cast<char*>(out.data_ptr()),
    /*block_idx_to_tensor_idx=*/packed.second[1],
    /*block_idx_to_start_tensor_bytes=*/packed.second[2],
    /*start_block_idx_per_tensor_chunk=*/packed.second[3],
    /*actual_tensor_sizes=*/packed.second[4],
    /*pad_tensor_chunk_sizes=*/packed.second[5],
    /*num_blocks_per_tensor_chunk=*/packed.second[6],
    slice_size,
    chunk_size
  );
  C10_CUDA_KERNEL_LAUNCH_CHECK();
  return out.view({leading_dim, num_chunks, trailing_dim});
}
} // namespace at::native
