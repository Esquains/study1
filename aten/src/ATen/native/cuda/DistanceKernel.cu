#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/cuda/Exceptions.h>
#include <math.h>
#include <THC/THCTensorMathReduce.cuh>

#include <ATen/native/Distance.h>

namespace at {
namespace native {

namespace {

static const int forward_threads = 256;

template <typename scalar_t>
static __forceinline__ __device__ scalar_t device_sqrt(scalar_t val);

template <>
__forceinline__ __device__ float device_sqrt(float val) {
  return ::sqrtf(val);
}

template <>
__forceinline__ __device__ double device_sqrt(double val) {
  return ::sqrt(val);
}

template <typename scalar_t>
struct dists {
  static __forceinline__ __device__ scalar_t sign(scalar_t val) {
    return (0 < val) - (val < 0);
  }

  // Zero norm
  struct zero {
    static __forceinline__ __device__ void inc(
        scalar_t& agg,
        const scalar_t diff,
        const scalar_t p) {
      agg += diff != 0.0;
    }
    static __forceinline__ __device__ scalar_t
    finish(const scalar_t agg, const scalar_t p) {
      return agg;
    }
    static __forceinline__ __device__ void agg(
        scalar_t& update,
        const scalar_t other) {
      update += other;
    }
  };

  // One norm
  struct one {
    static __forceinline__ __device__ void inc(
        scalar_t& agg,
        const scalar_t diff,
        const scalar_t p) {
      agg += diff;
    }
    static __forceinline__ __device__ scalar_t
    finish(const scalar_t agg, const scalar_t p) {
      return agg;
    }
    static __forceinline__ __device__ void agg(
        scalar_t& update,
        const scalar_t other) {
      update += other;
    }
    static __forceinline__ __device__ scalar_t backward(
        const scalar_t diff,
        const scalar_t grad,
        const scalar_t dist,
        const scalar_t p) {
      return grad * sign(diff);
    }
  };

  // Special case backward when p is less than two
  struct lt_two {
    static __forceinline__ __device__ scalar_t backward(
        const scalar_t diff,
        const scalar_t grad,
        const scalar_t dist,
        const scalar_t p) {
      return dist == 0.0 ? 0
                         : sign(diff) * std::pow(std::abs(diff), p - 1) * grad /
              std::pow(dist, p - 1);
    }
  };

  // Two norm
  struct two {
    static __forceinline__ __device__ void inc(
        scalar_t& agg,
        const scalar_t diff,
        const scalar_t p) {
      agg += diff * diff;
    }
    static __forceinline__ __device__ scalar_t
    finish(const scalar_t agg, const scalar_t p) {
      return device_sqrt<scalar_t>(agg);
    }
    static __forceinline__ __device__ void agg(
        scalar_t& update,
        const scalar_t other) {
      update += other;
    }
    static __forceinline__ __device__ scalar_t backward(
        const scalar_t diff,
        const scalar_t grad,
        const scalar_t dist,
        const scalar_t p) {
      return dist == 0.0 ? 0 : grad * diff / dist;
    }
  };

  // General p norm
  struct p {
    static __forceinline__ __device__ void inc(
        scalar_t& agg,
        const scalar_t diff,
        const scalar_t p) {
      agg += std::pow(diff, p);
    }
    static __forceinline__ __device__ scalar_t
    finish(const scalar_t agg, const scalar_t p) {
      return std::pow(agg, static_cast<scalar_t>(1) / p);
    }
    static __forceinline__ __device__ void agg(
        scalar_t& update,
        const scalar_t other) {
      update += other;
    }
    static __forceinline__ __device__ scalar_t backward(
        const scalar_t diff,
        const scalar_t grad,
        const scalar_t dist,
        const scalar_t p) {
      return dist == 0.0 ? 0
                         : diff * std::pow(std::abs(diff), p - 2) * grad /
              std::pow(dist, p - 1);
    }
  };

  // Inf norm
  struct inf {
    static __forceinline__ __device__ void inc(
        scalar_t& agg,
        const scalar_t diff,
        const scalar_t p) {
      if (diff > agg) {
        agg = diff;
      }
    }
    static __forceinline__ __device__ scalar_t
    finish(const scalar_t agg, const scalar_t p) {
      return agg;
    }
    static __forceinline__ __device__ void agg(
        scalar_t& update,
        const scalar_t other) {
      if (other > update) {
        update = other;
      }
    }
    static __forceinline__ __device__ scalar_t backward(
        const scalar_t diff,
        const scalar_t grad,
        const scalar_t dist,
        const scalar_t p) {
      return grad * sign(diff) * (std::abs(diff) == dist);
    }
  };
};

template <typename scalar_t, typename F>
__device__ static inline scalar_t reduce_agg(scalar_t agg) {
  for (int offset = warpSize / 2; offset > 0; offset /= 2) {
    F::agg(agg, WARP_SHFL_DOWN(agg, offset));
  }

  __shared__ scalar_t shared[forward_threads];
  int lane = threadIdx.x % warpSize;
  int warp_id = threadIdx.x / warpSize;
  if (lane == 0) {
    shared[warp_id] = agg;
  }

  __syncthreads();
  agg = (threadIdx.x < blockDim.x / warpSize) ? shared[lane] : 0.0;
  if (warp_id == 0) {
    for (int offset = blockDim.x / warpSize / 2; offset > 0; offset /= 2) {
      F::agg(agg, WARP_SHFL_DOWN(agg, offset));
    }
  }
  return agg;
}

template <typename scalar_t, typename F>
__global__ static void pdist_kernel_cuda_impl(
    scalar_t* result,
    const scalar_t* self,
    const int64_t n,
    const int64_t d,
    const scalar_t p,
    const double n2,
    const double n2_squared_minus_1) {
  const int64_t r_k = blockIdx.x;
  const int64_t b_l = blockIdx.y;
  const int64_t stride = blockDim.x;

  // The -1 accounts for floating point truncation issues
  int64_t n_i = static_cast<int64_t>(
      (n2 - device_sqrt<double>(n2_squared_minus_1 - 2 * r_k)));
  int64_t n_j = r_k - n * n_i + n_i * (n_i + 1) / 2 + n_i + 1;

  const scalar_t* const start = self + (b_l * n + n_i) * d;
  const scalar_t* const end = start + d;
  const scalar_t* a = start + threadIdx.x;
  const scalar_t* b = self + (b_l * n + n_j) * d + threadIdx.x;

  scalar_t agg = 0.0;
  for (; a < end; a += stride, b += stride) {
    F::inc(agg, std::abs(*a - *b), p);
  }

  agg = reduce_agg<scalar_t, F>(agg);
  if (threadIdx.x == 0) {
    const int64_t r = n * (n - 1) / 2;
    result[b_l * r + r_k] = F::finish(agg, p);
  }
}

template <typename scalar_t, typename F>
__global__ static void cdist_backward_kernel_cuda_impl(
    scalar_t* buffer,
    const scalar_t* grad,
    const scalar_t* x1,
    const scalar_t* x2,
    const scalar_t* dist,
    int64_t gs,
    const scalar_t p,
    const int64_t r1,
    const int64_t r2,
    const int64_t m,
    const int64_t count) {
  const int k = blockIdx.y * blockDim.y + threadIdx.y;
  const int init = blockIdx.x * blockDim.x + threadIdx.x;
  const int stride = blockDim.x * gridDim.x;

  if (k >= count) {
    return;
  }

  int64_t i = k / r2;
  int64_t j = k % r2;

  const scalar_t grad_k = grad[k * gs];
  const scalar_t dist_k = dist[k];

  const scalar_t* const start = x1 + i * m;
  const scalar_t* const end = start + m;
  const scalar_t* self_i = start + init;
  const scalar_t* self_j = x2 + j * m + init;

  scalar_t* buff_i = buffer + (r1 * j + i) * m + init;

  for (; self_i < end; self_i += stride, self_j += stride, buff_i += stride) {
    const scalar_t res = F::backward(*self_i - *self_j, grad_k, dist_k, p);
    *buff_i = res;
  }
}

template <typename scalar_t, typename F>
__global__ static void pdist_backward_kernel_cuda_impl(
    scalar_t* result,
    const scalar_t* grad,
    const scalar_t* self,
    const scalar_t* dist,
    int64_t gs_l,
    int64_t gs_k,
    const int64_t n,
    const int64_t d,
    const int64_t r,
    const scalar_t p,
    const double n2,
    const double n2_squared_minus_1) {
  const int64_t b_l = blockIdx.z;
  const int64_t r_k = blockIdx.x * blockDim.x + threadIdx.x;
  const int64_t init = blockIdx.y * blockDim.y + threadIdx.y;
  const int64_t stride = blockDim.x * gridDim.x;

  if (r_k >= r) {
    return;
  }

  // The -1 accounts for floating point truncation issues
  int64_t n_i = static_cast<int64_t>(
      (n2 - device_sqrt<double>(n2_squared_minus_1 - 2 * r_k)));
  int64_t n_j = r_k - n * n_i + n_i * (n_i + 1) / 2 + n_i + 1;
  int64_t n_ir = n_j - n_i - 1;
  int64_t n_jr = n - 2 - n_i;

  const scalar_t grad_lk = grad[b_l * gs_l + r_k * gs_k];
  const scalar_t dist_lk = dist[b_l * r + r_k];

  const scalar_t* const start = self + (b_l * n + n_i) * d;
  const scalar_t* const end = start + d;
  const scalar_t* self_i = start + init;
  const scalar_t* self_j = self_i + (n_j - n_i) * d;
  scalar_t* result_i = result + ((b_l * (n - 1) + n_ir) * n + n_i) * d + init;
  scalar_t* result_j = result_i + ((n_jr - n_ir) * n + n_j - n_i) * d;
  for (; self_i < end; self_i += stride,
                       self_j += stride,
                       result_i += stride,
                       result_j += stride) {
    const scalar_t res = F::backward(*self_i - *self_j, grad_lk, dist_lk, p);
    *result_i = res;
    *result_j = -res;
  }
}

template <typename scalar_t, typename F>
__global__ static void cdist_kernel_cuda_impl(
    scalar_t* result,
    const scalar_t* x1,
    const scalar_t* x2,
    const scalar_t p,
    const int64_t r1,
    const int64_t r2,
    const int64_t m) {
  const int k = blockIdx.x;
  const int64_t i = k / r2;
  const int64_t j = k % r2;
  const int stride = blockDim.x;

  const scalar_t* const start = x1 + i * m;
  const scalar_t* const end = start + m;
  const scalar_t* a = start + threadIdx.x;
  const scalar_t* b = x2 + j * m + threadIdx.x;

  scalar_t agg = 0.0;
  for (; a < end; a += stride, b += stride) {
    F::inc(agg, std::abs(*a - *b), p);
  }
  agg = reduce_agg<scalar_t, F>(agg);
  if (threadIdx.x == 0) {
    result[k] = F::finish(agg, p);
  }
}

void cdist_kernel_impl(
    Tensor& result,
    const Tensor& x1,
    const Tensor& x2,
    double p) {
  int64_t r1 = x1.size(-2);
  int64_t r2 = x2.size(-2);
  int64_t m = x1.size(-1);
  const dim3 grid(r1 * r2);
  const dim3 block(forward_threads);

  AT_DISPATCH_FLOATING_TYPES(x1.scalar_type(), "cdist_cuda", [&] {
    if (p == 0.0) {
      cdist_kernel_cuda_impl<scalar_t, dists<scalar_t>::zero><<<grid, block>>>(
          result.data<scalar_t>(),
          x1.data<scalar_t>(),
          x2.data<scalar_t>(),
          p,
          r1,
          r2,
          m);
    } else if (p == 1.0) {
      cdist_kernel_cuda_impl<scalar_t, dists<scalar_t>::one><<<grid, block>>>(
          result.data<scalar_t>(),
          x1.data<scalar_t>(),
          x2.data<scalar_t>(),
          p,
          r1,
          r2,
          m);
    } else if (p == 2.0) {
      cdist_kernel_cuda_impl<scalar_t, dists<scalar_t>::two><<<grid, block>>>(
          result.data<scalar_t>(),
          x1.data<scalar_t>(),
          x2.data<scalar_t>(),
          p,
          r1,
          r2,
          m);
    } else if (std::isinf(p)) {
      cdist_kernel_cuda_impl<scalar_t, dists<scalar_t>::inf><<<grid, block>>>(
          result.data<scalar_t>(),
          x1.data<scalar_t>(),
          x2.data<scalar_t>(),
          p,
          r1,
          r2,
          m);
    } else {
      cdist_kernel_cuda_impl<scalar_t, dists<scalar_t>::p><<<grid, block>>>(
          result.data<scalar_t>(),
          x1.data<scalar_t>(),
          x2.data<scalar_t>(),
          p,
          r1,
          r2,
          m);
    }
  });
  AT_CUDA_CHECK(hipGetLastError());
}

void pdist_forward_kernel_impl(Tensor& result, const Tensor& self, double p) {
  int64_t b = self.size(0);
  int64_t n = self.size(1);
  int64_t d = self.size(2);
  int64_t r = result.size(1);

  AT_CHECK(
      b < (int32_t(1) << 16),
      "The number of batches can't exceed ",
      (int32_t(1) << 16) - 1,
      " but was ",
      b);
  AT_CHECK(
      r < (int64_t(1) << 32),
      "The number of combinations can't exceed ",
      (int64_t(1) << 32) - 1,
      " but was ",
      r);

  const dim3 grid(r, b);
  const dim3 block(forward_threads);

  // https://github.com/pytorch/pytorch/issues/15511 demonstrated we need to do
  // some math in fp64 -- this is just minimizing the amount of fp64 math we do
  // on the device.
  const double n2 = n - .5;
  const double n2_squared_minus_1 = n2 * n2 - 1;

  AT_DISPATCH_FLOATING_TYPES(self.scalar_type(), "pdist_cuda", [&] {
    if (p == 0.0) {
      pdist_kernel_cuda_impl<scalar_t, dists<scalar_t>::zero><<<grid, block>>>(
          result.data<scalar_t>(),
          self.data<scalar_t>(),
          n,
          d,
          p,
          n2,
          n2_squared_minus_1);
    } else if (p == 1.0) {
      pdist_kernel_cuda_impl<scalar_t, dists<scalar_t>::one><<<grid, block>>>(
          result.data<scalar_t>(),
          self.data<scalar_t>(),
          n,
          d,
          p,
          n2,
          n2_squared_minus_1);
    } else if (p == 2.0) {
      pdist_kernel_cuda_impl<scalar_t, dists<scalar_t>::two><<<grid, block>>>(
          result.data<scalar_t>(),
          self.data<scalar_t>(),
          n,
          d,
          p,
          n2,
          n2_squared_minus_1);
    } else if (std::isinf(p)) {
      pdist_kernel_cuda_impl<scalar_t, dists<scalar_t>::inf><<<grid, block>>>(
          result.data<scalar_t>(),
          self.data<scalar_t>(),
          n,
          d,
          p,
          n2,
          n2_squared_minus_1);
    } else {
      pdist_kernel_cuda_impl<scalar_t, dists<scalar_t>::p><<<grid, block>>>(
          result.data<scalar_t>(),
          self.data<scalar_t>(),
          n,
          d,
          p,
          n2,
          n2_squared_minus_1);
    }
  });
  AT_CUDA_CHECK(hipGetLastError());
}

void pdist_backward_kernel_impl(
    Tensor& result,
    const Tensor& grad,
    const Tensor& self,
    const double p,
    const Tensor& dist) {
  if (p == 0.0 || grad.numel() == 0 || self.numel() == 0) {
    result.fill_(0);
    return;
  }
  // Be careful with changing these as they affect the maximum dimensions that
  // backward can run on, however these are currently more permissive than the
  // forward pass
  const int block_x = 16;
  const int block_y = 64;

  const int64_t b = self.size(0);
  const int64_t n = self.size(1);
  const int64_t d = self.size(2);
  const int64_t r = dist.size(1);

  AT_CHECK(
      b < (int32_t(1) << 16),
      "The number of batches can't exceed ",
      (int32_t(1) << 16) - 1,
      " but was ",
      b);
  AT_CHECK(
      r < (int64_t(1) << 32) * block_x,
      "The number of combinations can't exceed ",
      (int64_t(1) << 32) * block_x - 1,
      " but was ",
      r);
  AT_CHECK(
      d < (int32_t(1) << 16) * block_y * 8,
      "The number of dimensions can't exceed ",
      (int32_t(1) << 16) * block_y * 8 - 1,
      " but was ",
      d);

  const int grid_x = (r + block_x - 1) / block_x;
  const int grid_y = (d + block_y * 8 - 1) / (block_y * 8);
  const dim3 grid(grid_x, grid_y, b);
  const dim3 block(block_x, block_y);

  // https://github.com/pytorch/pytorch/issues/15511 demonstrated we need to do
  // some math in fp64 -- this is just minimizing the amount of fp64 math we do
  // on the device.
  const double n2 = n - .5;
  const double n2_squared_minus_1 = n2 * n2 - 1;

  Tensor buffer = at::empty({b, n - 1, n, d}, result.options());
  AT_DISPATCH_FLOATING_TYPES(self.type(), "pdist_cuda_backward", [&] {
    if (p == 1.0) {
      pdist_backward_kernel_cuda_impl<scalar_t, dists<scalar_t>::one>
          <<<grid, block>>>(
              buffer.data<scalar_t>(),
              grad.data<scalar_t>(),
              self.data<scalar_t>(),
              dist.data<scalar_t>(),
              grad.stride(0),
              grad.stride(1),
              n,
              d,
              r,
              p,
              n2,
              n2_squared_minus_1);
    } else if (p < 2.0) {
      pdist_backward_kernel_cuda_impl<scalar_t, dists<scalar_t>::lt_two>
          <<<grid, block>>>(
              buffer.data<scalar_t>(),
              grad.data<scalar_t>(),
              self.data<scalar_t>(),
              dist.data<scalar_t>(),
              grad.stride(0),
              grad.stride(1),
              n,
              d,
              r,
              p,
              n2,
              n2_squared_minus_1);
    } else if (p == 2.0) {
      pdist_backward_kernel_cuda_impl<scalar_t, dists<scalar_t>::two>
          <<<grid, block>>>(
              buffer.data<scalar_t>(),
              grad.data<scalar_t>(),
              self.data<scalar_t>(),
              dist.data<scalar_t>(),
              grad.stride(0),
              grad.stride(1),
              n,
              d,
              r,
              p,
              n2,
              n2_squared_minus_1);
    } else if (std::isinf(p)) {
      pdist_backward_kernel_cuda_impl<scalar_t, dists<scalar_t>::inf>
          <<<grid, block>>>(
              buffer.data<scalar_t>(),
              grad.data<scalar_t>(),
              self.data<scalar_t>(),
              dist.data<scalar_t>(),
              grad.stride(0),
              grad.stride(1),
              n,
              d,
              r,
              p,
              n2,
              n2_squared_minus_1);
    } else {
      pdist_backward_kernel_cuda_impl<scalar_t, dists<scalar_t>::p>
          <<<grid, block>>>(
              buffer.data<scalar_t>(),
              grad.data<scalar_t>(),
              self.data<scalar_t>(),
              dist.data<scalar_t>(),
              grad.stride(0),
              grad.stride(1),
              n,
              d,
              r,
              p,
              n2,
              n2_squared_minus_1);
    }
  });
  AT_CUDA_CHECK(hipGetLastError());

  at::sum_out(result, buffer, 1);
}

void cdist_backward_kernel_impl(
    Tensor& result,
    const Tensor& grad,
    const Tensor& x1,
    const Tensor& x2,
    const double p,
    const Tensor& dist) {
  if (p == 0.0 || grad.numel() == 0 || x1.numel() == 0 || x2.numel() == 0) {
    result.fill_(0);
    return;
  }

  const int64_t r1 = x1.size(-2);
  const int64_t r2 = x2.size(-2);
  const int64_t m = x1.size(-1);
  const int block_x = 64;
  const int block_y = 16;
  const int grid_x = (m + block_x * 8 - 1) / (block_x * 8);
  const int grid_y = (dist.numel() + block_y - 1) / block_y;

  const dim3 grid(grid_x, grid_y);
  const dim3 block(block_x, block_y);

  const int64_t count = dist.numel();

  Tensor buffer = at::empty({r2, r1, m}, result.options());
  AT_DISPATCH_FLOATING_TYPES(result.scalar_type(), "cdist_cuda_backward", [&] {
    if (p == 1.0) {
      cdist_backward_kernel_cuda_impl<scalar_t, dists<scalar_t>::one>
          <<<grid, block>>>(
              buffer.data<scalar_t>(),
              grad.data<scalar_t>(),
              x1.data<scalar_t>(),
              x2.data<scalar_t>(),
              dist.data<scalar_t>(),
              grad.stride(1),
              p,
              r1,
              r2,
              m,
              count);
    } else if (p < 2.0) {
      cdist_backward_kernel_cuda_impl<scalar_t, dists<scalar_t>::lt_two>
          <<<grid, block>>>(
              buffer.data<scalar_t>(),
              grad.data<scalar_t>(),
              x1.data<scalar_t>(),
              x2.data<scalar_t>(),
              dist.data<scalar_t>(),
              grad.stride(1),
              p,
              r1,
              r2,
              m,
              count);
    } else if (p == 2.0) {
      cdist_backward_kernel_cuda_impl<scalar_t, dists<scalar_t>::two>
          <<<grid, block>>>(
              buffer.data<scalar_t>(),
              grad.data<scalar_t>(),
              x1.data<scalar_t>(),
              x2.data<scalar_t>(),
              dist.data<scalar_t>(),
              grad.stride(1),
              p,
              r1,
              r2,
              m,
              count);
    } else if (std::isinf(p)) {
      cdist_backward_kernel_cuda_impl<scalar_t, dists<scalar_t>::inf>
          <<<grid, block>>>(
              buffer.data<scalar_t>(),
              grad.data<scalar_t>(),
              x1.data<scalar_t>(),
              x2.data<scalar_t>(),
              dist.data<scalar_t>(),
              grad.stride(1),
              p,
              r1,
              r2,
              m,
              count);
    } else {
      cdist_backward_kernel_cuda_impl<scalar_t, dists<scalar_t>::p>
          <<<grid, block>>>(
              buffer.data<scalar_t>(),
              grad.data<scalar_t>(),
              x1.data<scalar_t>(),
              x2.data<scalar_t>(),
              dist.data<scalar_t>(),
              grad.stride(1),
              p,
              r1,
              r2,
              m,
              count);
    }
  });
  AT_CUDA_CHECK(hipGetLastError());

  at::sum_out(result, buffer, 0);
}

} // anonymous namespace

REGISTER_DISPATCH(pdist_forward_stub, &pdist_forward_kernel_impl);
REGISTER_DISPATCH(pdist_backward_stub, &pdist_backward_kernel_impl);
REGISTER_DISPATCH(cdist_stub, &cdist_kernel_impl);
REGISTER_DISPATCH(cdist_backward_stub, &cdist_backward_kernel_impl);

} // namespace native
} // namespace at
