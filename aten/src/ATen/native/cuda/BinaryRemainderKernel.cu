#include "hip/hip_runtime.h"
#include <ATen/Dispatch.h>
#include <ATen/native/DispatchStub.h>
#include <ATen/native/cuda/Loops.cuh>
#include <ATen/native/TensorIterator.h>
#include <ATen/native/BinaryOps.h>
#include <ATen/native/cuda/Math.cuh>

// NOTE: CUDA on Windows requires that the enclosing function
// of a __device__ lambda not have internal linkage.

namespace at { namespace native {

template<typename T>
__host__ __device__ static inline c10::complex<T> floor_wrapper(c10::complex<T> v) {
  return c10::complex<T>(std::floor(v.real()), std::floor(v.imag()));
}

void remainder_kernel_cuda(TensorIterator& iter) {
  if (isIntegralType(iter.dtype(), /*includeBool*/ false)) {
    AT_DISPATCH_INTEGRAL_TYPES(iter.dtype(), "remainder_cuda", [&]() {
      gpu_kernel_with_scalars(iter, []GPU_LAMBDA(scalar_t a, scalar_t b) -> scalar_t {
        scalar_t r = a % b;
        if ((r != 0) && ((r < 0) != (b < 0))) {
          r += b;
        }
        return r;
      });
    });
  }
  else if (isComplexType(iter.dtype())) {
    AT_DISPATCH_COMPLEX_TYPES(iter.dtype(), "remainder_cuda", [&]() {
      gpu_kernel_with_scalars(iter,
        []GPU_LAMBDA(scalar_t a, scalar_t b) -> scalar_t {
          auto q = std::trunc(a.real() / b.real());
          auto r = std::fmod(a.real(), b.real()); 
          auto arg = (a - q * b) / std::abs(a - q * b);
          return arg * r;
        });
    });
  } else {
    AT_DISPATCH_FLOATING_TYPES_AND2(kHalf, kBFloat16, iter.dtype(), "remainder_cuda", [&]() {
      gpu_kernel_with_scalars(iter,
        []GPU_LAMBDA(scalar_t a, scalar_t b) __ubsan_ignore_float_divide_by_zero__ -> scalar_t {
          auto mod = ::fmod(a, b);
          if ((mod != 0) && ((b < 0) != (mod < 0))) mod += b;
          return mod;
        });
    });
  }
}

REGISTER_DISPATCH(remainder_stub, &remainder_kernel_cuda);

}} // namespace at::native
