#include <limits>
#include <ATen/native/UnaryOps.h>
#include <ATen/native/cuda/Loops.cuh>
#include <ATen/Context.h>
#include <ATen/Dispatch.h>
#include <ATen/native/cuda/Loops.cuh>
#include <ATen/native/DispatchStub.h>
#include <ATen/native/TensorIterator.h>

namespace at { namespace native {

void bitwise_not_kernel_cuda(TensorIterator& iter) {
  if (iter.dtype() == ScalarType::Bool) {
    gpu_kernel(iter, []GPU_LAMBDA(bool a) {
      return !a;
    });
  } else {
    AT_DISPATCH_INTEGRAL_TYPES(iter.dtype(), "bitwise_not_cuda", [&]() {
      gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
        return ~a;
      });
    });
  }
}


void sign_kernel_cuda(TensorIterator& iter){
    if (iter.dtype() == ScalarType::Bool) {
      gpu_kernel(iter, []GPU_LAMBDA(bool a){ 
        return a; 
      });
    }else{
      AT_DISPATCH_ALL_TYPES_AND(ScalarType::Half, iter.dtype(), "sign_cuda", [&]() {
          gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
              scalar_t zero = scalar_t(0);
              return (zero < a) - (a < zero);
          });
      });
    }
}

REGISTER_DISPATCH(bitwise_not_stub, &bitwise_not_kernel_cuda);
REGISTER_DISPATCH(sign_stub, &sign_kernel_cuda);
}}
