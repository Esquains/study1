#include "hip/hip_runtime.h"
#include <ATen/Dispatch.h>
#include <ATen/native/DispatchStub.h>
#include <ATen/native/cuda/Loops.cuh>
#include <ATen/native/cuda/Math.cuh>
#include <ATen/native/TensorIterator.h>
#include <ATen/native/BinaryOps.h>

#if defined(__HIPCC__)
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <c10/cuda/CUDAMathCompat.h>
#elif defined(__HIPCC__)
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <c10/hip/HIPMathCompat.h>
#endif

// NOTE: CUDA on Windows requires that the enclosing function
// of a __device__ lambda not have internal linkage.

namespace at { namespace native {

void nextafter_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_TYPES(iter.common_dtype(), "nextafter_cuda", [&]() {
    gpu_kernel_with_scalars(iter, []GPU_LAMBDA(scalar_t a, scalar_t b) -> scalar_t {
      return ::nextafter(a, b);
    });
  });
}

void heaviside_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_ALL_TYPES_AND3(kHalf, kBool, kBFloat16, iter.dtype(), "heaviside_cuda", [&]() {
    gpu_kernel_with_scalars(iter, []GPU_LAMBDA(scalar_t a, scalar_t b) -> scalar_t {
      return a == 0 ? b : static_cast<scalar_t>(a > 0);
    });
  });
}

REGISTER_DISPATCH(nextafter_stub, &nextafter_kernel_cuda);
REGISTER_DISPATCH(heaviside_stub, &heaviside_kernel_cuda);

}} // namespace at::native
