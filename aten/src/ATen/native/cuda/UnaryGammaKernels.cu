#include "hip/hip_runtime.h"
#define TORCH_ASSERT_NO_OPERATORS
#include <limits>
#include <ATen/native/UnaryOps.h>
#include <ATen/native/cuda/Loops.cuh>
#include <ATen/AccumulateType.h>
#include <ATen/Dispatch.h>
#include <ATen/native/DispatchStub.h>
#include <ATen/native/TensorIterator.h>
#include <ATen/native/cuda/Math.cuh>
#include <ATen/native/Math.h>

namespace at { namespace native {

// See note [Jiterator]
const char digamma_name[] = "digamma";
void digamma_kernel_cuda(TensorIteratorBase& iter) {
  #ifdef USE_JITERATOR
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(iter.common_dtype(), "digamma_cuda", [&]() {
      jitted_gpu_kernel</*name=*/digamma_name,
                        /*return_dtype=*/ scalar_t,
                        /*common_dtype=*/ scalar_t,
                        /*arity=*/ 1>(iter, digamma_string);
    });
  #else
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(iter.common_dtype(), "digamma_cuda", [&]() {
      gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
        return calc_digamma(a);
      });
    });
  #endif // USE_JITERATOR
}

// See note [Jiterator]
const char trigamma_name[] = "trigamma";
void trigamma_kernel_cuda(TensorIteratorBase& iter) {
  #ifdef USE_JITERATOR
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(iter.common_dtype(), "trigamma_cuda", [&]() {
      jitted_gpu_kernel</*name=*/trigamma_name,
                        /*return_dtype=*/ scalar_t,
                        /*common_dtype=*/ scalar_t,
                        /*arity=*/ 1>(iter, trigamma_string);
    });
  #else
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(iter.common_dtype(), "trigamma_cuda", [&]() {
      gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
        return calc_trigamma(a);
      });
    });
  #endif // USE_JITERATOR
}

// Note [polygamma jiterator]
// To pass a runtime argument (similar to lambda captures in non-JIT kernels),
// we need to pass to additional arguments to `jitted_gpu_kernel`
// 1. `vector<pair<string, string>>>` where first string is the
//     type of the arguments and second the corresponding name for
//     them in the kernel in the same order as they appear in kernel's function
//     signature.
// 2.  We also need to pass the address of these extra arguments to
//     `jitted_gpu_kernel`
//     in the same order as they appear in kernel's function signature.
//
// NOTE: One big restriction being that these arguments should be after the
// arguments provided by TensorIterator. Eg. While capturing `n`, where
// `scalar_t x` and `scalar_t y` are provided by TensorIterator,
// * foo(scalar_t x, scalar_t y, int n) works!
// * foo(int n, scalar_t x, scalar_y) doesn't work
// * foo(scalar_t x, int n, scalar_y) doesn't work
const char polygamma_name[] = "polygamma";
void polygamma_kernel_cuda(TensorIteratorBase& iter, int64_t n) {
  if (n == 0) {
    digamma_kernel_cuda(iter);
  } else if (n == 1) {
    trigamma_kernel_cuda(iter);
  } else {
#ifdef USE_JITERATOR
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
        iter.common_dtype(), "polygamma_cuda", [&]() {
          jitted_gpu_kernel<
              /*name=*/polygamma_name,
              /*return_dtype=*/scalar_t,
              /*common_dtype=*/scalar_t,
              /*arity=*/1>(
              iter,
              polygamma_string,
              at::cuda::jit::BinaryFuncVariant::NoScalar,
              0,
              {{"int", "n"}}, // extra args to the kernel
              &n); // pointer to the args
        });
#else
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
        iter.common_dtype(), "polygamma_cuda", [&]() {
          gpu_kernel(iter, [=] GPU_LAMBDA(scalar_t a) -> scalar_t {
            return calc_polygamma<scalar_t, /*is_cuda=*/true>(a, static_cast<int>(n));
          });
        });
#endif // USE_JITERATOR
  }
}

const char lgamma_name[] = "lgamma_kernel";
void lgamma_kernel_cuda(TensorIteratorBase& iter) {
  #ifdef USE_JITERATOR
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(iter.common_dtype(), "lgamma_cuda", [&]() {
      jitted_gpu_kernel</*name=*/lgamma_name,
                        /*return_dtype=*/ scalar_t,
                        /*common_dtype=*/ scalar_t,
                        /*arity=*/ 1>(iter, lgamma_string);
    });
  #else
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(iter.common_dtype(), "lgamma_cuda", [&]() {
      gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
        return ::lgamma(a);
      });
    });
  #endif
}

REGISTER_DISPATCH(digamma_stub, &digamma_kernel_cuda);
REGISTER_DISPATCH(polygamma_stub, &polygamma_kernel_cuda);
REGISTER_DISPATCH(lgamma_stub, &lgamma_kernel_cuda);

}} // namespace at::native
