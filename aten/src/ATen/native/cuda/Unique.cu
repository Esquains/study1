#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <THC/THCGeneral.h>
#include <THC/THCThrustAllocator.cuh>
#include <thrust/execution_policy.h>

#include <tuple>
#include <iterator>
#include <thrust/unique.h>
#include <thrust/sort.h>
#include <thrust/scan.h>
#include <thrust/scatter.h>

namespace at {
namespace native{

namespace {

template <
  typename scalar_t,
  typename less_t,
  typename equal_t,
  typename not_equal_t
>
std::tuple<Tensor, Tensor, int64_t> compute_unique(
  scalar_t *data,
  int64_t num_inp,
  const bool return_inverse,
  const bool return_counts,
  TensorOptions options,
  less_t less,
  equal_t equal,
  not_equal_t not_equal
) {
  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  auto allocator = THCThrustAllocator(globalContext().lazyInitCUDA());
  auto policy = thrust::cuda::par(allocator).on(stream);

  //sort
  Tensor inverse_indices;
  if (!return_inverse) {
      inverse_indices = at::empty({0}, options);
      thrust::sort(policy, data, data + num_inp, less);
  } else {
      Tensor sorted_indices = at::arange(0, num_inp, options);
      int64_t* sorted_indices_ptr = sorted_indices.data<int64_t>();
      thrust::sort_by_key<decltype(policy), scalar_t *, int64_t *, less_t>(policy, data, data + num_inp, sorted_indices_ptr, less);
      Tensor inv_loc = at::empty({num_inp}, options);
      inverse_indices = at::empty({num_inp}, options);
      int64_t* inv_loc_ptr = inv_loc.data<int64_t>();
      int64_t* inverse_indices_ptr = inverse_indices.data<int64_t>();
      thrust::adjacent_difference(policy, data, data + num_inp, inv_loc_ptr, not_equal);
      inv_loc[0] = 0;
      thrust::inclusive_scan(policy, inv_loc_ptr, inv_loc_ptr + num_inp, inv_loc_ptr);
      thrust::scatter(policy, inv_loc_ptr, inv_loc_ptr + num_inp, sorted_indices_ptr, inverse_indices_ptr);
  }

  // unique
  Tensor counts = at::empty({0}, options);
  int64_t num_out;
  if (!return_counts) {
      num_out = thrust::unique(policy, data, data + num_inp, equal) - data;
  } else {
      Tensor sorted_indices = at::arange(0, num_inp + 1, options);
      int64_t* sorted_indices_ptr = sorted_indices.data<int64_t>();
      num_out = thrust::unique_by_key(policy, data, data + num_inp, sorted_indices_ptr, equal).first - data;
      sorted_indices[num_out] = num_inp;
      counts.resize_(num_out);
      int64_t* counts_ptr = counts.data<int64_t>();
      thrust::adjacent_difference(policy, sorted_indices_ptr + 1, sorted_indices_ptr + num_out + 1, counts_ptr);
  }

  THCudaCheck(hipGetLastError());
  return std::tuple<Tensor, Tensor, int64_t>(inverse_indices, counts, num_out);
}

template <typename scalar_t>
std::tuple<Tensor, Tensor, Tensor> unique_cuda_template(
    const Tensor& self,
    const bool return_inverse,
    const bool return_counts) {

    Tensor output = self.clone().reshape(-1);
    int64_t num_inp = output.numel();
    scalar_t* output_data = output.data<scalar_t>();

    Tensor inverse_indices, counts;
    int64_t num_out;
    std::tie(inverse_indices, counts, num_out) =
    compute_unique<scalar_t, thrust::less<scalar_t>, thrust::equal_to<scalar_t>,
                   thrust::not_equal_to<scalar_t>>
      (
        output_data, num_inp, return_inverse, return_counts,
        self.options().dtype(kLong),
        thrust::less<scalar_t>(),
        thrust::equal_to<scalar_t>(),
        thrust::not_equal_to<scalar_t>()
      );
    output.resize_(num_out);

    if (return_inverse) {
        inverse_indices.resize_(self.sizes());
    }

    return std::tuple<Tensor, Tensor, Tensor>(output, inverse_indices, counts);
}

template <typename scalar_t>
class UniqueDimLess {
  scalar_t *data;
  int64_t n;
public:
  UniqueDimLess(scalar_t *data, int64_t n): data(data), n(n) {}
  __device__ bool operator()(int64_t a, int64_t b) {
    for (int64_t i = 0; i < n; ++i) {
      scalar_t lhs = data[i + a * n];
      scalar_t rhs = data[i + b * n];
      if (lhs < rhs) {
        return true;
      } else if (lhs > rhs) {
        return false;
      }
    }
    return false;
  }
};

template <typename scalar_t>
class UniqueDimEqual {
  scalar_t *data;
  int64_t n;
public:
  UniqueDimEqual(scalar_t *data, int64_t n): data(data), n(n) {}
  __device__ bool operator()(int64_t a, int64_t b) {
    for (int64_t i = 0; i < n; ++i) {
      scalar_t lhs = data[i + a * n];
      scalar_t rhs = data[i + b * n];
      if (lhs != rhs) {
        return false;
      }
    }
    return true;
  }
};

template <typename scalar_t>
class UniqueDimNotEqual {
  scalar_t *data;
  int64_t n;
public:
  UniqueDimNotEqual(scalar_t *data, int64_t n): data(data), n(n) {}
  __device__ int64_t operator()(int64_t a, int64_t b) {
    for (int64_t i = 0; i < n; ++i) {
      scalar_t lhs = data[i + a * n];
      scalar_t rhs = data[i + b * n];
      if (lhs != rhs) {
        return 1;
      }
    }
    return 0;
  }
};

template <typename scalar_t>
std::tuple<Tensor, Tensor, Tensor> unique_dim_cuda_template(
    const Tensor& self,
    const int64_t dim,
    const bool return_inverse,
    const bool return_counts) {

    /**
     * The idea for implementing this is basically the same as unique.
     * For unique_dim, we are taking the unique with respect to a index
     * tensor, but during the processes, we override the compare and equal
     * operator by checking the data underlying it instead. After the
     * algorithm, we would use index_select to map the resulting indicies
     * to the result on the actual data.
     */

    int64_t num_inp = self.size(dim);
    Tensor input_flat = self.transpose(dim, 0).contiguous().view({num_inp, -1});
    int64_t numel = input_flat.size(1);
    scalar_t *input_flat_ptr = input_flat.data<scalar_t>();

    Tensor indices = at::arange(0, num_inp, self.options().dtype(kLong));
    int64_t *indices_data = indices.data<int64_t>();

    Tensor inverse_indices, counts;
    int64_t num_out;
    std::tie(inverse_indices, counts, num_out) =
    compute_unique<int64_t, UniqueDimLess<scalar_t>, UniqueDimEqual<scalar_t>,
                   UniqueDimNotEqual<scalar_t>>
    (
      indices_data, num_inp, return_inverse, return_counts,
      self.options().dtype(kLong),
      UniqueDimLess<scalar_t>(input_flat_ptr, numel),
      UniqueDimEqual<scalar_t>(input_flat_ptr, numel),
      UniqueDimNotEqual<scalar_t>(input_flat_ptr, numel)
    );
    indices.resize_(num_out);

    return std::tuple<Tensor, Tensor, Tensor>(self.index_select(dim, indices), inverse_indices, counts);
  }

} // namespace


std::tuple<Tensor, Tensor>
_unique_cuda(const Tensor& self, const bool sorted, const bool return_inverse) {
  return AT_DISPATCH_ALL_TYPES(self.scalar_type(), "unique", [&] {
    // The current CUDA implementation of unique always sort due to the
    // lack of hashtable implementation in thrust
    Tensor output, inverse;
    std::tie(output, inverse, std::ignore) = unique_cuda_template<scalar_t>(self, return_inverse, false);
    return std::make_tuple(output, inverse);
  });
}

std::tuple<Tensor, Tensor, Tensor>
_unique2_cuda(const Tensor& self, const bool sorted, const bool return_inverse, const bool return_counts) {
  return AT_DISPATCH_ALL_TYPES(self.scalar_type(), "unique", [&] {
    // The current CUDA implementation of unique always sort due to the
    // lack of hashtable implementation in thrust
    return unique_cuda_template<scalar_t>(self, return_inverse, return_counts);
  });
}

std::tuple<Tensor, Tensor>
_unique_dim_cuda(const Tensor& self, const int64_t dim, const bool sorted, const bool return_inverse) {
  return AT_DISPATCH_ALL_TYPES(self.scalar_type(), "unique_dim", [&] {
    Tensor output, inverse;
    std::tie(output, inverse, std::ignore) = unique_dim_cuda_template<scalar_t>(self, dim, return_inverse, false);
    return std::make_tuple(output, inverse);
  });
}

std::tuple<Tensor, Tensor, Tensor>
_unique_dim2_cuda(const Tensor& self, const int64_t dim, const bool sorted, const bool return_inverse, const bool return_counts) {
  return AT_DISPATCH_ALL_TYPES(self.scalar_type(), "unique_dim", [&] {
    return unique_dim_cuda_template<scalar_t>(self, dim, return_inverse, return_counts);
  });
}

}  // namespace native
}  // namespace at
