#include "hip/hip_runtime.h"
#include <type_traits>

#include <ATen/ATen.h>
#include <ATen/AccumulateType.h>
#include <ATen/Dispatch.h>
#include <ATen/NestedTensorImpl.h>
#include <ATen/TensorAccessor.h>

#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/detail/KernelUtils.h>
#include <ATen/cuda/detail/IndexUtils.cuh>
#include <ATen/native/cuda/Loops.cuh>
#include <ATen/native/cuda/MemoryAccess.cuh>
#include <ATen/native/cuda/PersistentSoftmax.cuh>
#include <ATen/native/cuda/block_reduce.cuh>

#include <c10/cuda/CUDAMathCompat.h>

#include <ATen/native/nested/NestedTensorUtils.h>
#include <ATen/native/nested/NestedTensorTransformerFunctions.h>

#ifdef USE_FLASH_ATTENTION
#include <ATen/native/transformers/cuda/flash_attn/fmha_api.h>
#include <ATen/native/transformers/cuda/mem_eff_attention/kernel_forward.h>
#endif
namespace at {

namespace native {

namespace {

static constexpr int TRANSFORM_BIAS_RESCALE_VEC = 4;

template <typename scalar_t, typename accscalar_t, bool assume_aligned>
__global__ void transform_bias_rescale_qkv_kernel(
    // [B, T, 3 * D]
    const PackedTensorAccessor64<scalar_t, 3, RestrictPtrTraits> qkv,
    // [3 * D]
    const PackedTensorAccessor64<scalar_t, 1, RestrictPtrTraits> qkv_bias,
    // [3, B, NH, T, DH]
    PackedTensorAccessor64<scalar_t, 5, RestrictPtrTraits> q_k_v,
    const scalar_t inv_sqrt_dim_per_head) {
  // warp per DH.
  // so launch B * NH * T warps.
  auto NH = q_k_v.size(2);
  auto T = q_k_v.size(3);
  auto DH = q_k_v.size(4);

  auto t = blockIdx.x % T;
  auto b = blockIdx.x / T;

  auto D = NH * DH;

  if (assume_aligned) {
    constexpr int VEC = TRANSFORM_BIAS_RESCALE_VEC;
    using LoadT = memory::aligned_vector<scalar_t, VEC>;
    for (int32_t d_v = threadIdx.x; d_v < D / VEC; d_v += blockDim.x) {
      auto d = d_v * VEC;
      auto nh = d / DH;
      auto dh = d % DH;
      scalar_t qkv_bias_q[VEC];
      scalar_t qkv_bias_k[VEC];
      scalar_t qkv_bias_v[VEC];
      scalar_t qkv_q[VEC];
      scalar_t qkv_k[VEC];
      scalar_t qkv_v[VEC];

      // Here we require D % VEC == 0 for these vectorized loads.
      *reinterpret_cast<LoadT*>(&qkv_bias_q) =
          *reinterpret_cast<const LoadT*>(&qkv_bias[d + 0 * D]);
      *reinterpret_cast<LoadT*>(&qkv_bias_k) =
          *reinterpret_cast<const LoadT*>(&qkv_bias[d + 1 * D]);
      *reinterpret_cast<LoadT*>(&qkv_bias_v) =
          *reinterpret_cast<const LoadT*>(&qkv_bias[d + 2 * D]);

      *reinterpret_cast<LoadT*>(&qkv_q) =
          *reinterpret_cast<const LoadT*>(&qkv[b][t][d + 0 * D]);
      *reinterpret_cast<LoadT*>(&qkv_k) =
          *reinterpret_cast<const LoadT*>(&qkv[b][t][d + 1 * D]);
      *reinterpret_cast<LoadT*>(&qkv_v) =
          *reinterpret_cast<const LoadT*>(&qkv[b][t][d + 2 * D]);

#pragma unroll
      // TODO: specialize for float2half2/half2float2?
      for (auto ii = 0; ii < VEC; ++ii) {
        qkv_q[ii] = static_cast<scalar_t>(
            (static_cast<accscalar_t>(qkv_q[ii]) +
             static_cast<accscalar_t>(qkv_bias_q[ii])) *
            static_cast<accscalar_t>(inv_sqrt_dim_per_head));
        qkv_k[ii] = static_cast<scalar_t>(
            (static_cast<accscalar_t>(qkv_k[ii]) +
             static_cast<accscalar_t>(qkv_bias_k[ii])));
        qkv_v[ii] = static_cast<scalar_t>(
            (static_cast<accscalar_t>(qkv_v[ii]) +
             static_cast<accscalar_t>(qkv_bias_v[ii])));
      }

      // Here we require DH % VEC == 0 for these vectorized stores.
      *reinterpret_cast<LoadT*>(&q_k_v[0][b][nh][t][dh]) =
          *reinterpret_cast<const LoadT*>(&qkv_q);
      *reinterpret_cast<LoadT*>(&q_k_v[1][b][nh][t][dh]) =
          *reinterpret_cast<const LoadT*>(&qkv_k);
      *reinterpret_cast<LoadT*>(&q_k_v[2][b][nh][t][dh]) =
          *reinterpret_cast<const LoadT*>(&qkv_v);
    }
  } else {
    // Same as above, but we can't vectorize memory access.
    for (int32_t d = threadIdx.x; d < D; d += blockDim.x) {
      auto nh = d / DH;
      auto dh = d % DH;
      scalar_t qkv_bias_q = qkv_bias[d + 0 * D];
      scalar_t qkv_bias_k = qkv_bias[d + 1 * D];
      scalar_t qkv_bias_v = qkv_bias[d + 2 * D];
      scalar_t qkv_q = qkv[b][t][d + 0 * D];
      scalar_t qkv_k = qkv[b][t][d + 1 * D];
      scalar_t qkv_v = qkv[b][t][d + 2 * D];
      qkv_q = static_cast<scalar_t>(
          (static_cast<accscalar_t>(qkv_q) +
           static_cast<accscalar_t>(qkv_bias_q)) *
          static_cast<accscalar_t>(inv_sqrt_dim_per_head));
      qkv_k = static_cast<scalar_t>(
          (static_cast<accscalar_t>(qkv_k) +
           static_cast<accscalar_t>(qkv_bias_k)));
      qkv_v = static_cast<scalar_t>(
          (static_cast<accscalar_t>(qkv_v) +
           static_cast<accscalar_t>(qkv_bias_v)));

      q_k_v[0][b][nh][t][dh] = qkv_q;
      q_k_v[1][b][nh][t][dh] = qkv_k;
      q_k_v[2][b][nh][t][dh] = qkv_v;
    }
  }
}

template <typename scalar_t, typename accscalar_t, bool assume_aligned = false>
__global__ void transform_bias_rescale_qkv_add_padding_kernel(
    // [B, T, 3 * D], but it's a NestedTensor buffer
    const PackedTensorAccessor64<scalar_t, 1, RestrictPtrTraits> qkv,
    // [3 * D]
    const PackedTensorAccessor64<scalar_t, 1, RestrictPtrTraits> qkv_bias,
    const int* offsets,
    const int* input_sizes,
    // [3, B, NH, T, DH]
    PackedTensorAccessor64<scalar_t, 5, RestrictPtrTraits> q_k_v,
    const scalar_t inv_sqrt_dim_per_head) {
  // warp per DH.
  // so launch B * NH * T warps.
  const auto NH = q_k_v.size(2);
  const auto T = q_k_v.size(3);
  const auto DH = q_k_v.size(4);

  const auto t = blockIdx.x % T;
  const auto b = blockIdx.x / T;

  const auto D = NH * DH;
  const auto _3D = 3 * D;

  const auto offset_for_batch = offsets[b];
  const auto input_dim = 1;
  const auto* sizes_i = input_sizes + b * input_dim;
  if (assume_aligned) {
    constexpr int VEC = TRANSFORM_BIAS_RESCALE_VEC;
    using LoadT = memory::aligned_vector<scalar_t, VEC>;
    for (int32_t d_v = threadIdx.x; d_v < D / VEC; d_v += blockDim.x) {
      auto d = d_v * VEC;
      auto nh = d / DH;
      auto dh = d % DH;
      scalar_t qkv_bias_q[VEC];
      scalar_t qkv_bias_k[VEC];
      scalar_t qkv_bias_v[VEC];
      scalar_t qkv_q[VEC];
      scalar_t qkv_k[VEC];
      scalar_t qkv_v[VEC];

      const auto first_item_offset = t * _3D + d;
      const auto last_item_offset = first_item_offset + VEC - 1;
      const bool first_item_in_bounds = first_item_offset < sizes_i[0];
      const bool entire_vec_in_bounds = last_item_offset < sizes_i[0];

      // Here we require D % VEC == 0 for these vectorized loads.
      *reinterpret_cast<LoadT*>(&qkv_bias_q) =
          *reinterpret_cast<const LoadT*>(&qkv_bias[d + 0 * D]);
      *reinterpret_cast<LoadT*>(&qkv_bias_k) =
          *reinterpret_cast<const LoadT*>(&qkv_bias[d + 1 * D]);
      *reinterpret_cast<LoadT*>(&qkv_bias_v) =
          *reinterpret_cast<const LoadT*>(&qkv_bias[d + 2 * D]);

      if (entire_vec_in_bounds) {
        const auto offset = offset_for_batch + first_item_offset;
        *reinterpret_cast<LoadT*>(&qkv_q) =
            *reinterpret_cast<const LoadT*>(&qkv[offset + 0 * D]);
        *reinterpret_cast<LoadT*>(&qkv_k) =
            *reinterpret_cast<const LoadT*>(&qkv[offset + 1 * D]);
        *reinterpret_cast<LoadT*>(&qkv_v) =
            *reinterpret_cast<const LoadT*>(&qkv[offset + 2 * D]);
#pragma unroll
        // TODO: specialize for float2half2/half2float2?
        for (auto ii = 0; ii < VEC; ++ii) {
          qkv_q[ii] = static_cast<scalar_t>(
              (static_cast<accscalar_t>(qkv_q[ii]) +
               static_cast<accscalar_t>(qkv_bias_q[ii])) *
              static_cast<accscalar_t>(inv_sqrt_dim_per_head));
          qkv_k[ii] = static_cast<scalar_t>(
              (static_cast<accscalar_t>(qkv_k[ii]) +
               static_cast<accscalar_t>(qkv_bias_k[ii])));
          qkv_v[ii] = static_cast<scalar_t>(
              (static_cast<accscalar_t>(qkv_v[ii]) +
               static_cast<accscalar_t>(qkv_bias_v[ii])));
        }
      } else if (first_item_in_bounds) {
        const auto offset = offset_for_batch + first_item_offset;
        qkv_q[0] = qkv[offset + 0 * D];
        qkv_k[0] = qkv[offset + 1 * D];
        qkv_v[0] = qkv[offset + 2 * D];
        qkv_q[0] = static_cast<scalar_t>(
              (static_cast<accscalar_t>(qkv_q[0]) +
               static_cast<accscalar_t>(qkv_bias_q[0])) *
              static_cast<accscalar_t>(inv_sqrt_dim_per_head));
        qkv_k[0] = static_cast<scalar_t>(
            (static_cast<accscalar_t>(qkv_k[0]) +
               static_cast<accscalar_t>(qkv_bias_k[0])));
          qkv_v[0] = static_cast<scalar_t>(
              (static_cast<accscalar_t>(qkv_v[0]) +
               static_cast<accscalar_t>(qkv_bias_v[0])));
#pragma unroll
        for (auto ii = 1; ii < VEC; ++ii) {
          const auto loop_offset = offset + ii;
          if (loop_offset < sizes_i[0]) {
            qkv_q[ii] = qkv[loop_offset + 0 * D];
            qkv_k[ii] = qkv[loop_offset + 1 * D];
            qkv_v[ii] = qkv[loop_offset + 2 * D];
            qkv_q[ii] = static_cast<scalar_t>(
                (static_cast<accscalar_t>(qkv_q[ii]) +
                 static_cast<accscalar_t>(qkv_bias_q[ii])) *
                static_cast<accscalar_t>(inv_sqrt_dim_per_head));
            qkv_k[ii] = static_cast<scalar_t>(
                (static_cast<accscalar_t>(qkv_k[ii]) +
                 static_cast<accscalar_t>(qkv_bias_k[ii])));
            qkv_v[ii] = static_cast<scalar_t>(
                (static_cast<accscalar_t>(qkv_v[ii]) +
                 static_cast<accscalar_t>(qkv_bias_v[ii])));
          } else {
            qkv_q[ii] = 0;
            qkv_k[ii] = 0;
            qkv_v[ii] = 0;
          }
        }
      } else {
#pragma unroll
        for (auto ii = 0; ii < VEC; ++ii) {
          qkv_q[ii] = 0;
          qkv_k[ii] = 0;
          qkv_v[ii] = 0;
        }
      }

      // Here we require DH % VEC == 0 for these vectorized stores.
      *reinterpret_cast<LoadT*>(&q_k_v[0][b][nh][t][dh]) =
          *reinterpret_cast<const LoadT*>(&qkv_q);
      *reinterpret_cast<LoadT*>(&q_k_v[1][b][nh][t][dh]) =
          *reinterpret_cast<const LoadT*>(&qkv_k);
      *reinterpret_cast<LoadT*>(&q_k_v[2][b][nh][t][dh]) =
          *reinterpret_cast<const LoadT*>(&qkv_v);
    }
  } else {
    for (int32_t d = threadIdx.x; d < D; d += blockDim.x) {
      auto nh = d / DH;
      auto dh = d % DH;
      scalar_t qkv_bias_q = qkv_bias[d + 0 * D];
      scalar_t qkv_bias_k = qkv_bias[d + 1 * D];
      scalar_t qkv_bias_v = qkv_bias[d + 2 * D];

      const auto item_offset = t * _3D + d;
      const bool in_bounds = item_offset < sizes_i[0];
      scalar_t qkv_q, qkv_k, qkv_v;
      if (in_bounds) {
        const auto qkv_offset = offset_for_batch + item_offset;
        qkv_q = qkv[qkv_offset + 0 * D];
        qkv_k = qkv[qkv_offset + 1 * D];
        qkv_v = qkv[qkv_offset + 2 * D];
        qkv_q = static_cast<scalar_t>(
            (static_cast<accscalar_t>(qkv_q) +
             static_cast<accscalar_t>(qkv_bias_q)) *
            static_cast<accscalar_t>(inv_sqrt_dim_per_head));
        qkv_k = static_cast<scalar_t>(
            (static_cast<accscalar_t>(qkv_k) +
             static_cast<accscalar_t>(qkv_bias_k)));
        qkv_v = static_cast<scalar_t>(
            (static_cast<accscalar_t>(qkv_v) +
             static_cast<accscalar_t>(qkv_bias_v)));
      } else {
        qkv_q = 0;
        qkv_k = 0;
        qkv_v = 0;
      }

      q_k_v[0][b][nh][t][dh] = qkv_q;
      q_k_v[1][b][nh][t][dh] = qkv_k;
      q_k_v[2][b][nh][t][dh] = qkv_v;
    }
  }
}

Tensor collapse_dims_1_and_2(const Tensor& sizes) {
  auto sizes_dim1 = at::native::narrow(sizes, 1, 0, 1);
  auto sizes_dim2 = at::native::narrow(sizes, 1, 1, 1);

  return (sizes_dim1 * sizes_dim2).contiguous();
}

} // namespace
// compute q = (q + q_bias) / sqrt(dim_per_head), k = k + k_bias, v = v + v_bias
__host__ std::tuple<Tensor, Tensor, Tensor> transform_bias_rescale_qkv_cuda(
    const Tensor& qkv,
    const Tensor& qkv_bias,
    const int64_t num_head) {
  auto B = qkv.is_nested()
      ? get_nested_tensor_impl(qkv)->get_nested_size_tensor().size(0)
      : qkv.size(0);
  // TODO: calculate this without the std::vector -- NestedTensor_to_mask wants
  // this too
  auto T = qkv.is_nested()
      ? NestedTensor_get_max_size(*get_nested_tensor_impl(qkv))[0]
      : qkv.size(1);
  if (qkv.is_nested()) {
    // Don't mess with non-nested case for now since it's not set up to fiddle
    // with mask size.

    // Round T up to next multiple of 8 so as to be able to utilize Tensor
    // cores. Otherwise, sometimes with padding, *no* row will have the maximum
    // sequence length and so we'll have a non-divisible-by-8 dimension even if
    // the model author chose a multiple of 8.
    T = T + (8 - (T % 8)) % 8;
  }
  auto _3D = qkv_bias.size(0);
  auto D = _3D / 3;
  TORCH_CHECK(D % num_head == 0);
  const auto dim_per_head = D / num_head;
  auto q_k_v = at::empty({3, B, num_head, T, dim_per_head}, qkv_bias.options());
#define CALL_KERNEL(assume_aligned)                                        \
  transform_bias_rescale_qkv_kernel<scalar_t, accscalar_t, assume_aligned> \
      <<<blocks, threads, 0, at::cuda::getCurrentCUDAStream()>>>(          \
          qkv.packed_accessor64<scalar_t, 3, RestrictPtrTraits>(),         \
          qkv_bias.packed_accessor64<scalar_t, 1, RestrictPtrTraits>(),    \
          q_k_v.packed_accessor64<scalar_t, 5, RestrictPtrTraits>(),       \
          1.0 / std::sqrt(static_cast<scalar_t>(dim_per_head)))
#define CALL_ADD_PADDING_KERNEL(assume_aligned)                         \
  transform_bias_rescale_qkv_add_padding_kernel<                        \
      scalar_t,                                                         \
      accscalar_t,                                                      \
      assume_aligned>                                                   \
      <<<blocks, threads, 0, at::cuda::getCurrentCUDAStream()>>>(       \
          nt_qkv_buffer                                          \
              .packed_accessor64<scalar_t, 1, RestrictPtrTraits>(),     \
          qkv_bias.packed_accessor64<scalar_t, 1, RestrictPtrTraits>(), \
          offsets_ptr,                                                  \
          sizes_ptr,                                                    \
          q_k_v.packed_accessor64<scalar_t, 5, RestrictPtrTraits>(),    \
          1.0 / std::sqrt(static_cast<scalar_t>(dim_per_head)))

  AT_DISPATCH_FLOATING_TYPES_AND2(
      ScalarType::Half,
      ScalarType::BFloat16,
      qkv.scalar_type(),
      "transform_bias_rescale_qkv",
      [&] {
        using accscalar_t = acc_type<scalar_t, true>;
        auto threads = std::max(
            std::min<int32_t>(1024, D / TRANSFORM_BIAS_RESCALE_VEC), 1);
        auto blocks = B * T;
        const bool aligned =
            ((dim_per_head % TRANSFORM_BIAS_RESCALE_VEC) == 0) &&
            ((reinterpret_cast<intptr_t>(qkv_bias.data_ptr()) %
              TRANSFORM_BIAS_RESCALE_VEC) == 0);
        if (aligned) {
          TORCH_INTERNAL_ASSERT_DEBUG_ONLY(
              D % TRANSFORM_BIAS_RESCALE_VEC == 0,
              "D = num_heads * dim_per_head, so we should have dim_per_head % "
              "TRANSFORM_BIAS_RESCALE_VEC == 0 => "
              "D % TRANSFORM_BIAS_RESCALE_VEC == 0");
        }
        if (qkv.is_nested()) {
          auto* nt_qkv = get_nested_tensor_impl(qkv);
          const at::Tensor& nt_qkv_buffer = nt_qkv->get_buffer();
          auto sizes = collapse_dims_1_and_2(nt_qkv->get_nested_size_tensor());
          auto offsets =
              NestedTensor_batch_offsets_from_size_tensor(sizes, sizes.numel());
          at::native::narrow(offsets, 0, sizes.numel() + 1, sizes.numel())
              .copy_(sizes.reshape({-1}));
          auto metadata = offsets.to(at::Device(kCUDA), at::kInt, true, true);
          const auto offsets_ptr = metadata.data_ptr<int>();
          const auto sizes_ptr = offsets_ptr + sizes.numel() + 1;
          const auto input_dim = sizes.sizes()[1];
          TORCH_INTERNAL_ASSERT_DEBUG_ONLY(input_dim == 1);
          if (aligned &&
              ((reinterpret_cast<intptr_t>(qkv.data_ptr()) %
                TRANSFORM_BIAS_RESCALE_VEC) == 0)) {
            CALL_ADD_PADDING_KERNEL(true);
          } else {
            CALL_ADD_PADDING_KERNEL(false);
          }
        } else if (aligned) {
          CALL_KERNEL(true);
        } else {
          CALL_KERNEL(false);
        }
        C10_CUDA_KERNEL_LAUNCH_CHECK();
      });
#undef CALL_ADD_PADDING_KERNEL
#undef CALL_KERNEL
  auto q_k_v_s =
      at::native::split(q_k_v.view({3 * B, num_head, T, dim_per_head}), B, 0);
  return std::make_tuple(q_k_v_s[0], q_k_v_s[1], q_k_v_s[2]);
}

Tensor triton_scaled_dot_attention(const Tensor& q, const Tensor& k, const Tensor& v, double dropout_p){
  TORCH_CHECK(false, "This operator should be overridden in python before use");
  return at::Tensor();
}

Tensor flash_scaled_dot_product_attention(
    const Tensor& query,
    const Tensor& key,
    const Tensor& value,
    const Tensor& cumulative_sequence_length_q,
    const Tensor& cumulative_sequence_length_k,
    const int64_t max_seqlen_batch_q,
    const int64_t max_seqlen_batch_k,
    double dropout_p,
    bool causal) {
#if defined(USE_FLASH_ATTENTION)
  auto softmax_scale = std::pow(query.size(-1), -0.5);
  std::vector<Tensor> output = fmha::mha_fwd(
      query,
      key,
      value,
      cumulative_sequence_length_q,
      cumulative_sequence_length_k,
      max_seqlen_batch_q,
      max_seqlen_batch_k,
      dropout_p,
      softmax_scale,
      false,
      causal,
      false,
      c10::nullopt);
  return output[0];
#endif
  TORCH_CHECK(false, "USE_FLASH_ATTENTION was not enabled for build.")
  return Tensor{};
}

std::tuple<at::Tensor, at::Tensor> efficient_attention_forward_generic(
    const at::Tensor& query,
    const at::Tensor& key,
    const at::Tensor& value,
    bool compute_logsumexp,
    const c10::optional<at::Tensor>& attn_bias_,
    double p,
    bool causal) {
#if defined(USE_FLASH_ATTENTION)
  TORCH_CHECK(p == 0.0, "Dropout is not supported at the moment");
  TORCH_CHECK(
      !attn_bias_.has_value(), "attn_bias is not supported at the moment");

  TORCH_CHECK(query.dim() == 3);
  TORCH_CHECK(key.dim() == 3);
  TORCH_CHECK(value.dim() == 3);

  TORCH_CHECK(query.size(2) == key.size(2));
  TORCH_CHECK(query.size(0) == key.size(0));

  TORCH_CHECK(query.is_cuda(), "query must be a CUDA tensor");
  TORCH_CHECK(key.is_cuda(), "key must be a CUDA tensor");
  TORCH_CHECK(value.is_cuda(), "value must be a CUDA tensor");

  TORCH_CHECK(!query.is_sparse(), "query must be a dense tensor");
  TORCH_CHECK(!key.is_sparse(), "key must be a dense tensor");
  TORCH_CHECK(!value.is_sparse(), "value must be a dense tensor");

  TORCH_CHECK(query.is_contiguous());
  TORCH_CHECK(key.is_contiguous());
  TORCH_CHECK(value.is_contiguous());

  at::cuda::CUDAGuard device_guard(query.device());
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  int64_t B = query.size(0);
  int64_t M = query.size(1);
  int64_t N = key.size(1);
  int64_t K = query.size(2);

  at::Tensor res;
  at::Tensor logsumexp;

  hipDeviceProp_t* properties =
      at::cuda::getDeviceProperties(query.device().index());
  const int computeCapability = properties->major * 10 + properties->minor;

#define DISPATCH_ARCHTAG(func)                                            \
  {                                                                       \
    if (computeCapability >= 80) {                                        \
      using ArchTag = cutlass::arch::Sm80;                                \
      func();                                                             \
    } else if (computeCapability >= 75) {                                 \
      using ArchTag = cutlass::arch::Sm75;                                \
      func();                                                             \
    } else if (computeCapability >= 70) {                                 \
      using ArchTag = cutlass::arch::Sm70;                                \
      func();                                                             \
    } else if (computeCapability >= 50) {                                 \
      using ArchTag = cutlass::arch::Sm50;                                \
      func();                                                             \
    } else {                                                              \
      TORCH_CHECK(                                                        \
          false,                                                          \
          "Your device is too old. We require compute capability >= 50"); \
    }                                                                     \
  }
// Dispatch to the right kernel
#define DISPATCH_TYPES(func)                                          \
  {                                                                   \
    if (query.scalar_type() == at::ScalarType::Float) {               \
      using scalar_t = float;                                         \
      func();                                                         \
    } else if (query.scalar_type() == at::ScalarType::Half) {         \
      using scalar_t = cutlass::half_t;                               \
      func();                                                         \
    } else {                                                          \
      TORCH_CHECK(false, "Only fp32 & half supported at the moment"); \
    }                                                                 \
  }

#define DISPATCH_BLOCKSIZE(VALUE_HEAD_DIM, BLOCK_6464, SINGLE_VALUE_ITER, FN) \
  {                                                                           \
    if (VALUE_HEAD_DIM <= 64) {                                               \
      constexpr bool BLOCK_6464 = true;                                       \
      constexpr bool SINGLE_VALUE_ITER = true;                                \
      FN();                                                                   \
    } else {                                                                  \
      constexpr bool BLOCK_6464 = false;                                      \
      if (VALUE_HEAD_DIM <= 128) {                                            \
        constexpr bool SINGLE_VALUE_ITER = true;                              \
        FN();                                                                 \
      } else {                                                                \
        constexpr bool SINGLE_VALUE_ITER = false;                             \
        FN();                                                                 \
      }                                                                       \
    }                                                                         \
  }

  DISPATCH_BLOCKSIZE(
      value.size(2), kIs64x64, kSingleValueIteration, ([&]() {
        static constexpr int64_t kQueriesPerBlock = kIs64x64 ? 64 : 32;
        static constexpr int64_t kKeysPerBlock = kIs64x64 ? 64 : 128;
        DISPATCH_TYPES(([&]() {
          DISPATCH_ARCHTAG(([&]() {
            // Run a more efficient kernel (with `isAligned=True`) if
            // memory is correctly aligned
            bool isAligned;
            using AlignedAK = AttentionKernel<
                scalar_t,
                ArchTag,
                true,
                kQueriesPerBlock,
                kKeysPerBlock,
                kSingleValueIteration>;
            isAligned =
                (query.stride(1) % AlignedAK::kAlignmentQ == 0 &&
                 key.stride(1) % AlignedAK::kAlignmentK == 0 &&
                 value.stride(1) % AlignedAK::kAlignmentV == 0);
            // TODO: Should we warn or log somewhere when we use a less
            // efficient kernel due to wrong alignment?
            DISPATCH_BOOL(
                isAligned, kIsAligned, ([&]() {
                  using Kernel = AttentionKernel<
                      scalar_t,
                      ArchTag,
                      kIsAligned,
                      kQueriesPerBlock,
                      kKeysPerBlock,
                      kSingleValueIteration>;
                  // Might happen on Sm80/half, where the minimum
                  // alignment is 32bits
                  TORCH_CHECK(
                      query.stride(1) % Kernel::kAlignmentQ == 0,
                      "query is not correctly aligned");
                  TORCH_CHECK(
                      key.stride(1) % Kernel::kAlignmentK == 0,
                      "key is not correctly aligned");
                  TORCH_CHECK(
                      value.stride(1) % Kernel::kAlignmentV == 0,
                      "value is not correctly aligned");

                  res = at::zeros(
                      {B, M, K},
                      query.options().dtype(
                          TypeTraits<
                              typename Kernel::output_t>::atScalarType()));
                  // NOTE: Should be aligned (by padding) in case M is not
                  // a good number for loading during backward
                  constexpr decltype(M) kAlignLSE =
                      32; // block size of backward
                  logsumexp = at::empty(
                      {B,
                       compute_logsumexp ? ceil_div(M, kAlignLSE) * kAlignLSE
                                         : 0},
                      query.options().dtype(at::ScalarType::Float));

                  constexpr auto kernel_fn = attention_kernel_batched<Kernel>;
                  size_t smem_bytes = sizeof(typename Kernel::SharedStorage);
                  if (smem_bytes > 0xc000) {
                    TORCH_INTERNAL_ASSERT(
                        computeCapability >= 70,
                        "This kernel requires too much shared memory on this machine!");
                    AT_CUDA_CHECK(hipFuncSetAttribute(reinterpret_cast<const void*>(
                        kernel_fn),
                        hipFuncAttributeMaxDynamicSharedMemorySize,
                        smem_bytes));
                  }

                  typename Kernel::Params p;
                  p.query_ptr = (scalar_t*)query.data_ptr();
                  p.key_ptr = (scalar_t*)key.data_ptr();
                  p.value_ptr = (scalar_t*)value.data_ptr();
                  p.logsumexp_ptr = compute_logsumexp
                      ? (typename Kernel::lse_scalar_t*)logsumexp.data_ptr()
                      : nullptr;
                  p.output_ptr = (typename Kernel::output_t*)res.data_ptr();
                  p.head_dim = query.size(2);
                  p.head_dim_value = value.size(2);
                  p.num_queries = query.size(1);
                  p.num_keys = key.size(1);
                  p.num_batches = B;
                  p.causal = causal;
                  kernel_fn<<<
                      p.getBlocksGrid(),
                      p.getThreadsGrid(),
                      smem_bytes>>>(p);
                }));
          }));
        }));
      }));

  AT_CUDA_CHECK(hipGetLastError());
  return std::make_tuple(res, logsumexp);
#endif
  TORCH_CHECK(false, "USE_FLASH_ATTENTION was not enabled for build.")
  return std::make_tuple(Tensor{}, Tensor{});
}

} // namespace native
} // namespace at
